#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hipsparse.h"
#include "hip/hip_runtime.h"
#include "memory.h"
#include "data_type.h"
#include <iostream>
#include <ctime>
#include <fstream>
#include <vector>
#include <Windows.h>
#include <iomanip>
#include "help_func.cuh"
#include "pre_process.cuh"
#include <thrust/device_vector.h>
#include <thrust/copy.h>

#pragma comment(lib,"cusparse.lib")
#pragma comment(lib,"cublas.lib")
using namespace std;

extern __global__ void standardAndThresholdingKernel(real__t* devCormat, int Batch_size, bool diagnoal, double thres);

extern __global__ void initialone(real__t *vec,int N);

extern void updataPointer(real__t** pointer, real__t* devBOLD, int L, int Batch_size, const int Transferblocknum);

extern void updateDevBOLD(int jj, real__t* devBOLD, real__t* BOLD_t, int L, int Batch_size, int* hosPoint, int* count, real__t* pointer, const int Num_Blocks);

/*************************************************************************************************/
/* GPU-based Cormat function,
output the CSR format connectivity matrix,
and functional connectivity strength.*/
/*************************************************************************************************/
int CorMat_gpu_blocking(string OutCor, real__t * BOLD_t, const int &N, const int &N0, const int &Num_Blocks, const int &L, const int &Batch_size, real__t* r_thresh, const int &NumS, const int Transferblocknum)
{			
	size_t L2 = L * ( L - 1 ) / 2;
	/*************************************************************************************************/
	/*							  Setup CUBLAS and CUSPARSE parameters                               */
	/*************************************************************************************************/
	//cout<<"*r_thresh:"<<*r_thresh<<endl;
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipsparseStatus_t  sparseStat;
	hipblasHandle_t handle;
	hipsparseHandle_t sparseHandle;
	//hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;
	hipsparseMatDescr_t descrA = 0;
	sparseStat= hipsparseCreate(&sparseHandle);
    if (sparseStat != HIPSPARSE_STATUS_SUCCESS)
		return sparseStat;
	
	/* create and setup matrix descriptor */ 
	sparseStat= hipsparseCreateMatDescr(&descrA); 
	if (sparseStat != HIPSPARSE_STATUS_SUCCESS) 
		return sparseStat; 
	hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO);  
	//sparseStat = cusparse_create_mat_descr(descrA); 
	
	/*****************     Correlation matrix variables for each block   ******************/
	/*          GPU variables        */
	real__t * devBOLD, * devCormat, * devBOLD_x, * devBOLD_y, * csrValA;
	size_t * tiecount;
	real__t * pointer;
	int count = 0; //used to control pointer
	int hosPoint = Transferblocknum - 1;
	int *nnzPerRowColumn,*csrRowPtrA,*csrColIndA;
	checkCudaErrors (hipMalloc ((void**)&nnzPerRowColumn, sizeof(int) * Batch_size)) ;
	checkCudaErrors (hipMalloc ((void**)&csrRowPtrA, sizeof(int) * (Batch_size+1)) ) ;
	checkCudaErrors (hipMalloc ((void**)&devBOLD, sizeof(real__t) * L * Batch_size * Transferblocknum)) ;
	checkCudaErrors (hipMalloc ( (void**)&devCormat, sizeof(real__t) * Batch_size * Batch_size)) ;
	checkCudaErrors ( hipMalloc ((void**)&tiecount, sizeof(size_t) * N0) );
	checkCudaErrors ( hipMalloc ((void**)&devBOLD_x, sizeof(real__t) * L2 * Batch_size) );
	checkCudaErrors ( hipMalloc ((void**)&devBOLD_y, sizeof(real__t) * L2 * Batch_size) );
	checkCudaErrors( hipMemcpy(devBOLD, BOLD_t, sizeof(real__t) * L * Batch_size * Transferblocknum, hipMemcpyHostToDevice) );
	pointer = devBOLD;
	stat = hipblasCreate(&handle) ;
	if (stat != HIPBLAS_STATUS_SUCCESS)
		return stat;
	
	/*          CPU variables        */
	uint__t Overall_Num_Blocks = Num_Blocks * Num_Blocks;
	uint__t **Column = new uint__t* [Overall_Num_Blocks];
	real__t **Value = new real__t* [Overall_Num_Blocks];
	uint__t **Rown = new uint__t* [Overall_Num_Blocks];
	//int *nnzOfEachBlock = new int [Overall_Num_Blocks];
	R_type totalNonzero = 0;
			
	/****************  Functional connectivity strength variables  ****************/
	real__t *fcsGPU,*vec;
	
	checkCudaErrors (hipMalloc ((void**)&fcsGPU, sizeof(real__t) * N0)) ;
	checkCudaErrors (hipMemset(fcsGPU,0,sizeof(real__t) * N0));
	checkCudaErrors (hipMalloc ((void**)&vec, sizeof(real__t) * Batch_size)) ;
	
	initialone<<<block_num,thread_num>>>(vec,Batch_size);			                                       

	cout<<"matrix block number: "<<Num_Blocks<<endl;
	//const float gamma = 1.0;
	
	clock_t correlationTime = clock();
		
	/*************************************************************************************************/
	/*						         Start the correlation computation                               */
	/*************************************************************************************************/
    bool flag = true;
	size_t shareSize = sizeof(real__t) * L + sizeof(size_t) * L;
	dim3 Grid(Batch_size/thread_num2D, Batch_size/thread_num2D), Block(thread_num2D,thread_num2D);
	real__t* biPointer =  devBOLD_y;
	for (int ii = 0; ii < Num_Blocks; ii++)
	{
#ifdef myDebug
		cout<<"loop flag:"<<ii<<" "<<endl;
#endif
		if ( ii > 0 )
			flag = false;
		for (int jj = ii; jj < Num_Blocks; jj++)
		{
			//1. Matrix multiplication																	//very different from step1!											
			biPointer = ii == jj ? devBOLD_y : devBOLD_x;
#ifdef figure
			int thread_num = L;
			int block_num = 180;
#endif
			pre_process <<<block_num, thread_num, shareSize>>>(biPointer, pointer, L, L2, Batch_size, tiecount + jj * Batch_size, flag);
			checkCublasErrors( hipblasSgemm(handle, HIPBLAS_OP_T,  HIPBLAS_OP_N, Batch_size, Batch_size, L2,  &alpha, devBOLD_y, L2, biPointer, L2, &beta, devCormat, Batch_size) ); //Is this really right?
			dividedByDenominator<<<Grid, Block>>>(devCormat, Batch_size, L, tiecount + ii * Batch_size, tiecount + jj * Batch_size);
			updateDevBOLD(jj, devBOLD, BOLD_t, L, Batch_size, &hosPoint, &count, pointer, Num_Blocks); //precedence order need to be gone by
			updataPointer(&pointer, devBOLD, L, Batch_size, Transferblocknum);
			//gpuOutput(devCormat, sizeof(real__t) * Batch_size * Batch_size, OutCor, true);
#ifdef myDebug
			/*if (jj == 5)
			{
				cout<<"ii jj:"<<ii<<" "<<jj<<endl;
				thrust::copy(thrust::device_pointer_cast(devCormat + Batch_size * 837 ), thrust::device_pointer_cast(devCormat + Batch_size * 837 + 10), std::ostream_iterator<real__t>(std::cout, " "));
				cout<<"breakpoint"<<endl;	
			}*/
#endif

			//Calculating FCS
			standardAndThresholdingKernel<<<block_num, thread_num>>>(devCormat, Batch_size, ii==jj,0);  
#ifdef myDebug
			gpuOutput(devCormat, sizeof(real__t) * Batch_size * Batch_size, OutCor, false);	
			cout<<"loop flag:"<<ii<<" "<<jj<<endl;
#endif
			stat = hipblasSgemv(handle, HIPBLAS_OP_N, Batch_size, Batch_size, &alpha, devCormat, Batch_size, vec, 1, &alpha, fcsGPU + ii * Batch_size, 1);
			if (stat != HIPBLAS_STATUS_SUCCESS)
				return stat;					
			if (ii!=jj)
			{
				stat = hipblasSgemv(handle, HIPBLAS_OP_T, Batch_size, Batch_size, &alpha, devCormat, Batch_size, vec, 1, &alpha, fcsGPU + jj * Batch_size, 1);
				if (stat != HIPBLAS_STATUS_SUCCESS)
				return stat;
			}
//#ifdef myDebug
//			gpuOutput(devCormat, sizeof(real__t) * Batch_size * Batch_size, OutCor, false);
//			cout<<"ii jj:"<<ii<<" "<<jj<<endl;
//#endif
			//2.thresholding	
#ifdef myLiteDebug
			//r_thresh[0] = 0.381512;
#endif
			standardAndThresholdingKernel<<<block_num,thread_num>>>(devCormat, Batch_size, ii==jj, r_thresh[0]);
			//standardAndThresholdingKernel<<<block_num,thread_num>>>(devCormat, Batch_size, ii==jj, 0);
			//3. dense2csr
			int nnzTotalDevHostPtr = 0;
			sparseStat = hipsparseSnnz(sparseHandle, HIPSPARSE_DIRECTION_ROW, Batch_size, Batch_size, descrA, devCormat,  Batch_size, nnzPerRowColumn, &nnzTotalDevHostPtr);
			if (sparseStat !=HIPSPARSE_STATUS_SUCCESS)
				return sparseStat;
			//nnzOfEachBlock[ii * Num_Blocks + jj] = nnzTotalDevHostPtr;
			//if (ii!=jj)
			//	nnzOfEachBlock[jj * Num_Blocks + ii] = nnzTotalDevHostPtr;
			
			Column[ii * Num_Blocks + jj] = new uint__t [nnzTotalDevHostPtr];
			Value[ii * Num_Blocks + jj] = new real__t [nnzTotalDevHostPtr];
			Rown[ii * Num_Blocks + jj] = new uint__t [Batch_size+1];
			if (ii!=jj)
			{
				Column[jj * Num_Blocks + ii] = new uint__t [nnzTotalDevHostPtr];
				Value[jj * Num_Blocks + ii] = new real__t [nnzTotalDevHostPtr];
				Rown[jj * Num_Blocks + ii] = new uint__t [Batch_size+1];
			}
			if (nnzTotalDevHostPtr==0)
			{
				//Rown[ii * Num_Blocks + jj] = new uint__t [Batch_size+1];
				for (int i = 0; i < (Batch_size + 1); i++)
				{
					Rown[ii * Num_Blocks + jj][i] = 0;
				}
				if (ii!=jj)
				{
					//Rown[jj * Num_Blocks + ii] = new uint__t [Batch_size+1];
					for (int i = 0; i < (Batch_size + 1); i++)
					{
						Rown[jj * Num_Blocks + ii][i] = 0;
					}
				}
				continue;
			}
			//malloc GPU csr column index and value,cusparseSdens2csr
			checkCudaErrors (hipMalloc ((void**)&csrValA, sizeof(real__t) * nnzTotalDevHostPtr)) ;
			checkCudaErrors (hipMalloc ((void**)&csrColIndA, sizeof(int) * nnzTotalDevHostPtr)) ;
			
			sparseStat = hipsparseSdense2csr(sparseHandle, Batch_size, Batch_size, descrA, devCormat, Batch_size, nnzPerRowColumn, csrValA, csrRowPtrA, csrColIndA);
			if (sparseStat != HIPSPARSE_STATUS_SUCCESS)
				return sparseStat;

			//3. transfer			
			checkCudaErrors (hipMemcpy(Column[ii * Num_Blocks + jj], csrColIndA, sizeof(int) * nnzTotalDevHostPtr, hipMemcpyDeviceToHost));
			checkCudaErrors (hipMemcpy(Value[ii * Num_Blocks + jj], csrValA, sizeof(real__t) * nnzTotalDevHostPtr, hipMemcpyDeviceToHost));
			checkCudaErrors (hipMemcpy(Rown[ii * Num_Blocks + jj], csrRowPtrA, sizeof(int) * (Batch_size+1), hipMemcpyDeviceToHost));
						
			if(Rown[ii * Num_Blocks + jj][Batch_size]!=nnzTotalDevHostPtr)
			{
				cout<<"checking error diagnoal:"<<Rown[ii * Num_Blocks + jj][Batch_size]<<endl;
			}

			//need transposition if ii!=jj
			if (ii!=jj)
			{
				real__t *cscVal;
				int *cscRowInd, *cscColPtr;
				
				checkCudaErrors (hipMalloc ((void**)&cscVal, sizeof(real__t) * nnzTotalDevHostPtr)) ;
				checkCudaErrors (hipMalloc ((void**)&cscColPtr, sizeof(int) * (Batch_size + 1))) ; //transposed R
				checkCudaErrors (hipMalloc ((void**)&cscRowInd, sizeof(int) *  nnzTotalDevHostPtr)) ; //transposed C
				
				sparseStat = hipsparseScsr2csc(sparseHandle, Batch_size, Batch_size, nnzTotalDevHostPtr, csrValA, csrRowPtrA, csrColIndA, cscVal, cscRowInd, cscColPtr, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO);
				if (sparseStat !=HIPSPARSE_STATUS_SUCCESS)
					return sparseStat;
									
				checkCudaErrors (hipMemcpy(Column[jj * Num_Blocks + ii], cscRowInd, sizeof(int) * nnzTotalDevHostPtr, hipMemcpyDeviceToHost));
				checkCudaErrors (hipMemcpy(Value[jj * Num_Blocks + ii], cscVal, sizeof(real__t) * nnzTotalDevHostPtr, hipMemcpyDeviceToHost));
				checkCudaErrors (hipMemcpy(Rown[jj * Num_Blocks + ii], cscColPtr, sizeof(int) * (Batch_size+1), hipMemcpyDeviceToHost));
				
				if (Rown[jj * Num_Blocks + ii][Batch_size]!=nnzTotalDevHostPtr)
				{
					cout<<"checking error non-diagnoal:"<<Rown[jj * Num_Blocks + ii][Batch_size]<<endl;
				}
				/*	for (int i = 0; i < nnzTotalDevHostPtr; i++)
					{
						if (Value[jj * Num_Blocks + ii][i] == 0)
						{
							cout<<"How come?"<<endl;
						}
					}*/
				checkCudaErrors (hipFree(cscVal));
				checkCudaErrors (hipFree(cscRowInd));
				checkCudaErrors (hipFree(cscColPtr));
			}

			if(ii==jj)
				totalNonzero += nnzTotalDevHostPtr;
			else
			{
				totalNonzero += nnzTotalDevHostPtr * 2 ;
			}

			//4.1 free GPU CSR column index and value.
			checkCudaErrors (hipFree(csrValA));
			checkCudaErrors (hipFree(csrColIndA));
		}
		//cout<<"Fulfill the "<<ii+1<<"th disposition."<<endl;
	}

	//4.2 free nnzPerRowColumn and csrRowPtrA.
	checkCudaErrors (hipFree(nnzPerRowColumn));
	checkCudaErrors (hipFree(csrRowPtrA));

	sparseStat = hipsparseDestroyMatDescr(descrA);
	if (sparseStat != HIPSPARSE_STATUS_SUCCESS)
		return sparseStat;
	sparseStat = hipsparseDestroy(sparseHandle);
	if (sparseStat != HIPSPARSE_STATUS_SUCCESS)
		return sparseStat;

	correlationTime = clock() - correlationTime;
	cout<<"correlation time: "<<correlationTime<<"ms"<<endl;
	//cout<<"overall time for histogram plus correlation: "<<*aggregrate<<"ms"<<endl;

	/****************    Write FCS information   ****************/
	real__t *fcs = new real__t[N];
	memset(fcs,0,sizeof(real__t)*N);	
	checkCudaErrors (hipMemcpy(fcs, fcsGPU, sizeof(real__t) * N, hipMemcpyDeviceToHost));
	
	ofstream fcs_fout;
	string fcs_out_str = OutCor;
	fcs_out_str.append("_fcs.nm");
	fcs_fout.open(fcs_out_str.c_str(), ios::binary | ios::out);
	if (!fcs_fout)
	{
		cout<<"create unsuccessfully. error code:  "<<GetLastError()<<endl;
		exit(false);
	}
	int length = N;
	fcs_fout.write((char*)&length, sizeof(int));
	for (int i = 0; i < N; i++)
	{
		fcs_fout.write((char*)&fcs[i], sizeof(real__t));
	}
	fcs_fout.close();
	delete[] fcs;
	checkCudaErrors (hipFree(fcsGPU));
	checkCudaErrors (hipFree(vec));

		
	/************************** multiple thresholds ***************************/
	
	//5. vector Column Index and Value
	R_type *Row = new R_type[N+1];
	memset(Row,0,sizeof(R_type)*(N+1));
	vector<C_type> C;
	vector<V_type> V;

	//first r_threshold
	
	//R_type Rcheck = 0;
	/*for (uint__t ii = 0; ii < Num_Blocks; ii++)
	{
		for (uint__t jj = 0; jj < Num_Blocks; jj++)
		{
			Rcheck += Rown[ii*Num_Blocks+jj][Batch_size];
			for (uint__t x = 0; x < Batch_size+1; x++)
			{
				Row[x+ii*Batch_size] += Rown[ii*Num_Blocks+jj][x];
			}
		}
		for (uint__t y = ii*Batch_size+Batch_size+1; y < (Num_Blocks * Batch_size+1); y++)
		{
			Row[y] = Row[Batch_size+ii*Batch_size];
		}
	}*/

			
	
	if ( totalNonzero > C.max_size() )
	{
		cout<<"error:"<<"Vector max_size exceeds!"<<endl;
		return false;
	}	

	C.reserve(totalNonzero);
	V.reserve(totalNonzero);
	
	for (int ii = 0; ii < Num_Blocks; ii++)
	{
		for (int i = 0; i < Batch_size && ii*Batch_size+i < N; i++)
		{			
			for (int jj = 0; jj < Num_Blocks; jj++)
			{
				if (Rown[ii*Num_Blocks+jj][i]==Rown[ii*Num_Blocks+jj][i+1])
					continue;
				else
					for (uint__t j = Rown[ii*Num_Blocks+jj][i]; j < Rown[ii*Num_Blocks+jj][i+1]; j++)
					{
						C.push_back(Column[ii*Num_Blocks+jj][j] + (C_type) jj*Batch_size);
						V.push_back(Value[ii*Num_Blocks+jj][j]);						
					}
			}		
			Row[ii*Batch_size+i+1] = C.size();			
		}
		for (int jj = 0; jj < Num_Blocks; jj++)
		{
			delete[] Rown[ii*Num_Blocks+jj];
			delete[] Column[ii*Num_Blocks+jj];
			delete[] Value[ii*Num_Blocks+jj];
		}
	}

	delete[] Rown;
	delete[] Column;
	delete[] Value;
	
	
	cout<<"Row[N]:"<<Row[N]<<endl;
	if (Row[N] != totalNonzero )
	{
		cout<<"error:"<<"R values abnormal!"<<endl;
		return false;
	}
	//checking point!
	if (Row[N] != C.size() || Row[N] != V.size() )
	{
		cout<<"error:"<<"R values abnormal!"<<endl;
		return false;
	}

	MEMORYSTATUS MemStat;
	MemStat.dwLength = sizeof(MEMORYSTATUS);
	GlobalMemoryStatus(&MemStat);	
	cout << "bytes of physical memory: " << TOM(MemStat.dwTotalPhys) <<"M" <<endl;
	cout << "percent of memory in use: " << MemStat.dwMemoryLoad <<"%" <<endl;
	cout << "free physical memory bytes: " << TOM(MemStat.dwAvailPhys) <<"M" <<endl;
	cout<<"number of non-zero elements: "<<Row[N]<<endl;	
	cout<<"Transmition finished."<<endl;
	
	long long M1 = (N-1);
	M1 *= N;	
	real__t spa = 100.0 * Row[N] / M1;
	cout<<"sparsity: "<<spa<<endl;
	char sparsity[30];
	sprintf(sparsity, "_spa%.3f%%_cor%.3f", spa, r_thresh[0]);
	string Outfilename = OutCor;
	Outfilename.append(string(sparsity)).append("_weighted.csr");
	ofstream fout;
	cout<<"generating "<<Outfilename.c_str()<< "..."<<endl;
	fout.open(Outfilename.c_str(), ios::binary | ios::out);
	if (!fout)
	{
		cout<<"create outfile unsuccessfully. error code:  "<<GetLastError()<<endl;
		exit(false);
	}	
#ifdef figure
	uint__t Rlength = N+1;
	fout.write((const char*)&Rlength, sizeof(uint__t));
	fout.write((const char*)Row, sizeof(R_type)*Rlength);
	R_type nnzlength = C.size();
	fout.write((const char*)&nnzlength, sizeof(R_type));
	fout.write((const char*)&C[0],sizeof(C_type)*nnzlength);
	fout.write((const char*)&nnzlength, sizeof(R_type));
	fout.write((const char*)&V[0],sizeof(V_type)*nnzlength);
#endif
	//Other thresholds
	

	for (int s = 1; s != NumS; s++)
	{
#ifdef figure
		int idx = 0;
		R_type j = Row[0];
		for (int i = 0; i != N ; i++)
		{
			for ( ; j != Row[i+1]; j++)
				if (V[j] > r_thresh[s]-ep)
				{
					C[idx] = C[j];
					V[idx] = V[j];
					++idx;
				}
			Row[i+1] = idx;
		}	

		spa = 100.0 * Row[N] / M1;
		cout<<"sparsity: "<<spa<<endl;
		char sparsity[30];
		sprintf(sparsity, "_spa%.3f%%_cor%.3f", spa, r_thresh[s]);
		Outfilename = OutCor;
		Outfilename.append(string(sparsity)).append("_weighted.csr");
		ofstream fout;
		cout<<"generating "<<Outfilename.c_str()<< "..."<<endl;
		fout.open(Outfilename.c_str(), ios::binary | ios::out);
		if (!fout)
		{
			cout<<"create outfile unsuccessfully. error code:  "<<GetLastError()<<endl;
			exit(false);
		}	
		fout.write((const char*)&Rlength, sizeof(uint__t));
		fout.write((const char*)Row, sizeof(R_type)*Rlength);
		nnzlength = Row[N];
		fout.write((const char*)&nnzlength, sizeof(R_type));
		fout.write((const char*)&C[0],sizeof(C_type)*nnzlength);
		fout.write((const char*)&nnzlength, sizeof(R_type));
		fout.write((const char*)&V[0],sizeof(V_type)*nnzlength);
#endif
	}

	C.clear();
	V.clear();
	checkCudaErrors ( hipFree (devBOLD));
	checkCudaErrors ( hipFree (devCormat));
	checkCudaErrors ( hipFree (devBOLD_x));
	checkCudaErrors ( hipFree (devBOLD_y));
	checkCudaErrors ( hipFree (tiecount));
	fout.close();
	
	return 1;

}









	