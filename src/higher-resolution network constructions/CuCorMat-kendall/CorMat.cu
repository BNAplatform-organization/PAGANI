#include "hip/hip_runtime.h"
﻿#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "memory.h"

#include <iostream>
#include <ctime>

using namespace std;

typedef float real__t;
typedef unsigned int uint__t;

#define WARP 32
const int thread_num = 256;
//const int Bv_size = 256;
const int block_num = 48;
const int blocksize = 1024*1024*48;

#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)
inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
			file, line, errorMessage, (int)err, hipGetErrorString( err ) );
		exit(-1);
	}
}

__global__ void pre_process (real__t * devBOLD, real__t * BOLD_ori, int L, int L2, int Batch_size, real__t * tiecount, bool sumtieflag )
{
	__shared__ real__t BOLD_v[thread_num]; //wrong!supposed to load entire sequence,but this kernel can only handle sequence with 1024 length, alterstion needed later  
	__shared__ int tcount[thread_num];  //sole for each sequence.
	int tid = threadIdx.x; //each thead in distinct block
	int current_offs;
              //each block have a v   
	for(int v = blockIdx.x; v < Batch_size; v += gridDim.x)
	{
		long long offset_v_obj = v*L2; //??
	
		int tie_count = 0;
		real__t tmp = 0;
		
		if (tid<L)       //not load completely??
			BOLD_v[tid] = BOLD_ori[v*L+tid];//distinct block handle distinct quantity.
		else
			BOLD_v[tid] = 0; 
	
		syncthreads();
	
		for (int ii = tid/WARP; ii <= (L-2)/2; ii+=thread_num/WARP) //question: two loop? //elements NO. 1 warp for 1 elements
		{
			current_offs = (2*L-ii-1)*ii/2;//offset address fornula: (L- 1 + L - ii) * ii / 2.0 
			for (int j = tid%WARP; j < L-1-ii; j+=WARP) //number of pair of each elements
			{
				tmp = (real__t)(BOLD_v[j+ii+1]>BOLD_v[ii]) - (BOLD_v[j+ii+1]<BOLD_v[ii]); // greater than benchmark 1; less than benchmark -1; equal to benchmark:0;
				tie_count += real__t (tmp==0);
				devBOLD[offset_v_obj + current_offs + j] =  tmp;			
			}		
		}
	                 
		for (int ii = L-2-tid/WARP; ii >(L-2)/2; ii-=thread_num/WARP) 
		{
			current_offs = (2*L-1-ii)*ii/2;
			for (int j = tid%WARP; j < L-1-ii; j+=WARP)
			{
				tmp = (real__t) (BOLD_v[ii+j+1]>BOLD_v[ii]) - (BOLD_v[ii+j+1]<BOLD_v[ii]);
				tie_count += real__t (tmp==0);
				devBOLD[offset_v_obj + current_offs + j] = tmp;			
			}
		}
		syncthreads();

		if(sumtieflag) // two definition; in order to be immune to repetitive calculation.
		{		
			tcount[tid] = tie_count; 
			syncthreads();
			for (int i = thread_num/2; i > 0; i /= 2) //add together like tree
			{
				if (tid<i) tcount[tid] += tcount[tid + i];
				syncthreads();
			}
			if (tid==0)
				tiecount[v] = (real__t) tcount[tid];
		}		
		syncthreads();
	}
}

/*
__global__ void calctie(real__t *vec, real__t *tiecount, int L2, int Batch_size )
{
	__shared__ int tmp[thread_num];
	int tid = threadIdx.x;
	int tcount = 0;
	//int tidy = threadIdx.x/WARP;
	//int n_per_block = thread_num/WARP;
	for (int v = blockIdx.x; v<Batch_size; v+=gridDim.x )
	{
		tmp[threadIdx.x] = 0;
		for(int i = tid; i < L2; i+=thread_num)
		{
			tcount+=(vec[v*L2+i]==0);
		}
		tmp[tid] = tcount;
		syncthreads();

		for (int j = thread_num/2; j> 0; j = j/2)
		{
			if(tid <j) tmp[tid] += tmp[tid + j];
			syncthreads();
		}

		if (tid==0)
			tiecount[v] = (real__t) tmp[tid];
		syncthreads();
	}
}*/

int CorMat_gpu(real__t * Cormat, real__t * BOLD, int N, int L, int Batch_size,real__t * tie_count)
{
	//real__t *BOLD_t1, *BOLD_t2;
	real__t * out, * tempout;
	int L2 = L*(L-1)/2;
	int Num_Blocks = (N + Batch_size - 1) / Batch_size;
	uint__t N0 = Num_Blocks * Batch_size;
	

	// transposing the BOLD signal
	real__t * BOLD_t = new real__t [L * N0];
	//tempout = new real__t[Batch_size * Batch_size];
	memset(BOLD_t, 0, sizeof(real__t) * L * N0);
	for (int i = 0; i < L; i ++)
		for (int j = 0; j < N; j++)
		{
			BOLD_t[j * L + i] = BOLD[i * N + j];
		}

		// Normalize
	/*
	for (int i = 0; i < N; i++)
	{
			real__t * row = BOLD_t + i * L;
			double sum1 = 0, sum2 = 0;
			for (int l = 0; l < L; l++)
			{
				sum1 += row[l];
			}
			sum1 /= L;
			for (int l = 0; l < L; l++)
			{
				sum2 += (row[l] - sum1) * (row[l] - sum1);
			}
			sum2 = sqrt(sum2);
			for (int l = 0; l < L; l++)
			{
				row[l] = (row[l] - sum1) / sum2;;
			}
	}*/

		//// column major in every block
		//real__t * BOLD_t_col = new real__t [L * N0];
		//for (int k = 0; k < Num_Blocks; k++)
		//{
		//	for (int i = 0; i < Batch_size; i ++)
		//		for (int j = 0; j < L; j++)
		//		{
		//			BOLD_t_col[k * Batch_size * L + j * Batch_size + i] = BOLD_t[k * Batch_size * L + i * L + j];
		//		}
		//}

		hipError_t cudaStat;
		hipblasStatus_t stat;
		hipblasHandle_t handle;
		real__t * devBOLD_x, * devBOLD_y, * devBOLD_ori, * devCormat, * tiecount;
//		stat = cublasAlloc(L*N0, sizeof(real__t), (void**)&devBOLD);
		cudaStat = hipMalloc ((void**)&devBOLD_ori, sizeof(real__t) * L * N0) ;
		if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
			return cudaStat;
//		stat = cublasAlloc(Batch_size * Batch_size, sizeof(real__t), (void**)&devCormat);		
		cudaStat = hipMalloc ( (void**)&devCormat, sizeof(real__t) * Batch_size * Batch_size) ;
		if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
			return cudaStat;
		stat = hipblasSetMatrix(N0, L, sizeof(real__t), BOLD_t, N0, devBOLD_ori, N0);
//		cudaStat = hipMemcpy(devBOLD_ori, BOLD_t, sizeof(real__t) * L * N0, hipMemcpyHostToDevice);
		
		cudaStat = hipMalloc ((void**)&tiecount, sizeof(real__t) * N0) ;
		if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
			return cudaStat;
		cudaStat = hipMalloc ((void**)&devBOLD_x, sizeof(real__t) * L2 * Batch_size) ;
		if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
			return cudaStat;
		cudaStat = hipMalloc ((void**)&devBOLD_y, sizeof(real__t) * L2 * Batch_size) ;
		if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
			return cudaStat;		

		/******************************************/
		/*
		clock_t time = clock();
		real__t * temp_host = new real__t [L2*Batch_size];
		long long c = 0;
		for (int i = 0; i < Batch_size; i++)
			for (int j = 0; j < L-1; j++)
				for (int k = j+1; k < L; k++)
					temp_host[c++] = (BOLD_t[i*L+j]<BOLD_t[i*L+k]) - (BOLD_t[i*L+j]>BOLD_t[i*L+k]);
		cout<<"check c:"<<c-L2*Batch_size<<endl;
		time = clock()-time;
		cout<<"CPU time:"<<time<<endl;
		//hipMemset(devBOLD_x, 0, sizeof(real__t)*L2*Batch_size);
		//for (int i = 0; i < Batch_size/32; i++)
		time = clock();
		pre_process <<<64, thread_num>>>(devBOLD_x, devBOLD_ori, L, L2,Batch_size); 
		getLastCudaError("Kernel execution failed");
		real__t * temp_device = new real__t[L2*Batch_size];
		time = clock()-time;
		cout<<"GPU time:"<<time<<endl;
		cudaStat = hipMemcpy(temp_device, devBOLD_x, sizeof(real__t) * L2 * Batch_size, hipMemcpyDeviceToHost);
		if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
			return cudaStat;

		
		int check;
		for (c=0; c<L2*Batch_size; c++)
			if (temp_host[c]!=temp_device[c]) 
			{   cout<<c/L2<<"  "<<c%L2<<";  host:"<<temp_host[c]<< ";  device:"<<temp_device[c]<<endl;
				cin>>check;
			}

		//cout<<"check device host: "<<check<<endl;
		*/		
		/****************************************/

		stat = hipblasCreate(&handle) ;
		if (stat != HIPBLAS_STATUS_SUCCESS)
			return stat;

		const float alpha = 1.0;
		const float beta = 0;
		for (int kk = 0, ii = 0; ii < Num_Blocks; ii++)
		{                                                //iith dataWidth == batch_size
			pre_process <<<64, thread_num>>>(devBOLD_y, devBOLD_ori+ii * Batch_size * L, L, L2,Batch_size, tiecount+ii*Batch_size, true);
			out = Cormat + kk * Batch_size * Batch_size;
			kk++;

			stat = hipblasSgemm(handle, HIPBLAS_OP_T,  HIPBLAS_OP_N, Batch_size, Batch_size, L2,  &alpha, devBOLD_y, L2, devBOLD_y , L2, &beta, devCormat, Batch_size);
			if (stat != HIPBLAS_STATUS_SUCCESS)
				return stat;
			cudaStat = hipMemcpy(out, devCormat, sizeof(real__t) * Batch_size * Batch_size, hipMemcpyDeviceToHost);
			if (cudaStat != hipSuccess) 
				return cudaStat;

			for (int jj = ii+1; jj < Num_Blocks; jj++)
			{
				//BOLD_t1 = BOLD_t + ii * Batch_size * L;
				//BOLD_t2 = BOLD_t + jj * Batch_size * L;
				out = Cormat + kk * Batch_size * Batch_size;
				kk++;
								
				pre_process <<<64, thread_num>>>(devBOLD_x, devBOLD_ori+jj * Batch_size * L, L, L2,Batch_size, tiecount, false);
				stat = hipblasSgemm(handle, HIPBLAS_OP_T,  HIPBLAS_OP_N, Batch_size, Batch_size, L2,  &alpha, devBOLD_x, L2, devBOLD_y , L2, &beta, devCormat, Batch_size);
				if (stat != HIPBLAS_STATUS_SUCCESS)
					return stat;
				
				cudaStat = hipMemcpy(out, devCormat, sizeof(real__t) * Batch_size * Batch_size, hipMemcpyDeviceToHost);
				
				if (cudaStat != hipSuccess) 
					return cudaStat;
				
				/*		for (int i = 0; i < Batch_size; i ++)
					for (int j = 0; j < Batch_size; j++)
					{
						out[j * Batch_size + i] = tempout[i * Batch_size + j];
					}*/
				/*float *testA = new float[Batch_size * L];
				hipMemcpy(testA, devBOLD + ii * Batch_size * L, sizeof(real__t) * Batch_size * L, hipMemcpyDeviceToHost);
				float *testB = new float[Batch_size * L];
				hipMemcpy(testB, devBOLD + jj * Batch_size * L, sizeof(real__t) * Batch_size * L, hipMemcpyDeviceToHost);
				cout<<"A"<<endl;
				for (int i = 0; i < Batch_size; i++)
				{
				for (int j = 0; j < L; j++)
				{
				cout<<testA[i * L + j]<<"\t";
				}
				cout<<endl;
				}
				getchar();
				cout<<"B"<<endl;

				for (int i = 0; i < Batch_size; i++)
				{
				for (int j = 0; j < L; j++)
				{
				cout<<testB[i * L + j]<<"\t";
				}
				cout<<endl;
				}
				getchar();
				cout<<"C"<<endl;
				for (int i = 0; i < Batch_size; i++)
				{
				for (int j = 0; j < Batch_size; j++)
				{
				cout<<out[i * Batch_size + j]<<"\t";
				}
				cout<<endl;
				}
				getchar();*/
				/*	double sum3;
				for (int k = 0, i = 0; i < Batch_size; i++)
					for (int j = 0; j < Batch_size; j++)
					{
						sum3 = 0;
						for (int l = 0; l < L; l++)
						{
							sum3 += BOLD_t1[i*L+l] * BOLD_t2[j*L+l];
						}
						out[k++] = sum3;
					}*/
			}
		}
		cudaStat = hipMemcpy(tie_count, tiecount, sizeof(real__t) * N, hipMemcpyDeviceToHost);
		if (cudaStat != hipSuccess) 
			return cudaStat;

		hipFree (tiecount);
		hipFree (devBOLD_x);
		hipFree (devBOLD_y);
		hipFree (devBOLD_ori);
		hipFree (devCormat);
		stat = hipblasDestroy(handle);
		if (stat != HIPBLAS_STATUS_SUCCESS)
			return stat;
		delete []BOLD_t;
		return 1;
}




	