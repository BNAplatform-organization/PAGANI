#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "memory.h"
#include <iostream>
#include <ctime>
# include <fstream>
#include <vector>
#include <Windows.h>
#include<iomanip>
#include <thrust/device_vector.h>
#include"histogram.h"
#include <thrust/binary_search.h>
#include <sm_20_atomic_functions.h>
#include "help_func.cuh"
#include "pre_process.cuh"

//#define width  0.1//0.0001//best to be the multiples
#define width  0.000001//0.0001//best to be the multiples

using namespace std;
#pragma comment(lib,"cublas.lib")

__global__ void standardKernel(real__t* devCormat, int Batch_size, bool diagnoal)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;
	while(i<Batch_size*Batch_size)
	{
		if (!(devCormat[i]>0.0f && devCormat[i]<(1+ep)) )  //negative word is zero!
		{
			devCormat[i] = 0;
		}
		if(diagnoal==true)
		{
			if (i%(Batch_size+1)==0)
		    {
				devCormat[i] = 0;
		    }
		}
		i += offset;
	}
}



/* 
function CorMat_spa2rth
Calculate the correlation threshold for 
each sparsity threshold by a histogram method.
*/
real__t CorMat_spa2rth(string OutCor, real__t * BOLD_t, int N,  int L, int  Batch_size,real__t *s_thresh, real__t* result, int num_spa)
{
	const int Num_Blocks = (N + Batch_size - 1) / Batch_size;
	decltype(N) N0 = Num_Blocks * Batch_size;
	size_t L2 = L * ( L - 1 ) / 2;
	unsigned long long *zeroAmount = new unsigned long long [num_spa];
	for (uint__t i = 0; i < num_spa; i++)
	{
		unsigned long long amount = N * s_thresh[i] * (N-1)  / 100.0 ;///2.0 ;
#ifdef myLiteDebug
		cout<<"amount:"<<amount<<endl;
#endif
		//amount += amount%2;
		zeroAmount[i] = N;
		zeroAmount[i] *= N ;
		zeroAmount[i] -= amount;
		//cout<<zeroAmount[i]<<endl;
	}
		
	
	long long subtractor = (long long)N0 * N0 - (long long)N * N;
	uint__t Num_Bins = 1.0 / width + 1; //take care!
#ifdef myLiteDebug
	cout<<"Num_Bins:"<<Num_Bins<<endl;
#endif
	uint__t position = 0;
	// transposing the BOLD signal
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;               //optimize:bitmap+ for 1 -1 0; 
	real__t * devBOLD, * devCormat, * devBOLD_x, * devBOLD_y;
	size_t * tiecount; //perhaps _y _x tiecount should be integer type.
	//uint__t *devhisto;
	checkCudaErrors ( hipMalloc ((void**)&devBOLD, sizeof(real__t) * L * N0) ) ;
	checkCudaErrors ( hipMalloc ( (void**)&devCormat, sizeof(real__t) * Batch_size * Batch_size) );
	checkCudaErrors ( hipMalloc ((void**)&tiecount, sizeof(size_t) * N0) );
	checkCudaErrors ( hipMalloc ((void**)&devBOLD_x, sizeof(real__t) * L2 * Batch_size) );
	checkCudaErrors ( hipMalloc ((void**)&devBOLD_y, sizeof(real__t) * L2 * Batch_size) );
		
	stat = hipblasSetMatrix(N0, L, sizeof(real__t), BOLD_t, N0, devBOLD, N0);
	stat = hipblasCreate(&handle) ;
	if (stat != HIPBLAS_STATUS_SUCCESS)
		return stat;
	
	cout<<"generating histogram..."<<endl;
	cout<<"block number per row: "<<Num_Blocks<<endl;
	thrust::device_vector<long long> histogram(Num_Bins,0);
	clock_t time;
	time = clock();
	//uint__t blocknum = ( Batch_size * Batch_size + thread_num -1 ) / thread_num;
	bool flag = true;
	size_t shareSize = sizeof(real__t) * L + sizeof(size_t) * L;
	dim3 Grid(Batch_size/thread_num2D, Batch_size/thread_num2D), Block(thread_num2D,thread_num2D);
	//for (int ii = 0; ii < Num_Blocks; ii++)
	//{
	//	if ( ii > 0 )
	//		flag = false;
	//	cout<<"loop flag:"<<ii<<" "<<endl;
	//	pre_process <<<block_num, thread_num,  shareSize>>>(devBOLD_y, devBOLD + ii * Batch_size * L, L, L2, Batch_size, tiecount + ii * Batch_size, flag);
	//	//gpuOutput(devBOLD_y, sizeof(real__t) * L2 * Batch_size, OutCor, false);
	//	checkCublasErrors( hipblasSgemm(handle, HIPBLAS_OP_T,  HIPBLAS_OP_N, Batch_size, Batch_size, L2,  &alpha, devBOLD_y, L2, devBOLD_y , L2, &beta, devCormat, Batch_size) );
	//	//gpuOutput(devCormat, sizeof(real__t) * Batch_size * Batch_size, OutCor, true);
	//	dividedByDenominatorAndStandardedKernelWith2DBlock<<<Grid, Block>>>(devCormat, Batch_size, L, tiecount + ii * Batch_size, tiecount + ii * Batch_size, true);
	//	gpuOutput(devCormat, sizeof(real__t) * Batch_size * Batch_size, OutCor, true);
	//	dense_histogram(thrust::device_pointer_cast(devCormat),Batch_size * Batch_size, width, Num_Bins, histogram); //Somehow num_bins is generated wrongly in histogram.h files. So just transfer it directly.
	//	for (int jj = ii + 1; jj < Num_Blocks; jj++)
	//	{
	//		//cout<<"loop flag:"<<ii<<" "<<jj<<endl;
	//		pre_process <<<block_num, thread_num, shareSize>>>(devBOLD_x, devBOLD + jj * Batch_size * L, L, L2, Batch_size, tiecount + jj * Batch_size, flag);
	//		checkCublasErrors( hipblasSgemm(handle, HIPBLAS_OP_T,  HIPBLAS_OP_N, Batch_size, Batch_size, L2,  &alpha, devBOLD_x, L2, devBOLD_y , L2, &beta, devCormat, Batch_size) ); //devBold_x is equivalent to devBOLD + jj * Batch_size * L. So this is in accordance with previous formula.
	//		dividedByDenominatorAndStandardedKernelWith2DBlock<<<Grid, Block>>>(devCormat, Batch_size, L, tiecount + ii * Batch_size, tiecount + jj * Batch_size, false);
	//	    dense_histogram2(thrust::device_pointer_cast(devCormat),Batch_size * Batch_size, width, Num_Bins, histogram);//difference: Multiply temphisto by 2; Somehow num_bins is generated wrongly in histogram.h files. So just transfer it directly.
	//	}
	//}

	real__t* biPointer =  devBOLD_y;
	for (int ii = 0; ii < Num_Blocks; ii++)
	{
		if ( ii > 0 )
			flag = false;
#ifdef figure4
		cout<<"generate "<<ii+1<<"th batch. "<<endl;
#endif
		for (int jj = ii; jj < Num_Blocks; jj++)
		{
			biPointer = ii == jj ? devBOLD_y : devBOLD_x;
#ifdef figure
			int thread_num = L;
			int block_num = 180;
#endif
	        pre_process <<<block_num, thread_num, shareSize>>>(biPointer, devBOLD + jj * Batch_size * L, L, L2, Batch_size, tiecount + jj * Batch_size, flag);
	        checkCublasErrors( hipblasSgemm(handle, HIPBLAS_OP_T,  HIPBLAS_OP_N, Batch_size, Batch_size, L2,  &alpha, devBOLD_y, L2, biPointer, L2, &beta, devCormat, Batch_size) ); //Is this really right?
			dividedByDenominatorAndStandardedKernelWith2DBlock<<<Grid, Block>>>(devCormat, Batch_size, L, tiecount + ii * Batch_size, tiecount + jj * Batch_size, ii==jj);
#ifdef myDebug
			//cout<<"loop flag:"<<ii<<" "<<jj<<endl;
			gpuOutput(devCormat, sizeof(real__t) * Batch_size * Batch_size, OutCor, true);
			if (jj==1)
			{
				thrust::device_vector<size_t> dt(tiecount,tiecount+129);
				print_vector("common!",dt);
			}
#endif
			if (ii==jj)
				dense_histogram(thrust::device_pointer_cast(devCormat),Batch_size * Batch_size, width, Num_Bins, histogram); //Somehow num_bins is generated wrongly in histogram.h files. So just transfer it directly.
			else
			    dense_histogram2(thrust::device_pointer_cast(devCormat),Batch_size * Batch_size, width, Num_Bins, histogram);//difference: Multiply temphisto by 2; Somehow num_bins is generated wrongly in histogram.h files. So just transfer it directly.
		}
	}
	
	substract(histogram, subtractor); //subtract extra 0 emerged by ending batch
	for (size_t i = 0; i < num_spa; i++)
	{
		//cout<<zeroAmount[i]<<endl;
		position = thrust::upper_bound(histogram.begin(),histogram.end(),zeroAmount[i]) - histogram.begin();//flag~ N*N
		result[i] = width * position + width/2.0;
#ifdef myLiteDebug
		cout<<"threshold:"<<result[i]<<endl;
		cout<<position<<endl;
		cout<<"histogram[position-1]:"<<histogram[position-1]<<endl;
		cout<<"histogram[position]:"<<histogram[position]<<endl;
		cout<<"histogram[position+1]:"<<histogram[position+1]<<endl;
#endif
	}
	//display and put out
	time = clock() - time;
	cout<<"histogram time: "<<time<<"ms"<<endl;
	
	checkCudaErrors ( hipFree (devBOLD));
	checkCudaErrors ( hipFree (devCormat));
	checkCudaErrors ( hipFree (devBOLD_x));
	checkCudaErrors ( hipFree (devBOLD_y));
	checkCudaErrors ( hipFree (tiecount));

	/*thrust::adjacent_difference(histogram.begin(), histogram.end(), histogram.begin());
	print_vector("histogram:",histogram);*/
	thrust::device_vector<long long>().swap(histogram);
	
	delete[] zeroAmount;
	stat = hipblasDestroy(handle);
	if (stat != HIPBLAS_STATUS_SUCCESS)
		return stat;
		
}
