#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "memory.h"
#include <iostream>
#include <ctime>
# include <fstream>
#include <vector>
#include <Windows.h>
#include<iomanip>
#include <thrust/device_vector.h>
#include"histogram.h"
#include <thrust/binary_search.h>
#include <sm_20_atomic_functions.h>
#include "help_func.cuh"
#include "pre_process.cuh"

//#define width  0.1//0.0001//best to be the multiples
#define width  0.000001//0.0001//best to be the multiples

using namespace std;
#pragma comment(lib,"cublas.lib")

extern __global__ void standardKernel(real__t* devCormat, int Batch_size, bool diagnoal);

void updataPointer(real__t** pointer, real__t* devBOLD, int L, int Batch_size, const int Transferblocknum)
{
	*pointer =  *pointer ==  ( devBOLD + L * Batch_size * (Transferblocknum-1) )  ? devBOLD : *pointer + L * Batch_size; 
}
void updateDevBOLD(int jj, real__t* devBOLD, real__t* BOLD_t, int L, int Batch_size, int* hosPoint, int* count, real__t* pointer, const int Num_Blocks)
{
	if (*count + 1 < Num_Blocks)
	{
		real__t* devJiAddr = pointer;
		if (*hosPoint == Num_Blocks - 1)
		{
			(*count)++;
			(*hosPoint) =  (*count);	
		}
		else
			(*hosPoint) ++;
	real__t* hosJiAddr = BOLD_t + (*hosPoint) * Batch_size * L;
	checkCudaErrors( hipMemcpy(devJiAddr, hosJiAddr, sizeof(real__t) * L * Batch_size, hipMemcpyHostToDevice) );
	}
}

/* 
function CorMat_spa2rth
Calculate the correlation threshold for 
each sparsity threshold by a histogram method.
*/
real__t CorMat_spa2rth_blocking(string OutCor, real__t * BOLD_t, int N,  int L, int  Batch_size,real__t *s_thresh, real__t* result, int num_spa, const int Transferblocknum)
{
	const int Num_Blocks = (N + Batch_size - 1) / Batch_size;
	decltype(N) N0 = Num_Blocks * Batch_size;
	size_t L2 = L * ( L - 1 ) / 2;
	unsigned long long *zeroAmount = new unsigned long long [num_spa];
	for (uint__t i = 0; i < num_spa; i++)
	{
		unsigned long long amount = N * s_thresh[i] * (N-1)  / 100.0 ;///2.0 ;
#ifdef myLiteDebug
		cout<<"amount:"<<amount<<endl;
#endif
		//amount += amount%2;
		zeroAmount[i] = N;
		zeroAmount[i] *= N ;
		zeroAmount[i] -= amount;
		//cout<<zeroAmount[i]<<endl;
	}
		
	
	long long subtractor = (long long)N0 * N0 - (long long)N * N;
	uint__t Num_Bins = 1.0 / width + 1; //take care!
#ifdef myLiteDebug
	cout<<"Num_Bins:"<<Num_Bins<<endl;
#endif
	uint__t position = 0;
	// transposing the BOLD signal
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;               //optimize:bitmap+ for 1 -1 0; 
	real__t * devBOLD, * devCormat, * devBOLD_x, * devBOLD_y;
	size_t * tiecount; //perhaps _y _x tiecount should be integer type.
	real__t * pointer;
	int count = 0; //used to control pointer
	int hosPoint = Transferblocknum - 1;
	//uint__t *devhisto;
	checkCudaErrors ( hipMalloc ((void**)&devBOLD, sizeof(real__t) * L * Batch_size * Transferblocknum) ) ;
	checkCudaErrors ( hipMalloc ( (void**)&devCormat, sizeof(real__t) * Batch_size * Batch_size) );
	checkCudaErrors ( hipMalloc ((void**)&tiecount, sizeof(size_t) * N0) );
	checkCudaErrors ( hipMalloc ((void**)&devBOLD_x, sizeof(real__t) * L2 * Batch_size) );
	checkCudaErrors ( hipMalloc ((void**)&devBOLD_y, sizeof(real__t) * L2 * Batch_size) );
	checkCudaErrors( hipMemcpy(devBOLD, BOLD_t, sizeof(real__t) * L * Batch_size * Transferblocknum, hipMemcpyHostToDevice) );
	pointer = devBOLD;	
	stat = hipblasCreate(&handle) ;
	if (stat != HIPBLAS_STATUS_SUCCESS)
		return stat;
	
	cout<<"generating histogram..."<<endl;
	cout<<"block number per row: "<<Num_Blocks<<endl;
	thrust::device_vector<long long> histogram(Num_Bins,0);
	clock_t time;
	time = clock();
	//uint__t blocknum = ( Batch_size * Batch_size + thread_num -1 ) / thread_num;
	bool flag = true;
	size_t shareSize = sizeof(real__t) * L + sizeof(size_t) * L;
	dim3 Grid(Batch_size/thread_num2D, Batch_size/thread_num2D), Block(thread_num2D,thread_num2D);
	real__t* biPointer =  devBOLD_y;
	for (int ii = 0; ii < Num_Blocks; ii++)
	{
		if ( ii > 0 )
			flag = false;
		for (int jj = ii; jj < Num_Blocks; jj++)
		{
#ifdef myDebug
	cout<<"loop flag:"<<ii<<" "<<jj<<endl;
#endif
			biPointer = ii == jj ? devBOLD_y : devBOLD_x;
#ifdef figure
			int thread_num = L;
			int block_num = 180;
#endif
			pre_process <<<block_num, thread_num, shareSize>>>(biPointer, pointer, L, L2, Batch_size, tiecount + jj * Batch_size, flag);
	        checkCublasErrors( hipblasSgemm(handle, HIPBLAS_OP_T,  HIPBLAS_OP_N, Batch_size, Batch_size, L2,  &alpha, devBOLD_y, L2, biPointer, L2, &beta, devCormat, Batch_size) ); //Is this really right?
			dividedByDenominatorAndStandardedKernelWith2DBlock<<<Grid, Block>>>(devCormat, Batch_size, L, tiecount + ii * Batch_size, tiecount + jj * Batch_size, ii==jj);
			updateDevBOLD(jj, devBOLD, BOLD_t, L, Batch_size, &hosPoint, &count, pointer, Num_Blocks);
			updataPointer(&pointer, devBOLD, L, Batch_size, Transferblocknum);
#ifdef myDebug
			//cout<<"loop flag:"<<ii<<" "<<jj<<endl;
			gpuOutput(devCormat, sizeof(real__t) * Batch_size * Batch_size, OutCor, true);
			if (jj==1)
			{
				thrust::device_vector<size_t> dt(tiecount,tiecount+129);
				print_vector("common!",dt);
			}
#endif
			if (ii==jj)
				dense_histogram(thrust::device_pointer_cast(devCormat),Batch_size * Batch_size, width, Num_Bins, histogram); //Somehow num_bins is generated wrongly in histogram.h files. So just transfer it directly.
			else
			    dense_histogram2(thrust::device_pointer_cast(devCormat),Batch_size * Batch_size, width, Num_Bins, histogram);//difference: Multiply temphisto by 2; Somehow num_bins is generated wrongly in histogram.h files. So just transfer it directly.
		}
	}
	
	substract(histogram, subtractor); //subtract extra 0 emerged by ending batch
	for (size_t i = 0; i < num_spa; i++)
	{
		//cout<<zeroAmount[i]<<endl;
		position = thrust::upper_bound(histogram.begin(),histogram.end(),zeroAmount[i]) - histogram.begin();//flag~ N*N
		result[i] = width * position + width/2.0;
#ifdef myLiteDebug
		cout<<"threshold:"<<result[i]<<endl;
		cout<<position<<endl;
		/*cout<<"histogram[position-1]:"<<histogram[position-1]<<endl;
		cout<<"histogram[position]:"<<histogram[position]<<endl;
		cout<<"histogram[position+1]:"<<histogram[position+1]<<endl;*/
#endif
	}
	//display and put out
	time = clock() - time;
	cout<<"histogram time: "<<time<<"ms"<<endl;
	
	checkCudaErrors ( hipFree (devBOLD));
	checkCudaErrors ( hipFree (devCormat));
	checkCudaErrors ( hipFree (devBOLD_x));
	checkCudaErrors ( hipFree (devBOLD_y));
	checkCudaErrors ( hipFree (tiecount));

	/*thrust::adjacent_difference(histogram.begin(), histogram.end(), histogram.begin());
	print_vector("histogram:",histogram);*/
	thrust::device_vector<long long>().swap(histogram);
	
	delete[] zeroAmount;
	stat = hipblasDestroy(handle);
	if (stat != HIPBLAS_STATUS_SUCCESS)
		return stat;
		
}
