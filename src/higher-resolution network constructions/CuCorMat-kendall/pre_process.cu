#include "hip/hip_runtime.h"
#include "pre_process.cuh" 

__global__ void pre_process (real__t * devBOLD, real__t * BOLD_ori, int L, int L2, int Batch_size, size_t * tiecount, bool sumtieflag )
{
	extern __shared__ real__t share[];
	real__t* BOLD_v = share;
	size_t* tcount = (size_t*)( &BOLD_v[L] );
	int tid = threadIdx.x; //each thead in distinct block
	int current_offs;
              //each block have a v   
	for(int v = blockIdx.x; v < Batch_size; v += gridDim.x)
	{
		long long offset_v_obj = v*L2; 
	
		size_t tie_count = 0;
		real__t tmp = 0;
		      
		BOLD_v[tid] = BOLD_ori[v*L+tid];//distinct block handle distinct quantity.
		if (tid + thread_num < L)
		{
			BOLD_v[tid + thread_num] = BOLD_ori[v*L + tid + thread_num];;
		}
		syncthreads();
	
		for (int ii = tid/WARP; ii <= (L-2)/2; ii+=thread_num/WARP) //question: two loop? //elements NO. 1 warp for 1 elements
		{
			current_offs = (2*L-ii-1)*ii/2;//offset address fornula: (L- 1 + L - ii) * ii / 2.0 
			for (int j = tid%WARP; j < L-1-ii; j+=WARP) //number of pair of each elements
			{
				tmp = (real__t)(BOLD_v[j+ii+1]>BOLD_v[ii]) - (BOLD_v[j+ii+1]<BOLD_v[ii]); // greater than benchmark 1; less than benchmark -1; equal to benchmark:0;
				tie_count += (tmp==0);
				devBOLD[offset_v_obj + current_offs + j] =  tmp;			
			}		
		}
	                 
		for (int ii = L-2-tid/WARP; ii >(L-2)/2; ii-=thread_num/WARP) 
		{
			current_offs = (2*L-1-ii)*ii/2;
			for (int j = tid%WARP; j < L-1-ii; j+=WARP)
			{
				tmp = (real__t) (BOLD_v[ii+j+1]>BOLD_v[ii]) - (BOLD_v[ii+j+1]<BOLD_v[ii]);
				tie_count +=  (tmp==0);
				devBOLD[offset_v_obj + current_offs + j] = tmp;			
			}
		}
		syncthreads();

		if(sumtieflag) // two definition; in order to be immune to repetitive calculation.
		{		
			tcount[tid] = tie_count;
			syncthreads();
			for (int i = thread_num/2; i > 0; i /= 2) //add together like tree
			{
				if (tid<i) tcount[tid] += tcount[tid + i];
				syncthreads();
			}
			if (tid==0)
				tiecount[v] = tcount[tid];
		}		
		syncthreads();
	}
}
//hahahaha! another demo with fine or coar grained comparison.
__global__ void dividedByDenominatorAndStandardedKernel(real__t* devCormat, int Batch_size, int L, real__t * tieAddr1, real__t * tieAddr2, bool diagnoal)
{
	int tid = threadIdx.x;
	//may cause error.
	size_t n1 = 0; // obtained in broadcasted way
	size_t n2 = 0;
	size_t n0 = L * ( L - 1 ) / 2.0; //batch_size equals to L; This is best!
	size_t temp1 = 0;
	real__t temp2 = 0;

	if ( tid >= Batch_size ) {return;}  //may cause conflict or broadcast??
	n2 = tieAddr2[tid];
	
	for(int v = blockIdx.x; v < Batch_size; v += gridDim.x)//distinct block handle distinct quantity.
	{
		real__t* nominatorAddr = devCormat + v * Batch_size;
		n1 = tieAddr1[v];
		temp1 = ( n0 - n1 ) * ( n0 - n2 );
		temp2 = nominatorAddr[tid] / temp1;
		if ( temp1 == 0 || ( diagnoal && tid == v ) || temp2<0.0f || temp2 >= (1+ep) ) //three condition need to artificially set 0: denominator is 0; diagnoal elements; thrsholding
			temp2 = 0;
		nominatorAddr[tid]  = temp2; 
	}
}
// 32 * 32 threads and 32 block. 
//You need try without shared memory!                                                                        //other kernels also need to be modified    
__global__ void dividedByDenominatorAndStandardedKernelWith2DBlock(real__t* devCormat, int Batch_size, int L, size_t * tieAddr1, size_t * tieAddr2, bool diagnoal)
{
	
	__shared__ size_t rowTile[thread_num2D];
	__shared__ size_t colTile[thread_num2D];
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	/*rowTile[threadIdx.y] = tieAddr1[row*thread_num2D+threadIdx.y];
	colTile[threadIdx.x] = tieAddr2[col*thread_num2D+threadIdx.x];*/
	
	rowTile[threadIdx.y] = tieAddr2[row];
	colTile[threadIdx.x] = tieAddr1[col];

	__syncthreads();

	size_t n0 = L * ( L - 1 ) / 2.0; 
	size_t temp1 = 0;
	real__t temp2 = 0;
	real__t* nominator = devCormat + row * Batch_size + col;

#ifdef myDebug
	if ( row == 0 && col == 4 )
	{
		printf("nominator is %f \n", *nominator);
	}
#endif
	
	temp1 = ( n0 - rowTile[threadIdx.y] ) * ( n0 - colTile[threadIdx.x] );
	temp2 = ( *nominator ) / sqrt( (real__t) temp1 );

	if ( temp1 == 0 || ( diagnoal && row == col ) || temp2<0.0f || temp2 >= (1+ep) ) //three condition need to manually set 0: denominator is 0; diagnoal elements; out of range.
		temp2 = 0;
	
	*nominator = temp2;

#ifdef myDebug
	if ( row == 0 && col == 4 )
	{
		printf("nominator after operation is %f \n", *nominator);
		printf("temp1 is %d \n", temp1);
		printf("n0 is %d \n", n0);
		printf("rowTile[threadIdx.y] is %d \n", rowTile[threadIdx.y]);
		printf("colTile[threadIdx.x] is %d \n", colTile[threadIdx.x]);

	}
#endif

}
//without any set 0 operation; will also copare with nonshared memory kernel.
__global__ void dividedByDenominator(real__t* devCormat, int Batch_size, int L, size_t * tieAddr1, size_t * tieAddr2) 
{
	__shared__ size_t rowTile[thread_num2D];
	__shared__ size_t colTile[thread_num2D];
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	/*rowTile[threadIdx.y] = tieAddr1[row*thread_num2D+threadIdx.y];
	colTile[threadIdx.x] = tieAddr2[col*thread_num2D+threadIdx.x];
	*/
	
	rowTile[threadIdx.y] = tieAddr2[row];
	colTile[threadIdx.x] = tieAddr1[col];
	
	__syncthreads();
	
	size_t n0 = L * ( L - 1 ) / 2.0; 
	size_t temp1 = 0;
	real__t* nominator = devCormat + row * Batch_size + col;

	temp1 = ( n0 - rowTile[threadIdx.y] ) * ( n0 - colTile[threadIdx.x] );

	(*nominator) /= sqrt ( (real__t) temp1);
}

void gpuOutput( real__t* gpuAddr, unsigned int byteNo, string OutCor, bool nameFlag)
{
	ofstream fout;
	real__t* cpuAddr = new real__t[byteNo/sizeof(real__t)];
	checkCudaErrors ( hipMemcpy(cpuAddr, gpuAddr, byteNo, hipMemcpyDeviceToHost) );
	string filename;
	if(!nameFlag)
		filename = OutCor.append("_wrong.matrix");
	else
		filename = OutCor.append("_right.matrix");
	fout.open(filename.c_str(), ios::binary | ios::out);
	if (!fout)
	{
		cout<<"create outfile(gpu) unsuccessfully. error code:  "<<"fighting!"<<endl;		
		system("pause");
	}	
	fout.write((const char*)cpuAddr,byteNo);
	fout.close();
	delete[] cpuAddr;
}