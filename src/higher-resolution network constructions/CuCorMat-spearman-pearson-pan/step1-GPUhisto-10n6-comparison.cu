#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "memory.h"
#include <iostream>
#include <ctime>
# include <fstream>
#include <vector>
#include <Windows.h>
#include<iomanip>
#include <thrust/device_vector.h>
#include"histogram.h"
#include <thrust/binary_search.h>
#include <sm_20_atomic_functions.h>

using namespace std;

#define ep  1e-6  //third question
#define width  0.000001//0.0001//best to be the multiples 

#pragma comment(lib,"cublas.lib")
typedef float real__t;
typedef unsigned int uint__t;

const int thread_num = 256; //maybe redefinition
const int block_num = 48;     //(bat*bat+thrednum-1)/threadnum
const int blocksize = 1024*1024*48;
/*
bool IsNumber(double x)
{
	return (x == x);
}
 bool IsFiniteNumber(double x)
{
	return (x <= DBL_MAX && x >= -DBL_MAX);
}
*/
bool myfunction (real__t i,real__t j) { return (i>j); }
//void select(vector<real__t>::iterator A,long long n,long long k);
#define TOM(byteValue) (byteValue/1024/1024)
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)
inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
	if(hipSuccess != err)
	{
		fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
		//exit(-1);
		system("pause");
	}
}
__global__ void standardKernel(real__t* devCormat, int Batch_size, bool diagnoal)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;
	while(i<Batch_size*Batch_size) 
	{
		if ((!(devCormat[i] == devCormat[i]))||(!(devCormat[i] <= DBL_MAX && devCormat[i] >= -DBL_MAX))||devCormat[i]<0) 
		{
			devCormat[i] = 0;
		}
		if(diagnoal==true)
		{
			if (i%(Batch_size+1)==0)
		    {
			devCormat[i] = 0;
		    }
		}
		 i += offset;
	}
}
void MatrixMultiplication(real__t * BOLD_t1, real__t * BOLD_t2,real__t * out,int Batch_size,int L);
real__t CorMat_spa2rth(string OutCor, real__t * BOLD, int N, int L, int Batch_size,real__t *s_thresh, clock_t * aggregate)
{
	real__t * BOLD_t1, * BOLD_t2, * tempout;
	const int Num_Blocks = (N + Batch_size - 1) / Batch_size;
	uint__t N0 = Num_Blocks * Batch_size;
	uint__t amount = N * (*s_thresh) * (N-1)  / 100.0 ;///2.0 ;
	//amount += amount%2;
	uint__t zeroAmount = N * N - amount;
	cout<<"nonzero numbers: "<<amount<<endl;
	//uint__t invaccount = N - account;
	uint__t Num_Bins = 1.0 / width + 1; //take care! 
	uint__t position = 0;

	// transposing the BOLD signal
	real__t * BOLD_t = new real__t [L * N0];
	tempout = new real__t[Batch_size * Batch_size];
	memset(BOLD_t, 0, sizeof(real__t) * L * N0);
	for (int i = 0; i < L; i ++)
		for (int j = 0; j < N; j++)
		{
			BOLD_t[j * L + i] = BOLD[i * N + j];
		}
		
		for (long i = L * N; i < L * N0; i++)
		{
			BOLD_t[i] = 0;
		}	
		
		// Normalize
		for (int i = 0; i < N; i++)
		{
			real__t * row = BOLD_t + i * L;
			double sum1 = 0, sum2 = 0;
			for (int l = 0; l < L; l++)
			{
				sum1 += row[l];
			}
			sum1 /= L;
			for (int l = 0; l < L; l++)
			{
				sum2 += (row[l] - sum1) * (row[l] - sum1);
			}
			sum2 = sqrt(sum2);
			for (int l = 0; l < L; l++)
			{
				row[l] = (row[l] - sum1) / sum2;;
			}
		}
		hipError_t cudaStat;
		hipblasStatus_t stat;
		hipblasHandle_t handle;
		real__t * devBOLD, * devCormat;// * devCormatLower, * devCormatPacked;
		//uint__t *devhisto;
		cudaStat = hipMalloc ((void**)&devBOLD, sizeof(real__t) * L * N0) ;
		//hipError_t err = hipGetLastError();
        //printf("%s\n",hipGetErrorString(err));
		if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
			return cudaStat;
		cudaStat = hipMalloc ( (void**)&devCormat, sizeof(real__t) * Batch_size * Batch_size) ;
		if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
			return cudaStat;
		stat = hipblasSetMatrix(N0, L, sizeof(real__t), BOLD_t, N0, devBOLD, N0);
		stat = hipblasCreate(&handle) ;
		if (stat != HIPBLAS_STATUS_SUCCESS)
			return stat;
		delete []BOLD_t;

		//��ָGPU block�ĸ�����
		cout<<"generating histogram..."<<endl;
		cout<<"block numbers: "<<Num_Blocks<<endl;
		const float alpha = 1.0;
		const float beta = 0;
		const float pbeta = -1.0;
		vector< vector<real__t> > bin;
		thrust::device_vector<uint__t> histogram(Num_Bins,0); 
		clock_t time;
		time = clock();
		//uint__t blocknum = ( Batch_size * Batch_size + thread_num -1 ) / thread_num;
		for (int kk = 0, ii = 0; ii < Num_Blocks; ii++)
		{
			for (int jj = ii; jj < Num_Blocks; jj++)
			{
				  
				//BOLD_t1 = BOLD_t + ii * Batch_size * L;
				//BOLD_t2 = BOLD_t + jj * Batch_size * L;
   			//	real__t *out = new real__t[Batch_size * Batch_size];

#ifdef CPUCormat
                MatrixMultiplication_s(BOLD_t1, BOLD_t2, out, Batch_size,L);//need modify as well.
#else            
				//time = clock();
				stat = hipblasSgemm(handle, HIPBLAS_OP_T,  HIPBLAS_OP_N, Batch_size, Batch_size, L,  &alpha, devBOLD + jj * Batch_size * L, L, devBOLD + ii * Batch_size * L, L, &beta, devCormat, Batch_size);
				if (stat != HIPBLAS_STATUS_SUCCESS)
					return stat;
				//1clear NaN and negative number 
				standardKernel<<<block_num,thread_num>>>(devCormat, Batch_size,ii==jj);
				//2.statistics;the results of non-diagnoal batch shall multiply by 2 accordingly
				thrust::device_vector<uint__t> temphisto(Num_Bins,0); //dontforgettofree!
				thrust::device_ptr<real__t> dev_ptr(devCormat);
				
				if (ii==jj)
				{
					dense_histogram(dev_ptr,Batch_size * Batch_size, width,temphisto, histogram);
				}
				else
				{
					dense_histogram2(dev_ptr,Batch_size * Batch_size, width,temphisto, histogram);
				}
				thrust::device_vector<uint__t>().swap(temphisto);
				thrust::raw_pointer_cast(dev_ptr);//have a try
				//time = clock() - time;
				//cout<<"thrust::histogram time: "<<time<<"ms"<<endl;
				cout<<"loop flag "<<"ii: "<<ii<<" jj: "<<jj<<endl;
#endif
			}
			cout<<"Fulfill the "<<ii+1<<"th block."<<endl;
		}
		
		uint__t subtractor = N0 * N0 - N * N;
		substract(histogram, subtractor);
		position = thrust::upper_bound(histogram.begin(),histogram.end(),zeroAmount) - histogram.begin();
		real__t result = width * position + width/2.0; 
		//another interface is needed to return the subscript		
		    //display and put out 
			time = clock() - time;
			cout<<"histogram time: "<<time<<"ms"<<endl;
			*aggregate = time;
			//time_t nowTime;
			/*
			unsigned int FreeMem = 0;
			MEMORYSTATUS MemStat;
			MemStat.dwLength = sizeof(MEMORYSTATUS);
			GlobalMemoryStatus(&MemStat);
			FreeMem = TOM(MemStat.dwAvailPhys);
			cout << "bytes of physical memory: " << TOM(MemStat.dwTotalPhys) <<"M" <<endl;
			cout << "percent of memory in use: " << MemStat.dwMemoryLoad <<"%" <<endl;
			cout << "free physical memory bytes: " << TOM(MemStat.dwAvailPhys) <<"M" <<endl;
			*/
			hipFree (devBOLD); 

			hipFree (devCormat);
			stat = hipblasDestroy(handle);
			if (stat != HIPBLAS_STATUS_SUCCESS)
				return stat;
		//	delete []BOLD_t;
			return result;
}
real__t interval(void)
{
	return width;
}
//void MatrixMultiplication_s(real__t * BOLD_t1, real__t * BOLD_t2,real__t * out,int Batch_size,int L)//do not announce
//{
//	long kk = 0;
//	for (int k = 0; k < Batch_size; k++)
//	{
//		for (int i = 0; i < Batch_size; i++)
//		{   
//			double sum3 = 0.0;
//			for (int j = 0; j < L; j++)
//			{
//				sum3 += 1.0*BOLD_t1[k*L+j] * BOLD_t2[i*L+j];
//			}
//			out[kk++] = sum3;
//		}
//	}
//	
//}