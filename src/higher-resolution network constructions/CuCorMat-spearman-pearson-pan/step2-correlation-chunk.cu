#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "memory.h"
#include <iostream>
#include <ctime>
#include <fstream>
#include <vector>
#include <Windows.h>
#include<iomanip>

using namespace std;

#define ep  1e-6  //third question

#pragma comment(lib,"cublas.lib")
typedef float real__t;
typedef unsigned int uint__t;

#define TOM(byteValue) (byteValue/1024/1024)

//#define CPUCormat 0

typedef struct cv
		{  
		 int column;
		 real__t value;
		} ColumnValueInfo;    //Global definition is necessary



const int thread_num = 256;
const int block_num = 48;
const int blocksize = 1024*1024*48;

void select(real__t *A,long long n,long long k);
void MatrixMultiplication(real__t * BOLD_t1, real__t * BOLD_t2,real__t * out,int Batch_size,int L);

void Thrust(vector <vector<ColumnValueInfo>>::iterator begin, real__t *out, int ii, int Batch_size, real__t r_thresh, real__t er);
void ThrustAsymmetrical(vector <vector<ColumnValueInfo>>::iterator begin, real__t *out, int ii, int jj, int Batch_size, real__t r_thresh, real__t er);

int CorMat_gpu(string OutCor, real__t * BOLD, int N, int L, int Batch_size,real__t *r_thresh,clock_t* aggregrate)
{
	real__t * BOLD_t1, * BOLD_t2, * tempout;
	const int Num_Blocks = (N + Batch_size - 1) / Batch_size;
	uint__t N0 = Num_Blocks * Batch_size;

	// transposing the BOLD signal
	real__t * BOLD_t = new real__t [L * N0];
	tempout = new real__t[Batch_size * Batch_size];
	memset(BOLD_t, 0, sizeof(real__t) * L * N0);
	for (int i = 0; i < L; i ++)
		for (int j = 0; j < N; j++)
		{
			BOLD_t[j * L + i] = BOLD[i * N + j];
		}
		
		// Normalize
		for (int i = 0; i < N; i++)
		{
			real__t * row = BOLD_t + i * L;
			double sum1 = 0, sum2 = 0;
			for (int l = 0; l < L; l++)
			{
				sum1 += row[l];
			}
			sum1 /= L;
			for (int l = 0; l < L; l++)
			{
				sum2 += (row[l] - sum1) * (row[l] - sum1);
			}
			sum2 = sqrt(sum2);
			for (int l = 0; l < L; l++)
			{
				row[l] = (row[l] - sum1) / sum2;;
			}
		}

		hipError_t cudaStat;
		hipblasStatus_t stat;
		hipblasHandle_t handle;
		real__t * devBOLD, * devCormat;
//		stat = cublasAlloc(L*N0, sizeof(real__t), (void**)&devBOLD);
		cudaStat = hipMalloc ((void**)&devBOLD, sizeof(real__t) * L * N0) ;
		if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
			return cudaStat;
//		stat = cublasAlloc(Batch_size * Batch_size, sizeof(real__t), (void**)&devCormat);		
		cudaStat = hipMalloc ( (void**)&devCormat, sizeof(real__t) * Batch_size * Batch_size) ;
		if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
			return cudaStat;
		stat = hipblasSetMatrix(N0, L, sizeof(real__t), BOLD_t, N0, devBOLD, N0);
//		cudaStat = hipMemcpy(devBOLD, BOLD_t, sizeof(real__t) * L * N0, hipMemcpyHostToDevice);
		stat = hipblasCreate(&handle) ;
		if (stat != HIPBLAS_STATUS_SUCCESS)
			return stat;

		//��ָGPU block�ĸ�����
		cout<<"block numbers: "<<Num_Blocks<<endl;
		const float alpha = 1.0;
		const float beta = 0;
		vector <vector<ColumnValueInfo>> ColumnAndValue; 
		vector <int> Row; 
		ColumnAndValue.resize(Num_Blocks*Batch_size);  
		Row.resize(Num_Blocks*Batch_size+1);  //consider whether allocate space just here or other.
		for (int i = 0; i < Num_Blocks*Batch_size+1; i++)
		{
			Row.push_back(0);
		}
		clock_t correlationTime = clock();
		real__t *out = new real__t[Batch_size * Batch_size];
		clock_t time;
		time = clock();
		for (int kk = 0, ii = 0; ii < Num_Blocks; ii++)
		{
			for (int jj = ii; jj < Num_Blocks; jj++)
			{
				  
				BOLD_t1 = BOLD_t + ii * Batch_size * L;
				BOLD_t2 = BOLD_t + jj * Batch_size * L;
				//  real__t *v425 = new real__t[L];
#ifdef CPUCormat
                MatrixMultiplication(BOLD_t1, BOLD_t2, out, Batch_size,L);
#else
				stat = hipblasSgemm(handle, HIPBLAS_OP_T,  HIPBLAS_OP_N, Batch_size, Batch_size, L,  &alpha, devBOLD + jj * Batch_size * L, L, devBOLD + ii * Batch_size * L, L, &beta, devCormat, Batch_size);//virtually kernel
				if (stat != HIPBLAS_STATUS_SUCCESS)
					return stat;
				cudaStat = hipMemcpy(out, devCormat, sizeof(real__t) * Batch_size * Batch_size, hipMemcpyDeviceToHost);
				if (cudaStat != hipSuccess) 
					return cudaStat;
#endif	
				ColumnValueInfo tmp;
				if(ii==jj)
				{
					Thrust(ColumnAndValue.begin(), out, ii, Batch_size,  *r_thresh,  ep);
				
					//for (int i = 0; i < Batch_size; i ++)
					//{
					//	for (int j = 0; j < Batch_size; j++)
					//    { 
					//		if(out[i * Batch_size + j]>(*r_thresh-ep)&&out[i * Batch_size + j]<=(1+ep)&&(i!=j))
					//		{
					//			//count ++;
					//			nonzerocount++;
					//		    tmp.column = j;
					//			tmp.column += ii * Batch_size;
					//			tmp.value = out[i*Batch_size+j];
					//			ColumnAndValue[ii*Batch_size+i].push_back(tmp);
					//		}
					//	}
					// }
				}
				else
				{
					//ThrustAsymmetrical(ColumnAndValue.begin(), out, ii, jj, Batch_size, *r_thresh, ep);
					for (int i = 0; i < Batch_size; i ++)
					{
						for (int j = 0; j < Batch_size; j++)
						{ 
							if( out[i * Batch_size + j]>(*r_thresh-ep) && out[i * Batch_size + j]<=(1+ep) )
							{
							  	//nonzerocount += 2;
							    //1.push row
								tmp.column = j;
								tmp.column += jj * Batch_size;
								tmp.value = out[i*Batch_size+j];
								ColumnAndValue[ii*Batch_size+i].push_back(tmp);
								//2.push column
								tmp.column = i;
								tmp.column += ii * Batch_size;
								tmp.value = out[i*Batch_size+j];
								ColumnAndValue[jj*Batch_size+j].push_back(tmp);
							}
						}
					}
				}
				/*time = clock()-time;
				cout<<"1.gpu time:"<<time<<" ms"<<endl;
				time = clock();*/
				cout<<"Loop flag: "<<ii<<":"<<jj<<endl;
			}
			cout<<"Fulfill the "<<ii+1<<"th disposition."<<endl;
		}
		delete []out;
		Row[0] = 0;
		for (vector <vector<ColumnValueInfo>>::iterator x = ColumnAndValue.begin(); x != ColumnAndValue.end(); x++)
		{
			Row[x-ColumnAndValue.begin() + 1] = (*x).size();
			Row[x-ColumnAndValue.begin() + 1] += Row[x-ColumnAndValue.begin()];
		}
		//display and put out 
		correlationTime = clock() - correlationTime;
		cout<<"correlation time: "<<correlationTime<<"ms"<<endl;
		* aggregrate += correlationTime;
		cout<<"overall time for histogram plus correlation: "<<*aggregrate<<"ms"<<endl;
		//time_t nowTime;
		unsigned int FreeMem = 0;
		MEMORYSTATUS MemStat;
		MemStat.dwLength = sizeof(MEMORYSTATUS);
		GlobalMemoryStatus(&MemStat);
		FreeMem = TOM(MemStat.dwAvailPhys);
		cout << "bytes of physical memory: " << TOM(MemStat.dwTotalPhys) <<"M" <<endl;
		cout << "percent of memory in use: " << MemStat.dwMemoryLoad <<"%" <<endl;
		cout << "free physical memory bytes: " << TOM(MemStat.dwAvailPhys) <<"M" <<endl;
		cout<<"number of non-zero elements: "<<Row[N]<<endl;
		long long M1 = (N-1);
		M1 *= N;
		M1 /= 2;
		real__t spa = 100.0 * Row[N] / M1 / 2.0;
		char sparsity[100];
		sprintf(sparsity, "_spa%.3f%%_cor%.3f", spa,*r_thresh);
		string Outfilename = OutCor;
		Outfilename.append(string(sparsity)).append("_weighted.csr");
		ofstream fout;
		cout<<"generating "<<Outfilename.c_str()<< "..."<<endl;
		fout.open(Outfilename.c_str(), ios::binary | ios::out);
		//fout.open(OutCor.c_str(),ios::binary | ios::out);
		if (!fout)
		{
			cout<<"create unsuccessfully. error code:  "<<GetLastError()<<endl;
			exit(false);

		}
		int Rlength = N+1;
		fout.write((char*)&Rlength, sizeof(int));
		for (int i = 0; i < Rlength; i++)
		{
				int R =Row[i];
				fout.write((char*)&R, sizeof(int));
		}
		int Clength = Row[N];
		fout.write((char*)&Clength, sizeof(int));
		for (vector <vector<ColumnValueInfo>>::iterator i = ColumnAndValue.begin(); i != ColumnAndValue.end(); i++)
		{
			for (vector<ColumnValueInfo>::iterator j = (*i).begin(); j !=(*i).end(); j++)
			{
				int C = (*j).column;
				fout.write((char*)&C, sizeof(int));
			}
				
		}
		fout.write((char*)&Clength, sizeof(int));
		for (vector <vector<ColumnValueInfo>>::iterator i = ColumnAndValue.begin(); i != ColumnAndValue.end(); i++)
		{
			for (vector<ColumnValueInfo>::iterator j = (*i).begin(); j !=(*i).end(); j++)
			{
				real__t V = (*j).value;
				fout.write((char*)&V, sizeof(real__t));
			}
				
		}
		fout.close();
		cout<<"Transmition finished."<<endl;
		hipFree (devBOLD); 
		hipFree (devCormat);
		stat = hipblasDestroy(handle);
		if (stat != HIPBLAS_STATUS_SUCCESS)
			return stat;
		delete []BOLD_t;
		return true;
}
void MatrixMultiplication(real__t * BOLD_t1, real__t * BOLD_t2,real__t * out,int Batch_size,int L)
{
	long kk = 0;
	for (int k = 0; k < Batch_size; k++)
	{
		for (int i = 0; i < Batch_size; i++)
		{   
			double sum3 = 0.0;
			for (int j = 0; j < L; j++)
			{
				sum3 += 1.0*BOLD_t1[k*L+j] * BOLD_t2[i*L+j];
			}
			out[kk++] = sum3;
		}
	}
	
}








	