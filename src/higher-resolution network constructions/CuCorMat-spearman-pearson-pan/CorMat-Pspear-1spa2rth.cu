#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "memory.h"
#include <iostream>
#include <ctime>
# include <fstream>
#include <vector>
#include <Windows.h>
#include<iomanip>
#include"histogram.h"
#include <thrust/binary_search.h>

#include "help_func.cuh"
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <cmath>

using namespace std;

#define ep  1e-6  //third question
//#define width  0.1//0.0001//best to be the multiples
#define width  0.000001//0.0001//best to be the multiples

#pragma comment(lib,"cublas.lib")
typedef float real__t;
typedef unsigned int uint__t;

const int thread_num = 1024; 
const int block_num = 30; 
bool myfunction (real__t i,real__t j) { return (i>j); }
#define TOM(byteValue) (byteValue/1024/1024)

__global__ void standardKernel(real__t* devCormat, int Batch_size, bool diagnoal)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;
	while(i<Batch_size*Batch_size)
	{
		if (!(devCormat[i]>0.0f && devCormat[i]<(1+ep)) ) 
		{
			devCormat[i] = 0;
		}
		if(diagnoal==true)
		{
			if (i%(Batch_size+1)==0)
		    {
			devCormat[i] = 0;
		    }
		}
		i += offset;
	}
}

/* 
function CorMat_spa2rth
Calculate the correlation threshold for 
each sparsity threshold by a histogram method.
*/
real__t CorMat_spa2rth(string OutCor, real__t * BOLD_t, int N,  int L, int  Batch_size, real__t *s_thresh, real__t* result, int num_spa)
{
	//cout<<"*s_thresh: "<<*s_thresh<<endl;
	//real__t * BOLD_t1, * BOLD_t2;// * tempout;
	const int Num_Blocks = (N + Batch_size - 1) / Batch_size;
	decltype(N) N0 = Num_Blocks * Batch_size;
	
	//amount += amount%2;
	//modify1 zeroAmount to an array
	//uint__t num_spa = sizeof(s_thresh) / sizeof(real__t);
	long long *zeroAmount = new long long [num_spa];
	for (uint__t i = 0; i < num_spa; i++)
	{
		long long amount = N * s_thresh[i] * (N-1)  / 100.0 ;///2.0 ;
		//amount += amount%2;
	
		zeroAmount[i] = N;
		zeroAmount[i] *= N ;//
		zeroAmount[i] -= amount;
		//cout<<zeroAmount[i]<<endl;
	/*	zeroAmount[i] = (long long)N * s_thresh[i] * (N-1)  / 100.0 ;
		zeroAmount[i] -= ((long long)N * N);
		zeroAmount[i] *= -1;*/
	}
		
	
	long long subtractor = (long long)N0 * N0 - (long long)N * N;
	//cout<<N0 * N0<<endl;
	//cout<<N * N<<endl;
	//cout<<subtractor<<endl;
	//uint__t invaccount = N - account;
	uint__t Num_Bins = 1.0 / width + 1; //take care!
	uint__t position = 0;

	// transposing the BOLD signal
	
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	real__t * devBOLD, * devCormat;// * devCormatLower, * devCormatPacked;
	
	
	checkCudaErrors (hipMalloc ((void**)&devBOLD, sizeof(real__t) * L * N0)) ;
	checkCudaErrors (hipMalloc ( (void**)&devCormat, sizeof(real__t) * Batch_size * Batch_size));	
	//checkCudaErrors (hipMalloc ( (void**)&devCormat, sizeof(real__t) * Batch_size * Batch_size));	

	stat = hipblasSetMatrix(N0, L, sizeof(real__t), BOLD_t, N0, devBOLD, N0);
	
	stat = hipblasCreate(&handle) ;
	if (stat != HIPBLAS_STATUS_SUCCESS)
		return stat;
	cout<<"generating histogram..."<<endl;
	cout<<"block number per row: "<<Num_Blocks<<endl;
	const float alpha = 1.0;
	const float beta = 0;	
	thrust::device_vector<long long> histogram(Num_Bins,0);
	clock_t time;
	time = clock();
	//uint__t blocknum = ( Batch_size * Batch_size + thread_num -1 ) / thread_num;
	for (int kk = 0, ii = 0; ii < Num_Blocks; ii++)
	{
		for (int jj = ii; jj < Num_Blocks; jj++)
		{
			//time = clock();
			stat = hipblasSgemm(handle, HIPBLAS_OP_T,  HIPBLAS_OP_N, Batch_size, Batch_size, L,  &alpha, devBOLD + ii * Batch_size * L, L, devBOLD + jj * Batch_size * L, L, &beta, devCormat, Batch_size);
			if (stat != HIPBLAS_STATUS_SUCCESS)
				return stat;
			//1clear NaN and negative number
			standardKernel<<<block_num,thread_num>>>(devCormat, Batch_size,ii==jj);
			//2.statistics;the results of non-diagnoal batch shall multiply by 2 accordingly
			//thrust::device_vector<uint__t> temphisto(Num_Bins,0); //dontforgettofree!
			thrust::device_ptr<real__t> dev_ptr(devCormat);
			
			/*if (ii==jj)
				dense_histogram(dev_ptr,Batch_size * Batch_size, width,temphisto, histogram);
			else
				dense_histogram2(dev_ptr,Batch_size * Batch_size, width,temphisto, histogram);*/

			//it seems that new approach reach higher speed.
			if (ii==jj)
				dense_histogram_new(dev_ptr,Batch_size * Batch_size, width, Num_Bins, histogram);
			else
				dense_histogram2_new(dev_ptr,Batch_size * Batch_size, width, Num_Bins, histogram);
			
			//thrust::device_vector<uint__t>().swap(temphisto);
			
			//thrust::raw_pointer_cast(dev_ptr);//have a try
			//time = clock() - time;
			//cout<<"thrust::histogram time: "<<time<<"ms"<<endl;
			//cout<<"loop flag "<<"ii: "<<ii<<" jj: "<<jj<<endl;
		}
		//cout<<"Fulfill the "<<ii+1<<"th block."<<endl;
	}

	substract(histogram, subtractor);
	
	for (size_t i = 0; i < num_spa; i++)
	{
		//cout<<zeroAmount[i]<<endl;
		position = thrust::upper_bound(histogram.begin(),histogram.end(),zeroAmount[i]) - histogram.begin();//flag~ N*N
		//cout<<position<<endl;
		result[i] = width * position + width/2.0;
#ifdef myLiteDebug
		cout<<"threshold:"<<result[i]<<endl;
		cout<<"histogram[position-1]:"<<histogram[position-1]<<endl;
		cout<<"histogram[position]:"<<histogram[position]<<endl;
		cout<<"histogram[position+1]:"<<histogram[position+1]<<endl;
#endif
	}

		//another interface is needed to return the subscript
		//display and put out
	time = clock() - time;
	cout<<"histogram time: "<<time<<"ms"<<endl;
	//checkCudaErrors (hipFree(thrust::raw_pointer_cast(histogram.data())));
	thrust::device_vector<long long>().swap(histogram);
	checkCudaErrors ( hipFree (devBOLD));
	checkCudaErrors ( hipFree (devCormat));

	delete[] zeroAmount;
	stat = hipblasDestroy(handle);
	if (stat != HIPBLAS_STATUS_SUCCESS)
		return stat;
		
}
