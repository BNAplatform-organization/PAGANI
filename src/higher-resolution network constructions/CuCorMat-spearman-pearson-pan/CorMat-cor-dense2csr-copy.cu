#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hipsparse.h"
#include "hip/hip_runtime.h"
#include "memory.h"
#include <iostream>
#include <ctime>
#include <fstream>
#include <vector>
#include <Windows.h>
#include<iomanip>
#pragma comment(lib,"cusparse.lib")
//#include "Utilities.cuh"

using namespace std;

#define ep  1e-6  //third question

#pragma comment(lib,"cublas.lib")
typedef float real__t;
typedef unsigned int uint__t;

#define TOM(byteValue) (byteValue/1024/1024)
extern void cusparseSafeCall(hipsparseStatus_t err);

//#define CPUCormat 0

typedef struct cv
		{  
		 int column;
		 real__t value;
		} ColumnValueInfo;   



const int thread_num = 256;
const int block_num = 48;
const int blocksize = 1024*1024*48;

void select(real__t *A,long long n,long long k);
void MatrixMultiplication(real__t * BOLD_t1, real__t * BOLD_t2,real__t * out,int Batch_size,int L);

void Thrust(vector <vector<ColumnValueInfo>>::iterator begin, real__t *out, int ii, int Batch_size, real__t r_thresh, real__t er);
void ThrustAsymmetrical(vector <vector<ColumnValueInfo>>::iterator begin, real__t *out, int ii, int jj, int Batch_size, real__t r_thresh, real__t er);

__global__ void standardAndThresholdingKernel(real__t* devCormat, int Batch_size, bool diagnoal, real__t thres)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;
	while(i<Batch_size*Batch_size) 
	{
		if (!(devCormat[i]>(thres-ep) && devCormat[i]<=(1+ep)) ) //flag!
		{
			devCormat[i] = 0;
		}
		if(diagnoal==true)
		{
			if (i%(Batch_size+1)==0)
		    {
				devCormat[i] = 0;
		    }
		}
		i += offset;
	}
}

__global__ void initialone(real__t *vec,int N)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;
	while(i<N) 
	{
		vec[i] = 1.0;
		i += offset;
	}
}
/*************************************************************************************************/
/* GPU-based Cormat function,
output the CSR format connectivity matrix,
and functional connectivity strength.*/
/*************************************************************************************************/
int CorMat_gpu(string OutCor, real__t * BOLD_t, const int N, const int N0, const int Num_Blocks, const int L, const int Batch_size,real__t *r_thresh)
{			
	
	/*************************************************************************************************/
	/*							  Setup CUBLAS and CUSPARSE parameters                               */
	/*************************************************************************************************/
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipsparseStatus_t  sparseStat;
	hipblasHandle_t handle;
	hipsparseHandle_t sparseHandle;
	hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_ROW;
	hipsparseMatDescr_t descrA = 0;
	sparseStat= hipsparseCreate(&sparseHandle);
    if (sparseStat != HIPSPARSE_STATUS_SUCCESS)
		return sparseStat;
	
	/* create and setup matrix descriptor */ 
	sparseStat= hipsparseCreateMatDescr(&descrA); 
	if (sparseStat != HIPSPARSE_STATUS_SUCCESS) 
		return sparseStat; 
	hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO);  
	//sparseStat = cusparse_create_mat_descr(descrA); 
	
	/***********************     Correlation matrix variables    ************************/
	/*          GPU variables        */
	real__t * devBOLD, * devCormat,*csrValA;
	int *nnzPerRowColumn,*csrRowPtrA,*csrColIndA;
	
	cudaStat = hipMalloc ((void**)&nnzPerRowColumn, sizeof(int) * Batch_size) ;
	if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
		return cudaStat;
	cudaStat = hipMalloc ((void**)&csrRowPtrA, sizeof(int) * (Batch_size+1) ) ;
	if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
		return cudaStat;
//	stat = cublasAlloc(L*N0, sizeof(real__t), (void**)&devBOLD);
	cudaStat = hipMalloc ((void**)&devBOLD, sizeof(real__t) * L * N0) ;
	if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
		return cudaStat;
//	stat = cublasAlloc(Batch_size * Batch_size, sizeof(real__t), (void**)&devCormat);		
	cudaStat = hipMalloc ( (void**)&devCormat, sizeof(real__t) * Batch_size * Batch_size) ;
	if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
		return cudaStat;
	stat = hipblasSetMatrix(N0, L, sizeof(real__t), BOLD_t, N0, devBOLD, N0);
//	cudaStat = hipMemcpy(devBOLD, BOLD_t, sizeof(real__t) * L * N0, hipMemcpyHostToDevice);
	stat = hipblasCreate(&handle) ;
	if (stat != HIPBLAS_STATUS_SUCCESS)
		return stat;
	
	/*          CPU variables        */
	uint__t Overall_Num_Blocks = Num_Blocks * Num_Blocks;
	uint__t **Column = new uint__t* [Overall_Num_Blocks];
	real__t **Value = new real__t* [Overall_Num_Blocks];
	uint__t **Rown = new uint__t* [Overall_Num_Blocks];
	int *nnzOfEachBlock = new int [Overall_Num_Blocks];
	long long totalNonzero = 0;

	long long *Row = new long long [Num_Blocks * Batch_size+1];
	memset(Row,0,sizeof(long long)*(Num_Blocks * Batch_size+1));
		
	/****************  Functional connectivity strength variables  ****************/
	real__t *fcsGPU,*vec;
	
	cudaStat = hipMalloc ((void**)&fcsGPU, sizeof(real__t) * Batch_size * Num_Blocks) ;
	if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
		return cudaStat;	
	hipMemset(fcsGPU,0,sizeof(real__t) * Batch_size * Num_Blocks);
	
	cudaStat = hipMalloc ((void**)&vec, sizeof(real__t) * Batch_size) ;
	if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
		return cudaStat;
	initialone<<<block_num,thread_num>>>(vec,Batch_size);			                                       

	cout<<"matrix block number: "<<Num_Blocks<<endl;
	const float alpha = 1.0;
	const float beta = 0;
	//const float gamma = 1.0;
	
	clock_t correlationTime = clock();
		
	/*************************************************************************************************/
	/*						         Start the correlation computation                               */
	/*************************************************************************************************/
	for (int ii = 0; ii < Num_Blocks; ii++)
	{
		for (int jj = ii; jj < Num_Blocks; jj++)
		{
			
//#ifdef CPUCormat
//          real__t * BOLD_t1, * BOLD_t2;
//			BOLD_t1 = BOLD_t + ii * Batch_size * L;
//			BOLD_t2 = BOLD_t + jj * Batch_size * L;
//			//  real__t *v425 = new real__t[L];
//			MatrixMultiplication(BOLD_t1, BOLD_t2, out, Batch_size, L);
//#endif																																			//has been amending					
			stat = hipblasSgemm(handle, HIPBLAS_OP_T,  HIPBLAS_OP_N, Batch_size, Batch_size, L,  &alpha, devBOLD + ii * Batch_size * L, L, devBOLD + jj * Batch_size * L, L, &beta, devCormat, Batch_size);//virtually kernel
			if (stat != HIPBLAS_STATUS_SUCCESS)
				return stat;
						
			//Calculating FCS
			standardAndThresholdingKernel<<<block_num,thread_num>>>(devCormat, Batch_size, ii==jj,0);  
				
			stat = hipblasSgemv(handle, HIPBLAS_OP_N, Batch_size, Batch_size, &alpha, devCormat, Batch_size, vec, 1, &alpha, fcsGPU + ii * Batch_size, 1);
			if (stat != HIPBLAS_STATUS_SUCCESS)
				return stat;					
			if (ii!=jj)
			{
				stat = hipblasSgemv(handle, HIPBLAS_OP_T, Batch_size, Batch_size, &alpha, devCormat, Batch_size, vec, 1, &alpha, fcsGPU + jj * Batch_size, 1);
				if (stat != HIPBLAS_STATUS_SUCCESS)
				return stat;
			}
			
			//so 1.thresholding 2.dense2csr 3.transfer 4.free
			//start 1.thresholding	
			standardAndThresholdingKernel<<<block_num,thread_num>>>(devCormat, Batch_size, ii==jj, *r_thresh);
			
			//2. dense2csr
			/*sparseStat = hipsparseSnnz(sparseHandle, HIPSPARSE_DIRECTION_ROW, Batch_size, Batch_size, descrA, devCormat,  Batch_size, nnzPerRowColumn, &nnzTotalDevHostPtr);
			if (sparseStat !=HIPSPARSE_STATUS_SUCCESS)
				return sparseStat;
			cudaStat = hipMalloc ((void**)&csrValA, sizeof(real__t) * nnzTotalDevHostPtr) ;
			if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
			return cudaStat;
			cudaStat = hipMalloc ((void**)&csrColIndA, sizeof(int) * nnzTotalDevHostPtr) ;
			if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
				return cudaStat;
			sparseStat = hipsparseSdense2csr(sparseHandle, Batch_size, Batch_size, descrA, devCormat, Batch_size, nnzPerRowColumn, csrValA, csrRowPtrA, csrColIndA);
			if (sparseStat != HIPSPARSE_STATUS_SUCCESS)
				return sparseStat;*/
			int nnzTotalDevHostPtr = 0;
			sparseStat = hipsparseSnnz(sparseHandle, HIPSPARSE_DIRECTION_ROW, Batch_size, Batch_size, descrA, devCormat,  Batch_size, nnzPerRowColumn, &nnzTotalDevHostPtr);
			if (sparseStat !=HIPSPARSE_STATUS_SUCCESS)
				return sparseStat;
			nnzOfEachBlock[ii * Num_Blocks + jj] = nnzTotalDevHostPtr;
			if (ii!=jj)
				nnzOfEachBlock[jj * Num_Blocks + ii] = nnzTotalDevHostPtr;
			
			Column[ii * Num_Blocks + jj] = new uint__t [nnzTotalDevHostPtr];
			Value[ii * Num_Blocks + jj] = new real__t [nnzTotalDevHostPtr];
			Rown[ii * Num_Blocks + jj] = new uint__t [Batch_size+1];
			if (ii!=jj)
			{
				Column[jj * Num_Blocks + ii] = new uint__t [nnzTotalDevHostPtr];
				Value[jj * Num_Blocks + ii] = new real__t [nnzTotalDevHostPtr];
				Rown[jj * Num_Blocks + ii] = new uint__t [Batch_size+1];
			}
			if (nnzTotalDevHostPtr==0)
			{
				//Rown[ii * Num_Blocks + jj] = new uint__t [Batch_size+1];
				for (int i = 0; i < (Batch_size + 1); i++)
				{
					Rown[ii * Num_Blocks + jj][i] = 0;
				}
				if (ii!=jj)
				{
					//Rown[jj * Num_Blocks + ii] = new uint__t [Batch_size+1];
					for (int i = 0; i < (Batch_size + 1); i++)
					{
						Rown[jj * Num_Blocks + ii][i] = 0;
					}
				}
				continue;
			}
			
			//malloc GPU csr column index and value
			cudaStat = hipMalloc ((void**)&csrValA, sizeof(real__t) * nnzTotalDevHostPtr) ;
			if (cudaStat != HIPBLAS_STATUS_SUCCESS)
				return cudaStat;
			cudaStat = hipMalloc ((void**)&csrColIndA, sizeof(int) * nnzTotalDevHostPtr) ;
			if (cudaStat != HIPBLAS_STATUS_SUCCESS)
				return cudaStat;
			sparseStat = hipsparseSdense2csr(sparseHandle, Batch_size, Batch_size, descrA, devCormat, Batch_size, nnzPerRowColumn, csrValA, csrRowPtrA, csrColIndA);
			if (sparseStat != HIPSPARSE_STATUS_SUCCESS)
				return sparseStat;

			//3. transfer
						
			cudaStat = hipMemcpy(Column[ii * Num_Blocks + jj], csrColIndA, sizeof(int) * nnzTotalDevHostPtr, hipMemcpyDeviceToHost);
			if (cudaStat != hipSuccess)
				return cudaStat;
				 
			cudaStat = hipMemcpy(Value[ii * Num_Blocks + jj], csrValA, sizeof(real__t) * nnzTotalDevHostPtr, hipMemcpyDeviceToHost);
			if (cudaStat != hipSuccess)
				return cudaStat;
			
			cudaStat = hipMemcpy(Rown[ii * Num_Blocks + jj], csrRowPtrA, sizeof(int) * (Batch_size+1), hipMemcpyDeviceToHost);
			if (cudaStat != hipSuccess)
				return cudaStat;
			
			if(Rown[ii * Num_Blocks + jj][Batch_size]!=nnzTotalDevHostPtr)
			{
				cout<<"checking error diagnoal:"<<Rown[ii * Num_Blocks + jj][Batch_size]<<endl;
			}

			//3.5 need transposition if ii!=jj
			if (ii!=jj)
			{
				real__t *cscVal;
				int *cscRowInd, *cscColPtr;
				cudaStat = hipMalloc ((void**)&cscVal, sizeof(real__t) * nnzTotalDevHostPtr) ;
				if (cudaStat != HIPBLAS_STATUS_SUCCESS)
					return cudaStat;
				cudaStat = hipMalloc ((void**)&cscColPtr, sizeof(int) * (Batch_size + 1)) ; //transposed R
				if (cudaStat != HIPBLAS_STATUS_SUCCESS)
					return cudaStat;
				cudaStat = hipMalloc ((void**)&cscRowInd, sizeof(int) *  nnzTotalDevHostPtr) ; //transposed C
				if (cudaStat != HIPBLAS_STATUS_SUCCESS)
					return cudaStat;
				sparseStat = hipsparseScsr2csc(sparseHandle, Batch_size, Batch_size, nnzTotalDevHostPtr, csrValA, csrRowPtrA, csrColIndA, cscVal, cscRowInd, cscColPtr, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO);
				if (sparseStat !=HIPSPARSE_STATUS_SUCCESS)
					return sparseStat;
									
				cudaStat = hipMemcpy(Column[jj * Num_Blocks + ii], cscRowInd, sizeof(int) * nnzTotalDevHostPtr, hipMemcpyDeviceToHost);
				if (cudaStat != hipSuccess)
					return cudaStat;
				cudaStat = hipMemcpy(Value[jj * Num_Blocks + ii], cscVal, sizeof(real__t) * nnzTotalDevHostPtr, hipMemcpyDeviceToHost);
				if (cudaStat != hipSuccess)
					return cudaStat;
				cudaStat = hipMemcpy(Rown[jj * Num_Blocks + ii], cscColPtr, sizeof(int) * (Batch_size+1), hipMemcpyDeviceToHost);
				if (cudaStat != hipSuccess)
				   return cudaStat;
				if (Rown[jj * Num_Blocks + ii][Batch_size]!=nnzTotalDevHostPtr)
				{
					cout<<"checking error non-diagnoal:"<<Rown[jj * Num_Blocks + ii][Batch_size]<<endl;
				}
				/*	for (int i = 0; i < nnzTotalDevHostPtr; i++)
					{
						if (Value[jj * Num_Blocks + ii][i] == 0)
						{
							cout<<"How come?"<<endl;
						}
					}*/
				hipFree(cscVal);
				hipFree(cscRowInd);
				hipFree(cscColPtr);
			}

			if(ii==jj)
				totalNonzero += nnzTotalDevHostPtr;
			else
			{
				totalNonzero += nnzTotalDevHostPtr * 2 ;
			}

			//4.1 free GPU CSR column index and value.
			hipFree(csrValA);
			hipFree(csrColIndA);
		}
		cout<<"Fulfill the "<<ii+1<<"th disposition."<<endl;
	}

	//4.2 free nnzPerRowColumn and csrRowPtrA.
	hipFree(nnzPerRowColumn);
	hipFree(csrRowPtrA);
	sparseStat = hipsparseDestroyMatDescr(descrA);
	if (sparseStat != HIPSPARSE_STATUS_SUCCESS)
		return sparseStat;
	sparseStat = hipsparseDestroy(sparseHandle);
	if (sparseStat != HIPSPARSE_STATUS_SUCCESS)
		return sparseStat;


	long long Rcheck = 0;
	for (int ii = 0; ii < Num_Blocks; ii++)
	{
		for (uint__t jj = 0; jj < Num_Blocks; jj++)
		{
			Rcheck += Rown[ii*Num_Blocks+jj][Batch_size];
			for (int x = 0; x < Batch_size+1; x++)
			{
				Row[x+ii*Batch_size] += Rown[ii*Num_Blocks+jj][x];
			}
		}
		for (uint__t y = ii*Batch_size+Batch_size+1; y < (Num_Blocks * Batch_size+1); y++)
		{
			Row[y] = Row[Batch_size+ii*Batch_size];
		}
	}
	cout<<"Rcheck:"<<Rcheck<<endl;
	cout<<"Row[N]:"<<Row[N]<<endl;
	if (Row[N] != totalNonzero )
	{
		cout<<"error:"<<"R values abnormal!"<<endl;
		return false;
	}

	/****************    Write CSR networks   ****************/
	correlationTime = clock() - correlationTime;
	cout<<"correlation time: "<<correlationTime<<"ms"<<endl;
	//cout<<"overall time for histogram plus correlation: "<<*aggregrate<<"ms"<<endl;
	
	unsigned int FreeMem = 0;
	MEMORYSTATUS MemStat;
	MemStat.dwLength = sizeof(MEMORYSTATUS);
	GlobalMemoryStatus(&MemStat);
	FreeMem = TOM(MemStat.dwAvailPhys);
	cout << "bytes of physical memory: " << TOM(MemStat.dwTotalPhys) <<"M" <<endl;
	cout << "percent of memory in use: " << MemStat.dwMemoryLoad <<"%" <<endl;
	cout << "free physical memory bytes: " << TOM(MemStat.dwAvailPhys) <<"M" <<endl;
	cout<<"number of non-zero elements: "<<Row[N]<<endl;
	long long M1 = (N-1);
	M1 *= N;	
	real__t spa = 100.0 * Row[N] / M1;
	char sparsity[100];
	sprintf(sparsity, "_spa%.3f%%_cor%.3f", spa,*r_thresh);
	string Outfilename = OutCor;
	Outfilename.append(string(sparsity)).append("_weighted.csr");
	ofstream fout;
	cout<<"generating "<<Outfilename.c_str()<< "..."<<endl;
	fout.open(Outfilename.c_str(), ios::binary | ios::out);
	if (!fout)
	{
		cout<<"create outfile unsuccessfully. error code:  "<<GetLastError()<<endl;
		exit(false);
	}
	uint__t Rlength = N+1;
	fout.write((char*)&Rlength, sizeof(uint__t));
	fout.write((char*)Row, sizeof(long long) * Rlength);
	long long Clength = Row[N];
	fout.write((char*)&Clength, sizeof(long long));
	for (int ii = 0; ii < Num_Blocks; ii++)
	{
		for (int i = 0; i < Batch_size; i++)
		{
			for (int jj = 0; jj < Num_Blocks; jj++)
			{
				if (Rown[ii*Num_Blocks+jj][i]==Rown[ii*Num_Blocks+jj][i+1])
					continue;
				else
				{
					for (uint__t j = Rown[ii*Num_Blocks+jj][i]; j < Rown[ii*Num_Blocks+jj][i+1]; j++)
					{
						//cout<<Column[ii*Num_Blocks+jj][j]<<endl;
						Column[ii*Num_Blocks+jj][j] += jj * Batch_size;
						fout.write((char*)&Column[ii*Num_Blocks+jj][j], sizeof(uint__t));
					}
				}
			}
		}
	}
	fout.write((char*)&Clength, sizeof(long long));
	for (int ii = 0; ii < Num_Blocks; ii++)
	{
		for (int i = 0; i < Batch_size; i++)
		{
			for (int jj = 0; jj < Num_Blocks; jj++)
			{
				if (Rown[ii*Num_Blocks+jj][i]==Rown[ii*Num_Blocks+jj][i+1])
					continue;
				else
				{
					for (uint__t j = Rown[ii*Num_Blocks+jj][i]; j < Rown[ii*Num_Blocks+jj][i+1]; j++)
					{
						fout.write((char*)&Value[ii*Num_Blocks+jj][j], sizeof(real__t));
					}
				}
			}
		}
	}
	fout.close();
	cout<<"Transmition finished."<<endl;
	
	/****************    Write FCS information   ****************/
	real__t *fcs = new real__t[N];
	memset(fcs,0,sizeof(real__t)*N);	
	cudaStat = hipMemcpy(fcs, fcsGPU, sizeof(real__t) * N, hipMemcpyDeviceToHost);
	if (cudaStat != hipSuccess) 
		return cudaStat;
	string nmOutfilename = OutCor;
	nmOutfilename.append("_weighted_fcs.nm");
	ofstream fot;
	fot.open(nmOutfilename.c_str(), ios::binary | ios::out);
	if (!fot)
	{
		cout<<"create unsuccessfully. error code:  "<<GetLastError()<<endl;
		exit(false);
	}
	int length = N;
	fot.write((char*)&length, sizeof(int));
	for (int i = 0; i < N; i++)
	{
		fot.write((char*)&fcs[i], sizeof(real__t));
	}
	fot.close();
	delete[] fcs;
	hipFree(fcsGPU);

	return true;
}

void MatrixMultiplication(real__t * BOLD_t1, real__t * BOLD_t2,real__t * out,int Batch_size,int L)
{
	long kk = 0;
	for (int k = 0; k < Batch_size; k++)
	{
		for (int i = 0; i < Batch_size; i++)
		{   
			double sum3 = 0.0;
			for (int j = 0; j < L; j++)
			{
				sum3 += 1.0*BOLD_t1[k*L+j] * BOLD_t2[i*L+j];
			}
			out[kk++] = sum3;
		}
	}
	
}








	