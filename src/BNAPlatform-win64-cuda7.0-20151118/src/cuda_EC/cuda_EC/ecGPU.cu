#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <iomanip>
#include <memory.h>
#include <fstream>
#include <iostream>
#include <cstring>
#include "dirent.h" 
#include ""  //ͬ�������Ĳ����߲�������
#include "hip/device_functions.h"
#include "modularity_GPU.cuh"
#include <cmath>
#include <time.h> 
#include "hipblas.h"
#include "hipsparse.h"
#define CLEANUP(s)   printf ("%s\n", s)   //cusparse�ģ�ÿ�����֮��Ҫfree�����������еı�������������ٿ���                           
#define CUBLAS_ERROR_CHECK(sdata) if(HIPBLAS_STATUS_SUCCESS!=sdata){printf("ERROR at:%s:%d\n",__FILE__,__LINE__);}//exit(-1);}  
#pragma comment(lib,"cublas.lib")
#pragma comment(lib,"cusparse.lib")

const int MAX_ITER=10000 ;			// The maximum iteration times in the power method
const int ITERNUMBER=500;
const double BETA_Adjust = 0;		// An optional parameter for quicker convergence. Its effect is uncertain
const double Epsilon = 0.000001;	// If |x - x0| < Epsilon, quit iteraion 
const double LAMBDA = 0.01;		// if labmda > LAMBDA, initiate the division
const int MIN_GROUP = 1;

const int threadnumx = 16;
const int threadnumy = 16;
const int  threadnum = 256;
const int blocknum    = 48;
extern ofstream fout;

__global__ void init_AD(long N, double *AD)
{
	int tid = blockDim.x*blockIdx.x+threadIdx.x;  //����һά�ģ�
	for (int i = tid; i<N; i+=blockDim.x*gridDim.x) 
		AD[i] = i;
}
__global__ void init_unweightednet(long nnz, double *V)
{
	int tid = blockDim.x*blockIdx.x+threadIdx.x;  
	for (int i = tid; i<nnz; i+=blockDim.x*gridDim.x) 
		V[i] = 1;
}
 //�������
 __global__ void vvplus (long N, double *result, double *v0, double alpha, double *v1, double beta)    //���������ӷ�
 {
	 const int blockid   = blockIdx.x;
	 const int threadid  = threadIdx.x;
	 int offset;  	 
	 for(offset=threadid+blockid*threadnum; offset<N; offset+=threadnum*gridDim.x)
		 result[offset]=(alpha*v0[offset]+beta*v1[offset]);
 }
/**********�������ܣ���Գơ�ϡ�����������������Ķȣ��������ηֱ�Ϊ��ƫ�ơ��кš�Ԫ��ֵ��Ntemp*Ntemp*********************/
      
double Lead_Vector_GPU(int *R, int *C, int *V, long Ntemp,double *v)  //beta�����ò��ÿ��� ��beta��d_u0,d_uu��ϵ���ˣ���
{
	
	//��������
    hipError_t cudaStat1,cudaStat2,cudaStat3,cudaStat4,cudaStat5,cudaStat6,cudaStat7;
	//int* d_AD_init;
	long M=R[Ntemp]/2;//	Ntemp�Ǿ�������У�M��ϡ���ĸ�����һ�룡��
	int* d_r; 
    int* d_c;
	double* d_v;
	double* d_u;
	double* d_u0;
	double * d_vector;
	double* y;  
	
	long long i = 0, j = 0;

	hipsparseStatus_t status;
	hipsparseHandle_t cushandle;
	hipsparseMatDescr_t descrA=0;
	const double alpha_mv= 1.0;
	const double beta_mv= 0.0;

	//cublas��һЩ������ʼ��
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	
	stat = hipblasCreate(&handle) ;
	//cusparse��һЩ������ʼ���������Ӣ��ע���������й��е�
	//initialize cusparse library 
	status= hipsparseCreate(&cushandle); 
	if (status != HIPSPARSE_STATUS_SUCCESS) 
	{  printf("CUSPARSE Library initialization failed");
	   hipsparseDestroy(cushandle); 
	} 
	//create and setup matrix descriptor 
	status= hipsparseCreateMatDescr(&descrA); 
	if (status != HIPSPARSE_STATUS_SUCCESS) 
	{  printf("Matrix descriptor initialization failed"); 
	   hipsparseDestroyMatDescr(descrA);
	   return 1;                   
	} 
	hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL); 
	hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO); 
	// another parameters 
	hipsparseOperation_t transA= HIPSPARSE_OPERATION_NON_TRANSPOSE;


	double err1 = 1, err2 = 1;
	int ITER = 0;
	double vNorm = 0;
	double temp2= -1;
	double temp1=0;

    double v_k;
	//��һ��  ����ռ䣻��ʼ��d_u���൱�ڵ�����ʽ�е�x[k]�����������R,C����GPU��
	cudaStat1= hipMalloc( (void**) &d_v, sizeof(double) * (2*M));
	cudaStat2= hipMalloc( (void**) &d_r, sizeof(int) * (Ntemp + 1));
	cudaStat3= hipMalloc( (void**) &d_c, sizeof(int) *(2*M));
	cudaStat4= hipMalloc( (void**) &d_u, sizeof(double) * Ntemp);
	cudaStat5= hipMalloc( (void**) &d_u0, sizeof(double) * Ntemp);
	cudaStat6= hipMalloc( (void**) &y, sizeof(double) * Ntemp);	
	cudaStat7= hipMalloc( (void**) &d_vector, sizeof(double) * Ntemp);
	if( (cudaStat1 != hipSuccess)||
			(cudaStat2 != hipSuccess)||
			(cudaStat3 != hipSuccess)||
			(cudaStat4 != hipSuccess)||
			(cudaStat5 != hipSuccess)||
			(cudaStat6 != hipSuccess)||
			(cudaStat7 != hipSuccess))
	{
	 CLEANUP(" Device malloc failed");
	}	
    init_AD<<<blocknum,threadnum>>>(Ntemp, d_u);
	cudaStat1 = hipMemcpy(d_r, R, 
                           (size_t)((Ntemp+1)*sizeof(d_r[0])), 
                           hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_c, C, 
                           (size_t)(2*M*sizeof(d_c[0])), 
						    hipMemcpyHostToDevice);
	if ((cudaStat1 != hipSuccess) ||
        (cudaStat2 != hipSuccess) 
        ) {
        CLEANUP("Memcpy from Host to Device failed");
        return 1;
    }
	//�ڶ������ж��Ƿ�Ϊ��Ȩ���磬���ǣ���Ԫ��ֵ����gpu�������ǣ���gpuֱ������Ԫ��ֵ
	if(*V==NULL)  //����Ȩ ��ʼ��Ϊ1
	{
	init_unweightednet<<<blocknum,threadnum>>>(2*M, d_v);
	}
	else   //��Ȩ ����ȥ
	{
		cudaStat1 = hipMemcpy(d_v, V, 
                           (size_t)(2*M*sizeof(d_v[0])), 
                           hipMemcpyHostToDevice);
	if (cudaStat1 != hipSuccess) 
      {
        CLEANUP("Memcpy from Host to Device failed");
    }
	}
	                                         //wocao�����ֵ�����ˣ�������        //soga!�ѵ���ǰ��һ�£���������
	//��������ѭ��;<1>Y(k) = X(k)/�U X(k)�U��;<2>X(k+1) = AY(k) k=0,1,2,��;<3>�жϣ���k��ִ�ʱ���򵱨U X(k)- X(k+1)�U <��ʱ������ѭ��;<4>�����Y(k)��V1,max |Xj(k)| �� ��1 ,1��j��nΪx(k)�ĵ�j������
	while (err1 > Epsilon &&  err2 > Epsilon && ITER < MAX_ITER)
	{	  		
          //3.1�Ȱ�d_u����d_u0
		  hipblasDcopy(handle, (int) Ntemp, d_u, 1 ,d_u0, 1 );
         //3.2ѭ����һ�� ��һ��	   
		  hipblasDnrm2(handle, Ntemp, d_u, 1, &vNorm);            //˼�룺���ܳ������ַ�������������������������һ������ͬ�ģ�
		  temp1=1/vNorm;                                          //du����du�ķ�����Ŀ����Ϊ�˷�ֹ������ʧ�������ַ�������,��֤������2����
		  hipblasDscal (handle, (int) Ntemp, &temp1, d_u, 1);   //Normalize v, v[i] = v[i]/vNorm Ŷ�������v��i���������Լ��ķ�������һ����      
	     checkCudaErrors( hipMemcpy(v,d_u, sizeof(double) * Ntemp,  hipMemcpyDeviceToHost) );
		  //ע����һ����ʱ�䣬�Ż�ʱ�����Ż���
		 //3.3ѭ���ڶ��� �����������
		  status= hipsparseDcsrmv(cushandle,  transA,  Ntemp,  Ntemp,  2*M, 

		&alpha_mv,  descrA,  d_v,  d_r,      //��Ϊ���d_vҪ�����double ����ǰ�涼�øĳ�double��
		d_c,  d_u,  &beta_mv,  y);
	if (status != HIPSPARSE_STATUS_SUCCESS) 
	{ 
		CLEANUP("Matrix-vector multiplication failed");
		//	return 1; 
	} 
	cudaStat1 = hipMemcpy(d_u, y, 
		(size_t)(Ntemp*sizeof(d_v[0])), 
                           hipMemcpyDeviceToDevice);
	if (cudaStat1 != hipSuccess) 
      {
        CLEANUP("Memcpy from Device to Device failed");
    }
       //3.4 �ж�
		vvplus<<<blocknum, threadnum>>>((long) Ntemp, d_vector, d_u, 1.0, d_u0, -1.0);
	    hipblasDnrm2(handle, Ntemp, d_vector, 1, &err1);   //xk-x��k-1���ķ���
		vvplus<<<blocknum, threadnum>>>((long) Ntemp, d_vector, d_u, 1.0, d_u0, 1.0);
		hipblasDnrm2(handle, Ntemp, d_vector, 1, &err2);   //xk+x��k-1���ķ���                    //��Щ��������������Ҫ�Ĳο������ǰٶȰٿ�
				 
		ITER++;
	}	 
	cout<<"Iterations:\t"<<ITER<<'\t'<<"residual:\t"<<min(err1, err2)<<'\t';
	fout<<"Iterations:\t"<<ITER<<'\t'<<"residual:\t"<<min(err1, err2)<<'\t';
	
	//���Ĳ�:�ͷ��ڴ棬�������ֵ
	hipblasDestroy(handle);
	hipFree(y);
	hipsparseDestroyMatDescr(descrA);
    hipsparseDestroy(cushandle);
//	double *v = new double [Ntemp];
	double *v0 = new double [Ntemp];
//	checkCudaErrors( hipMemcpy( v, d_u, sizeof(double) * Ntemp , hipMemcpyDeviceToHost) ); 
	checkCudaErrors( hipMemcpy(v0,d_u0, sizeof(double) * Ntemp,  hipMemcpyDeviceToHost) );
/*		long long max_index = 0;
	for (i = 0; i < Ntemp; i++)
		if (fabs(v0[i]) > fabs(v0[max_index]))
			max_index = i;  */
	return vNorm ;   //what ??????????
	//return v0[max_index];
//	for (i = 0; i < Ntemp; i++)
//		v[i]/=v[max_index];
}
void main(){
	  /* create the following sparse test matrix in CSR format */
    /* |0.0     1.0 1.0|
       |    0.0 1.0    |
       |1.0 1.0 0.0 1.0|
       |1.0     1.0 0.0| */
	int C[10]={2,3,2,0,1,3,0,2};
	int R[5]={0,2,3,6,8};
	float x;
	double *v = new double [4];
	int V=NULL;
	x=Lead_Vector_GPU(R, C, &V, 4,v);
	 for(int i=0;i<4;i++)   {   
		 printf("\n%f\n",v[i]);
	 }
	 printf("eigenvalue is %f\n",x);
}
