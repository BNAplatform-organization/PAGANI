#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#define N 4
#define eps 1e-6
#define KM 30
float MaxValue(float x[],int n)
{     float Max=x[0];     
      int i;     
      for (i=1;i<n;i++)         
     if(fabs(x[i])>fabs(Max))Max=x[i];         
	  return Max; 
}
void PowerMethod(float *A)
{     float U[N],V[N],EC[N],r1,r2,temp;     
      int i,j,k=0;     
	  for(i=0;i<N;i++)U[i]=1;     
	  while(k<KM)     
	  {         
		  k++;         
		  for(i=0;i<N;i++)         
		  {             
			  temp=0;             
			  for(j=0;j<N;j++)
				  temp+=*(A+i*N+j)*U[j];             
			  V[i]=temp;         //V=A*U
		  }         
		  for(i=0;i<N;i++)
		  {
			  U[i]=V[i]/MaxValue(V,N);       //U=V/Lamda  
			  EC[i]=abs(U[i]);                //ȡ����ֵ
		  }
			  if(k==1)
			  r1=MaxValue(V,N);         
		  else 
			  r2=MaxValue(V,N);         
		      if
			  (fabs(r2-r1)<eps)
			  break;         
		   r1=r2;                        //��������
	  }          
	  printf("r=%f\n",r2);          
	  for(i=0;i<N;i++)
		  printf("x[%d]=%f\n",i+1,EC[i]);
} 
void main()
{     
	//float A[N][N]={{2,-1,0},{-1,2,-1},{0,-1,2}} ;     
	//float A[N][N]={{1,0,0,0},{1,2,0,0},{-3,-3,-1,0},{-2,-2,-2,-3}} ;     
	float A[N][N]={{0,0,1,1},{0,0,1,0},{1,3,0,1},{1,0,1,0}} ;     
	PowerMethod(A[0]);
} 