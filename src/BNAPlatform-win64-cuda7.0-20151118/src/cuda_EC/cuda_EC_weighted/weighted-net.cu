#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <iomanip>
#include <memory.h>
#include <fstream>
#include <iostream>
#include <cstring>
#include ""  //ͬ�������Ĳ����߲�������
#include "hip/device_functions.h"
#include "dirent.h" 
#include <cmath>
#include <time.h> 
#include "Timer.h" 
#include "hipblas.h"
#include<hip/hip_runtime.h>
#include "hipsparse.h"

#define CLEANUP(s)   printf ("%s\n", s)   //cusparse�ģ�ÿ�����֮��Ҫfree�����������еı�������������ٿ���                           
#define CUBLAS_ERROR_CHECK(sdata) if(HIPBLAS_STATUS_SUCCESS!=sdata){printf("ERROR at:%s:%d\n",__FILE__,__LINE__);}//exit(-1);}  
#pragma comment(lib,"cublas.lib")
#pragma comment(lib,"cusparse.lib")

 using namespace std;


const int MAX_ITER=10000 ;			// The maximum iteration times in the power method
const int ITERNUMBER=500;
const double BETA_Adjust = 0;		// An optional parameter for quicker convergence. Its effect is uncertain
const double Epsilon = 0.000001;	// If |x - x0| < Epsilon, quit iteraion 
const double LAMBDA = 0.01;		// if labmda > LAMBDA, initiate the division
const int MIN_GROUP = 1;

const int threadnumx = 16;
const int threadnumy = 16;
const int  threadnum = 256;
const int blocknum    = 48;

__global__ void init_AD(long N, float *AD)
{
	int tid = blockDim.x*blockIdx.x+threadIdx.x;  //����һά�ģ�
	for (int i = tid; i<N; i+=blockDim.x*gridDim.x) 
		AD[i] = i;
}
 //�������
__global__ void vvplus (long N, float *result, float *v0, double alpha, float *v1, double beta)    //���������ӷ�
 {
	 const int blockid   = blockIdx.x;
	 const int threadid  = threadIdx.x;
	 int offset;  	 
	 for(offset=threadid+blockid*threadnum; offset<N; offset+=threadnum*gridDim.x)
		 result[offset]=(alpha*v0[offset]+beta*v1[offset]);
 }
/**********�������ܣ���Գơ�ϡ�����������������Ķȣ��������ηֱ�Ϊ��ƫ�ơ��кš�Ԫ��ֵ��Ntemp*Ntemp*********************/
      
 double Lead_Vector_GPU(int *R, int *C, float *V, long Ntemp,double *v)  //beta�����ò��ÿ��� ��beta��d_u0,d_uu��ϵ���ˣ���
{
	
	//��������
    hipError_t cudaStat1,cudaStat2,cudaStat3,cudaStat4,cudaStat5,cudaStat6,cudaStat7;
	//int* d_AD_init;
	long M=R[Ntemp]/2.0;//	Ntemp�Ǿ�������У�M��ϡ���ĸ�����һ�룡��
	int* d_r; 
    int* d_c;
	float* d_v;
	float* d_u;
	float* d_u0;
	float * d_vector;
	float* y;  
	
	long long i = 0, j = 0;

	hipsparseStatus_t status;
	hipsparseHandle_t cushandle;
	hipsparseMatDescr_t descrA=0;
	const float alpha_mv= 1.0;
	const float beta_mv= 0.0;

	//cublas��һЩ������ʼ��
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	
	stat = hipblasCreate(&handle) ;
	//cusparse��һЩ������ʼ���������Ӣ��ע���������й��е�
	//initialize cusparse library 
	status= hipsparseCreate(&cushandle); 
	if (status != HIPSPARSE_STATUS_SUCCESS) 
	{  printf("CUSPARSE Library initialization failed");
	   hipsparseDestroy(cushandle); 
	} 
	//create and setup matrix descriptor 
	status= hipsparseCreateMatDescr(&descrA); 
	if (status != HIPSPARSE_STATUS_SUCCESS) 
	{  printf("Matrix descriptor initialization failed"); 
	   hipsparseDestroyMatDescr(descrA);
	   return 1;                   
	} 
	hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL); 
	hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO); 
	// another parameters 
	hipsparseOperation_t transA= HIPSPARSE_OPERATION_NON_TRANSPOSE;


	float err1 = 1, err2 = 1;
	int ITER = 0;
	float vNorm = 0;
	float temp2= -1;
	float temp1=0;

 //   double v_k;
	//��һ��  ����ռ䣻��ʼ��d_u���൱�ڵ�����ʽ�е�x[k]�����������R,C����GPU��
	cudaStat1= hipMalloc( (void**) &d_v, sizeof(float) * (2*M));
	cudaStat2= hipMalloc( (void**) &d_r, sizeof(int) * (Ntemp + 1));
	cudaStat3= hipMalloc( (void**) &d_c, sizeof(int) *(2*M));
	cudaStat4= hipMalloc( (void**) &d_u, sizeof(float) * Ntemp);
	cudaStat5= hipMalloc( (void**) &d_u0, sizeof(float) * Ntemp);
	cudaStat6= hipMalloc( (void**) &y, sizeof(float) * Ntemp);	
	cudaStat7= hipMalloc( (void**) &d_vector, sizeof(float) * Ntemp);
	if( (cudaStat1 != hipSuccess)||
			(cudaStat2 != hipSuccess)||
			(cudaStat3 != hipSuccess)||
			(cudaStat4 != hipSuccess)||
			(cudaStat5 != hipSuccess)||
			(cudaStat6 != hipSuccess)||
			(cudaStat7 != hipSuccess))
	{
	 CLEANUP(" Device malloc failed");
	}	
    init_AD<<<blocknum,threadnum>>>(Ntemp, d_u);
	cudaStat1 = hipMemcpy(d_r, R, 
                           (size_t)((Ntemp+1)*sizeof(d_r[0])), 
                           hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_c, C, 
                           (size_t)(2*M*sizeof(d_c[0])), 
						    hipMemcpyHostToDevice);
	if ((cudaStat1 != hipSuccess) ||
        (cudaStat2 != hipSuccess) 
        ) {
        CLEANUP("Memcpy from Host to Device failed");
        return 1;
    }
	//�ڶ�������Ԫ��ֵ����gpu;
	
		cudaStat1 = hipMemcpy(d_v, V, 
                           (size_t)(2*M*sizeof(d_v[0])), 
                           hipMemcpyHostToDevice);
	if (cudaStat1 != hipSuccess) 
      {
        CLEANUP("Memcpy from Host to Device failed");
    }
	
	                                         //wocao�����ֵ�����ˣ�������        //soga!��ǰ��һ�£��������������õ��Ƕ�����
	//��������ѭ��;<1>Y(k) = X(k)/�U X(k)�U��;<2>X(k+1) = AY(k) k=0,1,2,��;<3>�жϣ���k��ִ�ʱ���򵱨U X(k)- X(k+1)�U <��ʱ������ѭ��;<4>�����Y(k)��V1,max |Xj(k)| �� ��1 ,1��j��nΪx(k)�ĵ�j������
	while (err1 > Epsilon &&  err2 > Epsilon && ITER < MAX_ITER)
	{	  		
          //3.1�Ȱ�d_u����d_u0
		  hipblasScopy(handle, (int) Ntemp, d_u, 1 ,d_u0, 1 );
         //3.2ѭ����һ�� ��һ��	   
		  hipblasSnrm2(handle, Ntemp, d_u, 1, &vNorm);            //˼�룺���ܳ������ַ�������������������������һ������ͬ�ģ�
		  temp1=1/vNorm;                                          //du����du�ķ�����Ŀ����Ϊ�˷�ֹ������ʧ�������ַ�������,��֤������2����
		  hipblasSscal (handle, (int) Ntemp, &temp1, d_u, 1);   //Normalize v, v[i] = v[i]/vNorm Ŷ�������v��i���������Լ��ķ�������һ����      
	    cudaStat1= hipMemcpy(v,d_u, sizeof(double) * Ntemp,  hipMemcpyDeviceToHost) ;//���������յ�����ˣ�
		if (cudaStat1 != hipSuccess) 
      {
        CLEANUP("Memcpy from Device to Host failed");
       }
	
		  //ע����һ����ʱ�䣬�Ż�ʱ�����Ż���
		 //3.3ѭ���ڶ��� �����������
		  status= hipsparseScsrmv(cushandle,  transA,  Ntemp,  Ntemp,  2*M, 

		&alpha_mv,  descrA,  d_v,  d_r,      //��Ϊ���d_vҪ�����double ����ǰ�涼�øĳ�double��
		d_c,  d_u,  &beta_mv,  y);
	if (status != HIPSPARSE_STATUS_SUCCESS) 
	{ 
		CLEANUP("Matrix-vector multiplication failed");
		//	return 1; 
	} 
	cudaStat1 = hipMemcpy(d_u, y, 
		(size_t)(Ntemp*sizeof(d_v[0])), 
                           hipMemcpyDeviceToDevice);
	if (cudaStat1 != hipSuccess) 
      {
        CLEANUP("Memcpy from Device to Device failed");
    }
       //3.4 �ж�
		vvplus<<<blocknum, threadnum>>>((long) Ntemp, d_vector, d_u, 1.0, d_u0, -1.0);
	    hipblasSnrm2(handle, Ntemp, d_vector, 1, &err1);   //xk-x��k-1���ķ���
		vvplus<<<blocknum, threadnum>>>((long) Ntemp, d_vector, d_u, 1.0, d_u0, 1.0);
		hipblasSnrm2(handle, Ntemp, d_vector, 1, &err2);   //xk+x��k-1���ķ���                    //��Щ��������������Ҫ�Ĳο������ǰٶȰٿ�
				 
		ITER++;
	}	 
	cout<<"Iterations:\t"<<ITER<<'\t'<<"residual:\t"<<min(err1, err2)<<'\t'<<"eigenvalue:\t"<<vNorm<<'\t';
//	fout<<"Iterations:\t"<<ITER<<'\t'<<"residual:\t"<<min(err1, err2)<<'\t';
	
	//���Ĳ�:�ͷ��ڴ棬�������ֵ
	hipblasDestroy(handle);
	hipFree(y);
	hipFree(d_r);
	hipFree(d_c);
	hipFree(d_u);
	hipFree(d_vector);
	hipFree(d_v);
	hipsparseDestroyMatDescr(descrA);
    hipsparseDestroy(cushandle);
//	double *v = new double [Ntemp];
	double *v0 = new double [Ntemp];
//	checkCudaErrors( hipMemcpy( v, d_u, sizeof(double) * Ntemp , hipMemcpyDeviceToHost) ); 
	cudaStat1= hipMemcpy(v0,d_u0, sizeof(double) * Ntemp,  hipMemcpyDeviceToHost) ;
	if (cudaStat1 != hipSuccess) 
      {
        CLEANUP("Memcpy from Device to Host failed");
    }
	return vNorm ; 
	

} 
int main(int argc, char * argv[]){

	//step 1��file in
	 DIR *dp;
	struct dirent *dirp;
	if (NULL == (dp = opendir(argv[1])))
	{
		printf("can't open %s", argv[1]);
		exit (1);
	}
	int FileNumber = 0;
	string filenametmp;
	while((dirp = readdir(dp)) != NULL)
	{
		filenametmp = string(dirp->d_name);
		
		if (filenametmp.find_last_of('.') == -1)
			continue;
		if(filenametmp.length()>4 && filenametmp.substr(filenametmp.find_last_of('.'),4).compare(".csr") == 0 && filenametmp.size() - filenametmp.find_last_of('.') - 1 == 3)
		{
			FileNumber++;
		}
	}
	cout<<FileNumber<<" files to be processed."<<endl;

	closedir(dp);
	string *filename = new string[FileNumber];
	dp = opendir(argv[1]);
	int i = 0;
	while((dirp = readdir(dp)) != NULL)
	{
		filenametmp = string(dirp->d_name);
		if (filenametmp.find_last_of('.') == -1)
			continue;
		if(filenametmp.length()>4 && filenametmp.substr(filenametmp.find_last_of('.'),4).compare(".csr") == 0 && filenametmp.size() - filenametmp.find_last_of('.') - 1 == 3)
		{
			filename[i++] = filenametmp;
		}
	}

	string isolated_v_file = string(argv[1]).append("\\").append("isolated_v_mark.txt");
	ofstream iso_file;
	iso_file.open(isolated_v_file.c_str(), ios::out);

	for (int i = 0; i < FileNumber; i++)
	{
		string a = string(argv[1]).append("\\").append(filename[i]);
		cout<<"\ncalculating eigenvalue centrality for "<<a.c_str()<<" ..."<<endl;
		ifstream fin(a.c_str(), ios_base::binary);
		if (!fin.good())
		{	cout<<"Can't open\t"<<a.c_str()<<endl;	return 0;}

		// Read x.csr
		int Rlength = 0, Clength = 0, Clength1=0;
		fin.read((char*)&Rlength, sizeof(int));
		int * R = new int [Rlength];
		fin.read((char*)R, sizeof(int) * Rlength);
		fin.read((char*)&Clength, sizeof(int));
		int * C = new int [Clength];
		fin.read((char*)C, sizeof(int) * Clength);
		fin.read((char*)&Clength1, sizeof(int));
		float * V = new float [Clength];
		fin.read((char*)V, sizeof(float) * Clength);
		fin.close();
		int N = Rlength - 1;
		//step 2��use leading_vector function
		double *v = new double [N];
		//float *V=NULL;
	    Setup(0);
		Start(0);
		float  x=Lead_Vector_GPU(R, C, V, N,v);
		Stop(0);
		cout<<"calculate time: "<<GetElapsedTime(0)<<" s."<<endl;
		//step 3��file out
	   // Parse file name
		string X_cp = a.substr(0, a.find_last_of('.') + 1).append("ec");
		string X_cp_mas = a.substr(0, a.find_last_of('.')).append("_ec.txt");
		cout<<"Save eigenvector centrality for each node as "<<X_cp.c_str()<<endl;
		ofstream fout;
		fout.open(X_cp.c_str(), ios::binary|ios::out);
		fout.write((char*)&N, sizeof(int));
		fout.write((char*)v, sizeof(double) * N);
		fout.write((char*)&x, sizeof(float));
		fout.close();
		FILE *fp=fopen(X_cp_mas.c_str(),"w");
		fprintf(fp,"the eigenvector centrality for each node is \n");
		for(long w=0;w<N;w++)
		{
			fprintf(fp,"%.15lg\n",v[w]);
		}
		fclose(fp);

}
}