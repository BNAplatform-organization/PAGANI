#include <stdlib.h>
#include <stdio.h>
#include <iomanip>
#include <memory.h>
#include <fstream>
#include <iostream>
#include <cstring>

#include ""  //ͬ�������Ĳ����߲�������
#include "hip/device_functions.h"
#include<hip/hip_runtime.h>
#include <cmath>
#include <time.h> 
#include "hipblas.h"
#include "hipsparse.h"
#define CLEANUP(s)   printf ("%s\n", s)   //cusparse�ģ�ÿ�����֮��Ҫfree�����������еı�������������ٿ���                           
#define CUBLAS_ERROR_CHECK(sdata) if(HIPBLAS_STATUS_SUCCESS!=sdata){printf("ERROR at:%s:%d\n",__FILE__,__LINE__);}//exit(-1);}  
#pragma comment(lib,"cublas.lib")
#pragma comment(lib,"cusparse.lib")
void main()
{
double u[4]={2.0,5.0,2.0,4.0};
double* d_u;
double vNorm = 0;
hipblasHandle_t handle;
 hipError_t cudaStat4;
 hipblasStatus_t stat;
cudaStat4= hipMalloc( (void**) &d_u, sizeof(double) * 4);
if( cudaStat4 != hipSuccess){
CLEANUP(" Device malloc failed");
}
cudaStat4 = hipMemcpy(d_u, u, 
                           (size_t)(4*sizeof(d_u[0])), 
                           hipMemcpyHostToDevice);
if( cudaStat4 != hipSuccess){
 CLEANUP("Memcpy from Host to Device failed");
}
stat = hipblasCreate(&handle) ;  
hipblasDnrm2(handle, 4, d_u, 1, &vNorm);
  printf("%f",vNorm);
}