#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <cmath>
#include <ctime>
#include "help_func.cuh"
#include "histogram.h"
using namespace std;
typedef float real__t;
const int thread_num = 1024; //maybe redefinition
const int block_num = 30; 

// square<T> computes the square of a number f(x) -> x*x 
template <typename T> 
struct square { 
	T m;
	square(T _m){ m = _m; }
	__host__ __device__ T operator()(const T& x) 
		const { return (x-m) * (x-m); } 
}; 
template<class T>
struct normalize_functor{
	T m,n;
    
	normalize_functor(T _m,T _n){
        m = _m;
		n = _n;
    }

    __host__ __device__ T operator()(T &x) const{
		return (x - m)/n;
    }
};

__global__ void assignrank(real__t* L_begin, int L, int*  addr, real__t *devTimeSeriesRank)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;
	//record ties
	while(i<L)
	{
		
		if (i<(L-1)&&L_begin[i]<L_begin[i+1]&&L_begin[i]==L_begin[i-1] || i ==(L-1)&&L_begin[i]==L_begin[i-1]) //be care that all elements are identical.
		{
			int rightBound = i ;
			do{i--;}
			while(L_begin[i]== L_begin[rightBound]&&i!=0);
            int leftBound = i ;
			i = threadIdx.x + blockIdx.x * blockDim.x;
			real__t averageRank = (leftBound + rightBound + 1 + 1) / 2.0; //rank is 1-based index
			for (int j = leftBound; j <= rightBound; j++)
			{
				devTimeSeriesRank[addr[j]] = averageRank; 
			}
		}
		i += offset;
	}
	i = threadIdx.x + blockIdx.x * blockDim.x;
	while(i<L)
	{
		if (devTimeSeriesRank[addr[i]] == 0 )
		{
			devTimeSeriesRank[addr[i]] = i + 1; //rank is 1-based index
		}		    
		i += offset;
	}
}

//1 block for 1 sequence. L must greater than 1024!!!!!! sacrifice generality.
__global__ void assignrankBroadcast(real__t* begin, int L, int N)
{
	int I = threadIdx.x + blockIdx.x * blockDim.x;
	for(int k = blockIdx.x; k < N; k += gridDim.x)
	{
		int i = threadIdx.x ;
		int offset = blockDim.x ;
		//real__t* addr = begin + blockIdx.x * L;
		real__t* addr = begin + k * L;
		int lessOne = 0, lessTwo = 0;
		int equalOne = 0, equalTwo = 0;
		float benchmarkOne = 0, benchmarkTwo = 0;//interval is blockDim.x

		//1.update two benchmark	
		benchmarkOne = *(addr + i);
		
		benchmarkTwo = i+offset < L ? *(addr + i + offset):0;
		
		//2.update both equal and less
# ifdef myDebug
# if __CUDA_ARCH__>=200 //requires computing capability greater than 2.0
		if (I == 0 && k == 0)
		{
			printf("benchmarkOne is %d \n", benchmarkOne);
		}
# endif
# endif
		
		for (int j = 0; j < L; j++)
		{
			real__t temp = *(addr + j);
# ifdef myDebug
# if __CUDA_ARCH__>=200
			if (I == 0 && k == 0 && j == 0)
			{
				printf("temp is %d \n", temp); //note: error usage here since temp is real__t type.
			}
# endif
# endif
			lessOne += ( temp < benchmarkOne ? 1:0 );
			equalOne += ( temp == benchmarkOne ? 1:0 );
			lessTwo += ( temp < benchmarkTwo ? 1:0 );
			equalTwo += ( temp == benchmarkTwo ? 1:0 );
		}  
		//3. compute rank and assign
		__syncthreads();
# ifdef myDebug
# if __CUDA_ARCH__>=200
		if (I == 0 && k == 0)
		{
			printf("lessOne is %d \n", lessOne);
			printf("equalOne is %d \n", equalOne);
		}
# endif
# endif
		*(addr + i) = lessOne + 0.5 * ( 1.0 + (float)equalOne );
		if (i+offset<L)
		{
			*(addr + i + offset) = lessTwo + 0.5 * ( 1.0 + (float)equalTwo );
		}
		
	}		
}
//error scope for comparison to matlab results:less than 1e-5
//coud make comparison to algorithm of single variable handle but multiple all-broadcast.
__global__ void assignrankBroadcastSharedMemory(real__t* begin, int L, int N)
{
	extern __shared__ real__t s[];
	for(int k = blockIdx.x; k < N; k += gridDim.x)
	{
		int i = threadIdx.x ;
		int offset = blockDim.x ;
		//real__t* addr = begin + blockIdx.x * L;
		real__t* addr = begin + k * L;
		int lessOne = 0, lessTwo = 0;
		int equalOne = 0, equalTwo = 0;
		real__t benchmarkOne = 0, benchmarkTwo = 0;//interval is blockDim.x

		//1.transfer data from global memory to shared memory
		s[i] = addr[i];
		if (i+offset<L)
		{
			s[i + offset] = addr[i + offset];
		}
		 __syncthreads();
		//2.update two benchmark	
		benchmarkOne = s[i];
		
		benchmarkTwo = i+offset < L ? s[i + offset]:0;
		
		//3.update both equal and less
		
		for (int j = 0; j < L; j++)
		{
			real__t temp = s[j];
			
			lessOne += temp < benchmarkOne ? 1:0;
			equalOne += temp == benchmarkOne ? 1:0;
			lessTwo += temp < benchmarkTwo ? 1:0;
			equalTwo += temp == benchmarkTwo ? 1:0;
		}  
		//4. compute rank and assign
		__syncthreads();
		*(addr + i) = lessOne + 0.5 * ( 1.0 + (float)equalOne );
		if (i+offset<L)
		{
			*(addr + i + offset) = lessTwo + 0.5 * ( 1.0 + (float)equalTwo );
		}
		
	}

}

void normalization(real__t* devBOLD, int i, int L) 
{
	thrust::plus<float> binary_op;
	float init = 0;
	real__t mean = ( thrust::reduce(thrust::device_pointer_cast(devBOLD + i * L), thrust::device_pointer_cast(devBOLD + i * L + L)) )/ L; //caution:may be integers!
	real__t norm = sqrt( thrust::transform_reduce(thrust::device_pointer_cast(devBOLD + i * L), thrust::device_pointer_cast(devBOLD + i * L + L), square<real__t>(mean), init, binary_op) );
	thrust::transform(thrust::device_pointer_cast(devBOLD + i * L), thrust::device_pointer_cast(devBOLD + i * L + L), thrust::device_pointer_cast(devBOLD + i * L), normalize_functor<real__t>(mean,norm));

}

//note: this func may exists error cause matlab check accuracy is only 1e-4 at the most.
void SpearmanAssignmentAndNormalization(real__t* BOLD_t, int N, int L)
{
	
	/**********************   allocate and transfer data   *************************/	
	real__t * devTimeSeriesRank,*devBOLD;
	checkCudaErrors (hipMalloc ((void**)&devBOLD, sizeof(real__t) * L * N)) ; //N0 to N, may cause trouble.
	checkCudaErrors (hipMalloc ( (void**)&devTimeSeriesRank, sizeof(real__t) * L));
	checkCudaErrors (hipMemcpy (devBOLD, BOLD_t, sizeof(real__t)* L * N, hipMemcpyHostToDevice) );
	thrust::device_vector<int> address(L);
   
	/**********************   sort\assignRank\normalization   **********************/	
	clock_t spearTime = clock();
	for(int i = 0; i<N; i++)
	{
		//1.sorting
		thrust::sequence(address.begin(), address.end());
		thrust::sort_by_key(thrust::device_pointer_cast(devBOLD + i * L), thrust::device_pointer_cast(devBOLD + (i+1) * L), address.begin());
		//2.assign rank
		hipMemset(devTimeSeriesRank, 0, sizeof(real__t) * L);
		assignrank<<<block_num,thread_num>>>(devBOLD + i * L, L, thrust::raw_pointer_cast(address.data()), devTimeSeriesRank);//revisit block_num later.
		checkCudaErrors(hipMemcpy(devBOLD + i * L,devTimeSeriesRank, sizeof(real__t) * L,hipMemcpyDeviceToDevice));
		// normalization
		normalization(devBOLD,i,L);
		
	}
	spearTime = clock() - spearTime;
	cout<<"extra-elapsed time for calculating Spearman coefficient of correlation (rank assignment plus normalization): "<<spearTime/1000.0<<"s."<<endl;
	//system("pause");

	thrust::device_vector<int>().swap(address);
	checkCudaErrors (hipFree(devTimeSeriesRank));
	/************************  pass back data   ******************************/	
	checkCudaErrors (hipMemcpy (BOLD_t, devBOLD, sizeof(real__t)* L * N, hipMemcpyDeviceToHost) );
	hipFree(devBOLD);
} 

real__t* SpearmanAssignmentAndNormalizationPointer(real__t* BOLD_t, int N, int L)
{
	
	real__t * devTimeSeriesRank,*devBOLD;
	checkCudaErrors (hipMalloc ((void**)&devBOLD, sizeof(real__t) * L * N)) ; //N0 to N, may cause trouble.
	checkCudaErrors (hipMalloc ( (void**)&devTimeSeriesRank, sizeof(real__t) * L));
	checkCudaErrors (hipMemcpy (devBOLD, BOLD_t, sizeof(real__t)* L * N, hipMemcpyHostToDevice) );
	thrust::device_vector<int> address(L);
   
	/**********************   sort\assignRank\normalization   **********************/	
	clock_t spearTime = clock();
	for(int i = 0; i<N; i++)
	{
		//1.sorting
		thrust::sequence(address.begin(), address.end());
		thrust::sort_by_key(thrust::device_pointer_cast(devBOLD + i * L), thrust::device_pointer_cast(devBOLD + (i+1) * L), address.begin());
		//2.assign rank
		hipMemset(devTimeSeriesRank, 0, sizeof(real__t) * L);
		assignrank<<<block_num,thread_num>>>(devBOLD + i * L, L, thrust::raw_pointer_cast(address.data()), devTimeSeriesRank);//revisit block_num later.
		checkCudaErrors(hipMemcpy(devBOLD + i * L,devTimeSeriesRank, sizeof(real__t) * L,hipMemcpyDeviceToDevice));
		// normalization
		normalization(devBOLD,i,L);
		
	}
	spearTime = clock() - spearTime;
	cout<<"extra-elapsed time for calculating Spearman coefficient of correlation (rank assignment plus normalization): "<<spearTime/1000.0<<"s."<<endl;
	
	thrust::device_vector<int>().swap(address);
	checkCudaErrors (hipFree(devTimeSeriesRank));
	/************************  pass back pointer   ******************************/	
	return devBOLD;
} 

void SpearmanAssignmentBroadcast(real__t* BOLD_t, int N, int L, bool blocking)
{
	int blocknum = 0;
    if(blocking == true)
		blocknum = 2; //maybe future you need to define sth applied in entire project.

	if (!blocking)
	{
		real__t *devBOLD;
		checkCudaErrors (hipMalloc ((void**)&devBOLD, sizeof(real__t) * L * N)) ; //N0 to N, may cause trouble.
		checkCudaErrors (hipMemcpy (devBOLD, BOLD_t, sizeof(real__t)* L * N, hipMemcpyHostToDevice) );

		/**********************   sort\assignRank\normalization   **********************/	
		cout<<"assigning rank..."<<endl;
		hipEvent_t start, stop;
		float elapsedTime;

		hipEventCreate(&start);
		hipEventRecord(start,0);

		//assignrankBroadcast<<<block_num,thread_num>>>(devBOLD , L, N );
#ifdef figure
		int thread_num = L;
		int block_num = 180;
#endif
		assignrankBroadcastSharedMemory<<<block_num,thread_num, L * sizeof(real__t)>>>(devBOLD, L, N);
	
		hipEventCreate(&stop);
		hipEventRecord(stop,0);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&elapsedTime, start,stop);
		printf("assignment time: %f ms\n" ,elapsedTime);

		/************************  pass back pointer   ******************************/	
		checkCudaErrors ( hipMemcpy (BOLD_t, devBOLD, sizeof(real__t)* L * N, hipMemcpyDeviceToHost) );
		checkCudaErrors ( hipFree(devBOLD) );
	}else
	{
		//grnerally speaking, the smaller blocknum is, the better. 
		int generalDataWidth = N / blocknum , endDataWidth = generalDataWidth;
		real__t *devBOLD;
		if (N % blocknum != 0)
		{
			endDataWidth = N - ( blocknum - 1 ) *  generalDataWidth; //joint short tail
			checkCudaErrors (hipMalloc ((void**)&devBOLD, sizeof(real__t) * L * endDataWidth)) ; 
		}else
		{
			checkCudaErrors (hipMalloc ((void**)&devBOLD, sizeof(real__t) * L * generalDataWidth)) ; 
		}
		
		/**********************   blocked rank assignment   **************************/
		cout<<"assigning rank (blocked transmission)..."<<endl;
		hipEvent_t start, stop;
		float elapsedTime;

		hipEventCreate(&start);
		hipEventRecord(start,0);
		for (int blockid = 0; blockid < blocknum; blockid++)
		{
			int dataWidth = ( blockid == ( blocknum - 1 ) ) ?  endDataWidth : generalDataWidth;
			real__t* hostAddress = BOLD_t + blockid * dataWidth * L;
			checkCudaErrors (hipMemcpy (devBOLD, hostAddress, sizeof(real__t)* L * dataWidth, hipMemcpyHostToDevice) );
			
#ifdef figure
			int thread_num = L;
			int block_num = 180;
#endif
			assignrankBroadcastSharedMemory<<<block_num,thread_num, L * sizeof(real__t)>>>(devBOLD, L, dataWidth );
			//assignrankBroadcast<<<block_num,thread_num>>>(devBOLD , L, dataWidth );

			checkCudaErrors ( hipMemcpy ( hostAddress, devBOLD, sizeof(real__t)* L * dataWidth, hipMemcpyDeviceToHost) );

		}
		hipEventCreate(&stop);
		hipEventRecord(stop,0);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&elapsedTime, start,stop);
		printf("assignment time: %f ms\n" ,elapsedTime);

		checkCudaErrors ( hipFree(devBOLD) );

	}
	

	
} 

