#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "memory.h"
#include <iostream>
#include <ctime>
# include <fstream>
#include <vector>
#include <Windows.h>
#include<iomanip>
#include"histogram.h"
#include <thrust/binary_search.h>

#include "help_func.cuh"
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <cmath>

using namespace std;

#define ep  1e-6  //third question
//#define width  0.1//0.0001//best to be the multiples
#define width  0.000001//0.0001//best to be the multiples

#pragma comment(lib,"cublas.lib")
typedef float real__t;
typedef unsigned int uint__t;

const int thread_num = 1024; 
const int block_num = 30; 
extern bool myfunction (real__t i,real__t j);// { return (i>j); }
#define TOM(byteValue) (byteValue/1024/1024)

extern __global__ void standardKernel(real__t* devCormat, int Batch_size, bool diagnoal);


void updateOperationii(real__t** operationii, real__t* devBOLDii, int L, int Batch_size, int ii)
{
	*operationii = ii%2 ? devBOLDii + L * Batch_size : devBOLDii;
}
void updateDevBOLDii(int ii, real__t* devBOLDii, real__t* BOLD_t, int L, int Batch_size, const int Num_Blocks)
{
	if ((ii + 1)<Num_Blocks)
	{
		real__t* devIiAddr = ii%2 ? devBOLDii : devBOLDii + L * Batch_size; 
		real__t* hosIiAddr = BOLD_t + (ii + 1) * Batch_size * L;
		checkCudaErrors( hipMemcpy(devIiAddr, hosIiAddr, sizeof(real__t) * L * Batch_size, hipMemcpyHostToDevice) );
	}
}
void updatedevBOLDjj(int jj, real__t* devBOLDjj, real__t* BOLD_t, int L, int Batch_size, int* hosPoint, int* count, real__t* operationjj, const int Num_Blocks)
{
	if (*count + 1 < Num_Blocks)
	{
		real__t* devJiAddr = operationjj;
		if (*hosPoint == Num_Blocks - 1)
		{
			(*count)++;
			(*hosPoint) =  (*count);	
		}
		else
			(*hosPoint) ++;
		real__t* hosJiAddr = BOLD_t + (*hosPoint) * Batch_size * L;
		checkCudaErrors( hipMemcpy(devJiAddr, hosJiAddr, sizeof(real__t) * L * Batch_size, hipMemcpyHostToDevice) );
	}
			
}
void updateOperationjj(real__t** operationjj, real__t* devBOLDjj, int L, int Batch_size, const int numjj)
{
	*operationjj =  *operationjj ==  ( devBOLDjj + L * Batch_size * (numjj-1) )  ? devBOLDjj : *operationjj + L * Batch_size; 
	
}


#ifdef myDebug
void gpuOutput( real__t* gpuAddr, unsigned int byteNo, string OutCor, bool nameFlag)
{
	ofstream fout;
	real__t* cpuAddr = new real__t[byteNo/sizeof(real__t)];
	checkCudaErrors ( hipMemcpy(cpuAddr, gpuAddr, byteNo, hipMemcpyDeviceToHost) );
	string filename;
	if(!nameFlag)
		filename = OutCor.append("compareW_wrong.matrix");
	else
		filename = OutCor.append("compareW_right.matrix");
	fout.open(filename.c_str(), ios::binary | ios::out);
	if (!fout)
	{
		cout<<"create outfile(gpu) unsuccessfully. error code:  "<<"fighting!"<<endl;		
		system("pause");
	}	
	fout.write((const char*)cpuAddr,byteNo);
	fout.close();
	delete[] cpuAddr;
}

#endif


/* 
function CorMat_spa2rth
Calculate the correlation threshold for 
each sparsity threshold by a histogram method.
*/
real__t CorMat_spa2rth_blocking(string OutCor, real__t * BOLD_t, int N,  int L, int  Batch_size,real__t *s_thresh, real__t* result, int num_spa, const int Transferblocknum)
{
	//cout<<"*s_thresh: "<<*s_thresh<<endl;
	//real__t * BOLD_t1, * BOLD_t2;// * tempout;
	const int Num_Blocks = (N + Batch_size - 1) / Batch_size;
	decltype(N) N0 = Num_Blocks * Batch_size;
	
	//amount += amount%2;
	//modify1 zeroAmount to an array
	//uint__t num_spa = sizeof(s_thresh) / sizeof(real__t);
	long long *zeroAmount = new long long [num_spa];
	for (uint__t i = 0; i < num_spa; i++)
	{
		long long amount = N * s_thresh[i] * (N-1)  / 100.0 ;///2.0 ;
		//amount += amount%2;
	
		zeroAmount[i] = N;
		zeroAmount[i] *= N ;//
		zeroAmount[i] -= amount;
		//cout<<zeroAmount[i]<<endl;
	/*	zeroAmount[i] = (long long)N * s_thresh[i] * (N-1)  / 100.0 ;
		zeroAmount[i] -= ((long long)N * N);
		zeroAmount[i] *= -1;*/
	}
		
	
	long long subtractor = (long long)N0 * N0 - (long long)N * N;
	//cout<<N0 * N0<<endl;
	//cout<<N * N<<endl;
	//cout<<subtractor<<endl;
	//uint__t invaccount = N - account;
	uint__t Num_Bins = 1.0 / width + 1; //take care!
	uint__t position = 0;

	// transposing the BOLD signal
	
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	real__t * devBOLD, * devCormat;// * devCormatLower, * devCormatPacked;
	real__t * devBOLDii, * devBOLDjj; //only refer to head address; actually is a const.
	real__t * operationii, * operationjj;
	
	checkCudaErrors (hipMalloc ((void**)&devBOLD, sizeof(real__t) * L * Batch_size * Transferblocknum)) ;
	checkCudaErrors (hipMalloc ( (void**)&devCormat, sizeof(real__t) * Batch_size * Batch_size));	
	devBOLDii = devBOLD;
	devBOLDjj = devBOLD + L * Batch_size * 2;
	operationii = devBOLDii; 
	operationjj = devBOLDjj; 
	const int numjj = Transferblocknum - 2;
	int count = 0; //used to control devboldjj
	int hosPoint = numjj - 1;

	checkCudaErrors( hipMemcpy(devBOLDii, BOLD_t, sizeof(real__t) * L * Batch_size, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(devBOLDjj, BOLD_t, sizeof(real__t) * L * Batch_size * numjj, hipMemcpyHostToDevice) );
	
	stat = hipblasCreate(&handle) ;
	if (stat != HIPBLAS_STATUS_SUCCESS)
		return stat;
	cout<<"generating histogram (blocked transmission)..."<<endl;
	cout<<"block number per row: "<<Num_Blocks<<endl;
	const float alpha = 1.0;
	const float beta = 0;	
	thrust::device_vector<long long> histogram(Num_Bins,0);
	clock_t time;
	time = clock();
	//uint__t blocknum = ( Batch_size * Batch_size + thread_num -1 ) / thread_num;
	for (int kk = 0, ii = 0; ii < Num_Blocks; ii++)
	{
		updateOperationii(&operationii, devBOLDii, L, Batch_size, ii);
		updateDevBOLDii(ii,devBOLDii, BOLD_t, L, Batch_size, Num_Blocks);
		
		cout<<"loop flag "<<"ii: "<<ii<<endl;
		for (int jj = ii; jj < Num_Blocks; jj++)
		{
			//time = clock();
			//checkCublasErrors( hipblasSgemm(handle, HIPBLAS_OP_T,  HIPBLAS_OP_N, Batch_size, Batch_size, L,  &alpha, devBOLD + ii * Batch_size * L, L, devBOLD + jj * Batch_size * L, L, &beta, devCormat, Batch_size) );
			checkCublasErrors( hipblasSgemm(handle, HIPBLAS_OP_T,  HIPBLAS_OP_N, Batch_size, Batch_size, L,  &alpha, operationii, L, operationjj, L, &beta, devCormat, Batch_size) );
			updatedevBOLDjj(jj,devBOLDjj, BOLD_t, L, Batch_size, &hosPoint, &count, operationjj, Num_Blocks);
			updateOperationjj(&operationjj, devBOLDjj, L, Batch_size, numjj);
			//1clear NaN and negative number
			standardKernel<<<block_num,thread_num>>>(devCormat, Batch_size,ii==jj);
			//2.statistics;the results of non-diagnoal batch shall multiply by 2 accordingly
			//thrust::device_vector<uint__t> temphisto(Num_Bins,0); //dontforgettofree!
			thrust::device_ptr<real__t> dev_ptr(devCormat);
			
			/*if (ii==jj)
				dense_histogram(dev_ptr,Batch_size * Batch_size, width,temphisto, histogram);
			else
				dense_histogram2(dev_ptr,Batch_size * Batch_size, width,temphisto, histogram);*/

#ifdef myDebug
		gpuOutput( devCormat, sizeof(real__t) * Batch_size * Batch_size, OutCor, false);
#endif
			//it seems that new approach reach higher speed.
			if (ii==jj)
				dense_histogram_new(dev_ptr,Batch_size * Batch_size, width, Num_Bins, histogram);
			else
				dense_histogram2_new(dev_ptr,Batch_size * Batch_size, width, Num_Bins, histogram);
			//thrust::device_vector<uint__t>().swap(temphisto);
			//thrust::raw_pointer_cast(dev_ptr);//have a try
			//time = clock() - time;
			//cout<<"thrust::histogram time: "<<time<<"ms"<<endl;
		}
		//cout<<"Fulfill the "<<ii+1<<"th block."<<endl;
	}

	substract(histogram, subtractor);
	
	for (size_t i = 0; i < num_spa; i++)
	{
		//cout<<zeroAmount[i]<<endl;
		position = thrust::upper_bound(histogram.begin(),histogram.end(),zeroAmount[i]) - histogram.begin();//flag~ N*N
		//cout<<position<<endl;
		result[i] = width * position + width/2.0;
#ifdef myDebug
		cout<<"threshold:"<<result[i]<<endl;
#endif
	}

		//another interface is needed to return the subscript
		//display and put out
	time = clock() - time;
	cout<<"histogram time: "<<time<<"ms"<<endl;
	//checkCudaErrors (hipFree(thrust::raw_pointer_cast(histogram.data())));
	thrust::device_vector<long long>().swap(histogram);
	checkCudaErrors ( hipFree (devBOLD));
	checkCudaErrors ( hipFree (devCormat));

	delete[] zeroAmount;
	stat = hipblasDestroy(handle);
	if (stat != HIPBLAS_STATUS_SUCCESS)
		return stat;
		
}
