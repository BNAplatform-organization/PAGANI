#include "hip/hip_runtime.h"
﻿#include <stdlib.h>
#include <stdio.h>
#include <iomanip>
#include <memory.h>
#include <fstream>
#include <cstring>
#include "dirent.h" 
#include "hip/device_functions.h"
#include "modularity_GPU.cuh"
#include <cmath>
#include <time.h> 
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/fill.h>
#include <thrust/extrema.h>
#include <hip/hip_math_constants.h>
//#include <hipblas.h>
//#include <hipsparse.h>
using namespace std;
//
//void Maslov(int * R_dst, int * C_dst, int * R_src, int * C_src, int Rlength, int Clength);
//
//#define RANDOM_V0
//
//extern long long N, Ntemp;
//extern double * v, *vv; 
//extern double * v0, * verr; 
//extern	double * sumBG;
//extern long long seed;
//
typedef  unsigned int u_int;
// set some parameters

extern const int MAX_ITER;			// The maximum iteration times in the power method
extern const double BETA_Adjust;		// An optional parameter for quicker convergence. Its effect is uncertain
extern const double Epsilon;	// If |x - x0| < Epsilon, quit iteraion 
extern const double LAMBDA;		// if labmda > LAMBDA, initiate the division
extern const double DQ_MIN;
extern const int MIN_GROUP;			// The minimum nodes of an allowed module 

extern ofstream fout;	

const int threadnumx = 16;
const int threadnumy = 16;
const int  threadnum = 256;
const int blocknum    = 96;
//int* d_AD_init;
//int* d_AD;
//int* d_r; 
//int* d_c;
//int* d_orir;
//double* d_u;
//double* d_u0;
//double* d_uu;
//double * d_sumBG;
//double * d_norm;
////bool* d_G;
//double * temp_result;
//double * d_vector;
//double * d_vector1;
//double * d_vector2;
//double *d_k;
//double *d_orik;
//
//void Partition(int * R, int * C, int * Result);
//bool Sub_Partition(int * OriR, int * OriC, int * R, int * C, int M, long long innerM, int * Result, int * Max_Result,int * AD);
//double Lead_Vector(int * OriR, int * R, int * C, int M, double * sumBG1, double beta, int *AD, double *v, double *vv);
//

hipsparseHandle_t s_handle=0;
hipsparseMatDescr_t s_descr=0;
hipblasHandle_t handle;

double Lead_Vector(int N, int * R, int * C, float *V, double *K,  double * sumBG, double m, double beta, double *u);
double calculate_dQ(signed char *S, int N, int * R, int * C, float *V, double *K, double *sumBG, double m);
double fine_tune_S(double dQ, int N, int * R, int * C, float * V, double * K, double m, signed char *S);
double qmax(int N, double *Qit, char *indSub, int *imax);
template <class Type> double VectorNorm(Type * x, int N);

__global__ void init_du(int N, double *d_u)
{
	int tid = blockDim.x*blockIdx.x+threadIdx.x;
	int i;
	for (i = tid; i < N; i+=blockDim.x*gridDim.x)
		d_u[i] = (1.0 * i) / (N-1);
}

__global__ void sign_eigv(int N, double *d_eigv)
{
	int tid = blockDim.x*blockIdx.x+threadIdx.x;
	int i;
	double temp = 0;
	for (i = tid; i<N; i+=blockDim.x*gridDim.x)
	{	
		temp = ((d_eigv[i] >= 0) ? 1.0 : -1.0);
		syncthreads();
		d_eigv[i] = temp;
	}
}

//
//__global__ void cal_k_kernel(long N, int *AD,int *d_OriR, double *d_K )
//{		
//	int offset;
//	const int blockid   = blockIdx.x;
//	const int threadid = threadIdx.x;
//	
//	for(offset=(blockid/2)*threadnum*2+threadid*2+blockid%2; offset<N; offset+=blockDim.x*gridDim.x )
//		d_K[offset]= (double)(d_OriR[AD[offset]+1]-d_OriR[AD[offset]]);
//}
//
//__global__ void sum_kj_kernel(long N, double *d_K ,double *sum_kj)
//{	__shared__ int sum[threadnum];
//	int temp=0;
//	int offset;
//   const int threadid =blockIdx.x*blockDim.x + threadIdx.x;
//	
//	//for(offset=(blockid/2)*threadnum*2+threadid*2+blockid%2; offset<N; offset+=blockDim.x*gridDim.x )
//		// temp+= dG[offset]*(d_R[offset+1]-d_R[offset]);
//	//for (offset=blockid*threadnum; offset+threadid<N; offset+=blockDim.x*gridDim.x ) 
//	 	//  temp +=dG[offset+threadid]*(d_R[offset + threadid+1]-d_R[offset + threadid]);	 		
//	
//	for(offset=threadid; offset<N; offset+=blockDim.x*gridDim.x)
//	{
//		temp+=(int)d_K[offset] ;
//	}
//	sum[threadIdx.x]=temp;
//	syncthreads();
//	
//	for(offset=1;offset+threadIdx.x<threadnum;offset*=2){
//			if (threadIdx.x%(2*offset)==0)  sum[threadIdx.x]+=sum[threadIdx.x+offset] ;
//			syncthreads();
//	}
//	if(threadIdx.x==0)
//	  sum_kj[blockIdx.x]=(double) sum[threadIdx.x];
//}
//
//__global__ void sum_kernel(int size, double *data, double scale)		   //将一些前面操作中用树形结构相加每个block的结果求和。
//{	
//	__shared__ double sum[blocknum];
//	int offset;
//	sum[threadIdx.x]=data[threadIdx.x];
//	syncthreads();
//	
//	for(offset=1;offset+threadIdx.x<blocknum; offset*=2){
//		if (threadIdx.x%(2*offset)==0)  sum[threadIdx.x]+=sum[threadIdx.x+offset] ;
//		syncthreads();
//	}
//	if(threadIdx.x==0)
//	    data[0]=sum[0]*scale; 
//}
//
//__global__ void spmv_one_thread(long N, long M, double *result, int *R, int *C, double *vv, double *dk, double vk, double *d_sum, double beta, double *v0)	   //计算Ai*v0,   每16个threads计算一行
//{
//	__shared__ int R_shared[threadnum+1];
//
//	double temp1=0;
//	int offset;  
//	
//	for (offset=blockIdx.x*threadnum+threadIdx.x;offset<N; offset+=gridDim.x*blockDim.x)
//	{
//		R_shared[threadIdx.x+1] = R[offset+1];
//		if(threadIdx.x==0) R_shared[threadIdx.x]=R[offset] ;
//		syncthreads();	 
//		temp1 =0 ;
//		for(int i=R_shared[threadIdx.x]; i<R_shared[threadIdx.x+1];i++)
//		{		
//			temp1+=vv[C[i]];			
//		}		
//		temp1-=vk/(2*M)*dk[offset]+(d_sum[offset]-beta)*v0[offset];
//		result[offset]=temp1 ;
//			//syncthreads();
//	}		
//}
//
__global__ void vvdot_subtr (long N, double *result_vector, double *vx1, double *vx2, double *vy)    //计算向量加法
 {
	 const int blockid   = blockIdx.x;
	 const int threadid  = threadIdx.x;
	 int offset;  	 
	 for(offset=threadid+blockid*blockDim.x; offset < N; offset+=blockDim.x*gridDim.x)
		 result_vector[offset] = vx1[offset] * vx2[offset] - vy[offset];
 }

__global__ void calculate_du (int N, double alpha, double beta, double *d_u, double *d_K, double *d_sumBG, double *d_u0)
{
	int tid = blockDim.x*blockIdx.x+threadIdx.x;
	int i;
	double temp;
	// d_u = d_u - alpha*K - (sumBG+beta).*u0
	for (i = tid; i<N; i+=blockDim.x*gridDim.x)
	{
		temp = d_u[i];
		syncthreads();
		temp -= alpha * d_K[i];
		syncthreads();
		temp -= (d_sumBG[i]+beta)*d_u0[i];		
		syncthreads();
		
		d_u[i] = temp ;
	}
}

__global__ void calculate_Qit (int N, double Q, double *d_Qit, double *d_temp_vector, double *d_K, double *d_S, double m, char *d_indSub)
{
	int tid = blockDim.x*blockIdx.x+threadIdx.x;
	double temp = 0, ki = 0, si = 0;
	char f = 0;
	// 
	for (int i = tid; i<N; i+=blockDim.x*gridDim.x)
	{
		f = d_indSub[i];
		if (f)
		{
			temp = d_temp_vector[i];
			//syncthreads();
			ki = d_K[i];
			//syncthreads();
			si = d_S[i];
			temp = temp*si +  ki*ki/m;
			//syncthreads();
			temp = Q - 4 * temp ;
			d_Qit[i] = (temp >= 0 ? temp : 0);
			//d_Qit[i] = temp;
		}
		else 
			d_Qit[i] = 0;

	}
}

__global__ void update_array(long imax, double *d_Sit)
{
	d_Sit[imax] = d_Sit[imax];
	//indSub[imax] = HIP_NAN_F; //at risk
}
// 
// __global__ void sumBG_kernel (double *sumBG, double *d_orik, double *d_k, int *AD,long N, double innerMd2M)    //计算向量加法
// {
//	 //sumBG[i] = R[i+1] - R[i] - (OriR[AD[i]+1] - OriR[AD[i]]) * (double)innerM / 2 / M;
//	 const int blockid   = blockIdx.x;
//	 const int threadid  = threadIdx.x;
//	 int offset;  	 
//	 for(offset=threadid+blockid*threadnum; offset<N; offset+=threadnum*gridDim.x)
//	 {
//		sumBG[offset] = d_k[offset] - (d_orik[AD[offset]]) * innerMd2M;
//	 }
// }
// 
// __global__ void cal_VV_kernel ( long Ntemp, long N, int *AD, double *vv, double *v)
// {
//	 const int blockid   = blockIdx.x;
//	 const int threadid  = threadIdx.x;
//	 int offset;
//	 for(offset=threadid+blockid*threadnum; offset<Ntemp; offset+=threadnum*gridDim.x)
//		 vv[AD[offset]] = v[offset];
// }
//
// __global__ void calc_vector (long N, double *result, double *dk, double vk2m, double *d_sum, double beta, double *v0)	   //计算 ( v_k/2/M * (R[i+1] - R[i])+(sumBG1[i] - beta) * v0[i])，
// {		
//	 const int blockid   = blockIdx.x;
//	 const int threadid = threadIdx.x;
//	 int offset;
//	 double temp=0;
//	 
//	 for(offset=threadnum*blockid+threadid; offset<N; offset+=threadnum*gridDim.x)
//	 {	
//		  temp=vk2m*dk[offset]+(d_sum[offset]-beta)*v0[offset];
//		  result[offset]=temp;
//	 }
//  }
//
// /*__global__ void  Norm2_ph1(long N, double *norm,  double *v, bool *dG)	 //求向量的二范数，配合sum_kernel 得到最终结果
// {
//	 __shared__  double temp[threadnum];
//	 const int blockid   = blockIdx.x;
//	 const int threadid = threadIdx.x;
//	 double temp1=0;
//	 int offset;
//
//	 for(offset=blockid*threadnum+threadid; offset<N; offset+=threadnum*gridDim.x)
//		 temp1+=dG[offset]? v[offset]*v[offset] : 0;
//	 temp[threadid]=temp1;
//	 syncthreads();
//
//	 for(offset=1;offset+threadid<threadnum;offset*=2){
//		 if (threadid%(2*offset)==0)  temp[threadid]+=temp[threadid+offset] ;
//		 syncthreads();
//	 }
//	 if (threadid==0)
//		 norm[blockid] = temp[threadid];  	 
// }
// __global__ void Norm2_ph2(long N,  double norm, double *v)						  //将向量归一化，若为零向量，则每个元素除以1，保持不变。
// {	 	 
//	 for (int offset = threadIdx.x+blockIdx.x*threadnum; offset<N ; offset+= threadnum*gridDim.x  )
//		 v[offset]/=(norm? (norm) : 1);  
// } */
//
//
//
//
//
//
//
///* 
//This function returns the norm of the input vector x[G].
//G is the logic subscriber and N is the matrix dimension.
//*/
//
//
void cudaDevice_check()
{
	int devID;
	hipDeviceProp_t deviceProps;
	devID = findCudaDevice();
	// get number of SMs on this GPU
	checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
}

double Lead_Vector_GPU(int N, int nnz, int *d_R, int *d_C, double *d_V, double *d_K, double *d_sumBG, double m, double beta, double *d_u)
{
	int i = 0, j = 0;
	
	// Initialize d_u. Two methods are optional. Define RANDOM_V0 if you want to use random starting vector
	init_du<<<blocknum,threadnum>>>(N, d_u);
//#ifdef RANDOM_V0
//	srand(time(0));
//	//srand(2016);
//	u[0] = 1;                        //normalized;
//	for (i = 1; i < N; i++)
//		u[i] = rand()*1.0f/RAND_MAX;
//	
//#else
//	for (i = 0; i < N; i++)
//		u[i] = 1.0f*i/(N-1);
//#endif
	
	double *d_u0;
	checkCudaErrors( hipMalloc( (void**) &d_u0, sizeof(double) * (N)));
	//double err1 = 1, err2 = 1;
	double err = 1;
	u_int ITER = 0;
	double uNorm = 0, uNorm0 = 0;
	double u_k;
	double temp1;
	double alpha = 1.0;
	double cublasbeta = 0;
	double *K = new double [N];
	double *sumBG = new double [N];
	checkCudaErrors( hipMemcpy(K, d_K, sizeof(double)*N, hipMemcpyDeviceToHost));
	checkCudaErrors( hipMemcpy(sumBG, d_sumBG, sizeof(double)*N, hipMemcpyDeviceToHost));
	
	//double *u = new double [N];
	//double *u0 = new double [N];

	while (err > Epsilon && ITER < MAX_ITER)
	{
		//for (i = 0; i < N; i++)
		//	v0[i] =  v[i];
		//checkCudaErrors( hipMemcpy(d_u, u,sizeof(double)*N, hipMemcpyHostToDevice));
		checkCudaErrors( hipMemcpy(d_u0,d_u,sizeof(double)*N, hipMemcpyDeviceToDevice));
		
		// The dot product of u_k = k'*u, 
		u_k = 0;
		checkCublasErrors( hipblasDdot (handle, N, d_u0, 1, d_K, 1, &u_k), "Ddot err in  u*k"); 
		//for (i = 0; i < N; i++)
		//	u_k += u0[i] * (K[i]);
		//cout<<"u_k = "<<u_k<<endl;
		
		// Do the matrix-vector multiplication
		alpha = 1.0;
		checkCusparseErrors( hipsparseDcsrmv(s_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nnz, &alpha, s_descr, d_V, d_R, d_C, d_u0, &cublasbeta, d_u),
		"Dcsrmv err in calculate lead_vector");
		
		//checkCublasErrors (hipblasDaxpy(handle, N, &alpha, d_K, 1, d_u, 1), "Daxpy err in calculate lead_vector");
		
		
		//checkCudaErrors( hipMemcpy(u,d_u,sizeof(double)*N, hipMemcpyDeviceToHost));
		alpha = u_k/m;
		calculate_du<<<blocknum,threadnum>>>(N, alpha, beta, d_u, d_K, d_sumBG, d_u0);
		
		
		/*for (i = 0; i < N; i++)
		{
			temp1 = 0;
			for (j = R[i]; j < R[i+1]; j++)
				temp1 += V[j] * u0[C[j]];
			temp1 -= u_k / m * (K[i]) + (sumBG[i] + beta) * u0[i];
			u[i] = temp1;
		}*/
		
		//ofstream debugfile;
		//debugfile.open("debug_u",ios::binary|ios::out);
		//debugfile.write((char *)u, N*sizeof(double));
		//debugfile.close();
		
		int idx;
		//double *x = new double [N];
		//checkCudaErrors( hipMemcpy(d_u, u, sizeof(double)*N, hipMemcpyHostToDevice));
		checkCublasErrors(hipblasIdamax(handle, N, d_u, 1, &idx),"amax err in calculating lead_vector!\n");
		checkCudaErrors( hipMemcpy(&uNorm,d_u+idx-1,sizeof(double), hipMemcpyDeviceToHost));
		//uNorm = VectorNorm<double>(x, N);
		
		// Decide whether converge, using infinity norm
		err = fabs(uNorm-uNorm0);
		//cout<<uNorm<<" - "<<uNorm0<<" residual: "<<err<<endl;
		uNorm0 = uNorm;
		alpha = 1.0/uNorm;
		checkCublasErrors( hipblasDscal(handle, N, &alpha, d_u, 1),"Dscal err!\n");
		//for (i = 0; i < N; i++)
		//	u[i] = u[i] / uNorm ;
				
		ITER++;
	}
	//delete []u0;
	//delete []u;
	cout<<"Iterations:\t"<<ITER<<'\t'<<"residual:\t"<<err<<'\t';
	fout<<"Iterations:\t"<<ITER<<'\t'<<"residual:\t"<<err<<'\t';
	hipFree(d_u0);
	//delete []u0;
	//u0 = NULL;
	// return the eigenvalue
	//u_int max_index = 0;
	//u[0] = G[0] ? v[0] : 0;
	//for (i = 0; i < N; i++)
	//	if (fabs(u[i]) > fabs(u[max_index]))
	//		max_index = i;
	//cout<<uNorm<<endl;
	return (uNorm);
}




//	long long i = 0, j = 0;
//	/*double *k = new double [Ntemp];
//	for (int p=0; p<Ntemp; p++)
//	{
//		k[p] = p;
//	}*/
//	// Initialize v. Two methods are optional. Define RANDOM_V0 if you want to use random starting vector
//#ifdef RANDOM_V0
//	//srand(time(0));
//	srand(seed);
//	for (i = 0; i < Ntemp; i++){
//		v[i] = AD[i];
//		//if(i<20) cout<<v[i]<<endl;
//	}	 
//	for (i = 0; i < N; i++){
//		vv[i] = 0;
//		//if(i<20) cout<<v[i]<<endl;
//	}
//#else
//	for (i = 0; i < N && !G[i]; i++)
//#endif
//	hipError_t cudaStat;
//	hipblasStatus_t stat;
//	hipblasHandle_t handle;
//	stat = hipblasCreate(&handle) ;
//	checkCudaErrors( hipMemset (d_u, 0, sizeof(double) * (Ntemp)));
//	checkCudaErrors( hipMemcpy( d_u, v, sizeof(double) * Ntemp , hipMemcpyHostToDevice) );
//	checkCudaErrors( hipMemcpy( d_uu, vv, sizeof(double) * N , hipMemcpyHostToDevice) );
//	if (stat != HIPBLAS_STATUS_SUCCESS)
//		return stat;
//
//	double err1 = 1, err2 = 1;
//	int ITER = 0;
//	double vNorm = 0;
//	double temp2= -1;
//	double temp1=0;
//	//double *norm_v= new double ;
//	//double *check_du= new double [N];
//    double v_k;
//	
//	//int blocknum_spmv = N*HALF_WARP/threadnum+1;
//
//	//spmv_kernel<<<blocknum_spmv, threadnum>>>((long) N, d_r, d_c, d_G, d_u, d_u0);
//	dim3 blocknum_spmv ( Ntemp/threadnumy+(Ntemp%threadnumy?1:0) );
//	dim3 threadn(threadnumx,threadnumy);
//	//cal_k_kernel<<<6,threadnum>>>((long) Ntemp, d_AD, d_orir, d_k);
//
//	while (err1 > Epsilon &&  err2 > Epsilon && ITER < MAX_ITER)
//	{	  		
//		
//	   hipblasDcopy(handle, (int) Ntemp, d_u, 1 ,d_u0, 1 );
//	   //这里需要计算vv！
//	   cal_VV_kernel<<<blocknum,threadnum>>>((long) Ntemp, (long) N, d_AD, d_uu, d_u);
//	   
//	   hipblasDdot (handle, Ntemp, d_u0, 1, d_k, 1, &v_k);
//	   //calc_vector<<<blocknum, threadnum>>>((long) Ntemp, d_vector, d_k, v_k/(2*M), d_sumBG, beta, d_u0);
//	  /* checkCudaErrors( hipMemcpy( vvector, d_vector, sizeof(double) * (Ntemp), hipMemcpyDeviceToHost) );
//	   for (int ii = 0;ii < Ntemp;ii++)
//	   {
//		   cout<<"vector["<<ii<<"] = "<<vvector[ii]<<endl;
//	   }*/
//  		spmv_one_thread<<<blocknum , threadnum>>>((long)Ntemp, (long) M, d_u, d_r, d_c, d_uu, d_k, v_k, d_sumBG, beta, d_u0) ;
//		/*checkCudaErrors( hipMemcpy( v, d_u, sizeof(double) * Ntemp , hipMemcpyDeviceToHost) ); 
//	  for (int ii = 0;ii < Ntemp;ii++)
//	   {
//		   cout<<"v["<<ii<<"] = "<<v[ii]<<endl;
//	   }*/
//		//hipblasDaxpy(handle, (int) Ntemp, &temp2, d_vector, 1, d_u, 1);
//		//checkCudaErrors( hipMemcpy( v, d_u, sizeof(double) * (Ntemp), hipMemcpyDeviceToHost) );
//	   /*for(int ii=0;ii<Ntemp;ii++)
//	   {
//			if (v[ii]!=0)
//				cout<<"v["<<ii<<"] = "<<v[ii]<<endl;
//	   }*/
//	   
//	    hipblasDnrm2(handle, Ntemp, d_u, 1, &vNorm);
//		temp1=1/vNorm;
//		hipblasDscal (handle, (int) Ntemp, &temp1, d_u, 1);   //Normalize v, v[i] = v[i]/vNorm
//	
//		vvplus<<<blocknum, threadnum>>>((long) Ntemp, d_vector, d_u, 1.0, d_u0, -1.0);
//	    hipblasDnrm2(handle, Ntemp, d_vector, 1, &err1);
//		vvplus<<<blocknum, threadnum>>>((long) Ntemp, d_vector, d_u, 1.0, d_u0, 1.0);
//		hipblasDnrm2(handle, Ntemp, d_vector, 1, &err2);
//				 
//		ITER++;
//	}	 
//	//system("pause");
//	cout<<"Iterations:\t"<<ITER<<'\t'<<"residual:\t"<<min(err1, err2)<<'\t';
//	fout<<"Iterations:\t"<<ITER<<'\t'<<"residual:\t"<<min(err1, err2)<<'\t';
//	
//	checkCudaErrors( hipMemcpy( v, d_u, sizeof(double) * Ntemp , hipMemcpyDeviceToHost) ); 
//	checkCudaErrors( hipMemcpy(v0,d_u0, sizeof(double) * Ntemp,  hipMemcpyDeviceToHost) );
//	hipblasDestroy(handle);
//	long long max_index = 0;
//	for (i = 0; i < Ntemp; i++)
//		if (fabs(v[i]) > fabs(v[max_index]))
//			max_index = i;
//	return (v[max_index] * v0[max_index] > 0) ? vNorm: -vNorm;
//}
//
//
//

/* 
This funtion calculates dQ for each round to decide whether to split the current submodule.
It returns a double variable dQ.
*/
double calculate_dQ_GPU(double *d_S, int N, int nnz, int *d_R, int *d_C, double *d_V, double *d_K, double *d_sumBG, double m)
{
	double dQ = 0;
	//double * x = new double [N];
	int i,j;
	double temp;
	double k_s = 0;
		
	// x = S'*Bsub*S = S'*(Bsub*S) = S'*[(Asparse-k*k'/m)*S]

	checkCublasErrors( hipblasDdot (handle, N, d_S, 1, d_K, 1, &k_s), "Ddot err in  calculate dQ function"); 
	/*for (i = 0; i < N; i++)
			k_s += S[i] * K[i];*/
	
	double alpha = 1.0, beta = 0;
	double *d_temp_vector;
	checkCudaErrors( hipMalloc( (void**) &d_temp_vector, sizeof(double) * N ));
	
	checkCusparseErrors(hipsparseDcsrmv(s_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nnz, &alpha, s_descr, d_V, d_R, d_C, d_S, &beta, d_temp_vector),
		"Dcsrmv err in calculate dQ function");
	
	
	/*double *temp_vector = new double [N];
	checkCudaErrors( hipMemcpy( temp_vector, d_temp_vector, sizeof(double) * N, hipMemcpyDeviceToHost ));
	for (i = 0; i < N; i++)
	{
		if (i<10)
			cout<<temp_vector[i]<<endl;
		dQ += temp_vector[i];
	}*/

	vvdot_subtr <<<blocknum,threadnum>>> (N, d_temp_vector, d_S, d_temp_vector, d_sumBG); //d_S.*temp_vector - d_sumBG
	
	thrust::device_ptr<double> d_tpvector(d_temp_vector);
	dQ = thrust::reduce(d_tpvector, d_tpvector+N, (double) 0, thrust::plus<double>());
	//checkCublasErrors ( hipblasDasum(handle, N, d_temp_vector, 1, &dQ), "cublas sum dQ error!"); //asum is the sum of absolute value;
	
	
	/*for (i = 0; i < N; i++)
		{
			temp = 0;
			for (j = R[i]; j < R[i+1]; j++)
				temp += V[j] * S[C[j]];			
			dQ += S[i] * temp - sumBG[i];
		}*/
	
	dQ -= k_s*k_s/m;

	hipFree(d_temp_vector);

	//delete []x;
	return (dQ);
}

double fine_tune_S_GPU(double dQ, int N, int nnz, int * d_R, int * d_C, double * d_V, double * d_K, double m, double *d_S)
{
	
	
	//signed char *Sit = new signed char [N];
	double *d_Sit;
	checkCudaErrors( hipMalloc( (void**) &d_Sit, sizeof(double) * (N)));
	checkCudaErrors( hipMemcpy( d_Sit, d_S, sizeof(double) * N, hipMemcpyDeviceToDevice)) ;
	thrust::device_ptr<double> dev_Sit(d_Sit);
	thrust::device_ptr<double> dev_S(d_S);
	//memcpy(Sit,S,sizeof(signed char)*N);
	//signed char *S_Si = new signed char [N];
	
	double *d_Qit; // = new double [N];
	checkCudaErrors( hipMalloc( (void**) &d_Qit, sizeof(double) * (N)));
	checkCudaErrors(hipMemset(d_Qit,0,sizeof(double)*N));
	thrust::device_ptr<double> dev_Qit(d_Qit);

	char *d_indSub;
	checkCudaErrors( hipMalloc( (void**) &d_indSub, sizeof(char) * (N)));
	thrust::device_ptr<char> dev_indSub(d_indSub);
	thrust::fill(dev_indSub,dev_indSub+N,TRUE);
	//fill(indSub,indSub+N,1); 

	
	double k_s = 0;
	int i = 0,j = 0;
	int imax = 0;
	int ITER = 0;
	double Q = dQ;
	double Qmax = dQ;
	double alpha = 0;
	//bool flag = TRUE;
	double *d_temp_vector;
	checkCudaErrors( hipMalloc( (void**) &d_temp_vector, sizeof(double) * N ));

	//double *Qit = new double [N];
	//double *Sit = new double [N];
	//bool copy_S_flag = false;
	while (ITER<N)
	{
		//////////////////////////////////////////////////////////
		//calculate Qit[]=dQ - 4 *(Sit').*( Bsub-diag(Bsub) )*Sit
		checkCublasErrors( hipblasDdot (handle, N, d_Sit, 1, d_K, 1, &k_s), "Ddot err in fine_tune_S function");
		alpha = 1.0;
		double beta = 0;
		checkCusparseErrors(hipsparseDcsrmv(s_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nnz, &alpha, s_descr, d_V, d_R, d_C, d_Sit, &beta, d_temp_vector),
		"Dcsrmv err in fine_tune_S function");
		alpha = -1.0*k_s/m;
		//hipblasDaxpy(hipblasHandle_t handle, int n, const double *alpha, const double *x, int incx, double *y, int incy)


		checkCublasErrors(hipblasDaxpy(handle, N, &alpha, d_K, 1, d_temp_vector, 1),"Daxpy err in fine_tune_S function");
		
		calculate_Qit<<<blocknum,threadnum>>>(N, Qmax, d_Qit, d_temp_vector, d_K, d_Sit, m, d_indSub);
		
		//checkCudaErrors( hipMemcpy( Qit, d_Qit, sizeof(double) * N, hipMemcpyDeviceToHost)) ;
		
		//for (i = 0; i < N; i++)
		//{
		//	if (indSub[i] == 0)
		//		continue;

			//for (j = 0; j < N; j++)
			//	S_Si[j] = Sit[j]*Sit[i];

			//Sit[k] = -Sit[k];
			//S_Si[i] = 0;
			//k_s = 0;
			
					

			//temp = 0;
			//for (j = R[i]; j < R[i+1]; j++)
			//	temp += S_Si[C[j]] * V[j];			
			////	Qit[k] += S[i] * temp + K[i]*K[i]/m;
			////}
			//for (j = 0; j < N; j++)
			//	temp -= S_Si[j]*K[i]*K[j]/m;
			//Qit[i] = Qmax - 4*temp;
			
			//cout<<Qit[k]<<'\t';
			////////////////////////////////////////////////////
			//Sit[k] = -Sit[k];
		//}

		//for (i = 0; i < N; i++)
		//	Qit[i] *= indSub[i];
		
		//Qmax = qmax(N, Qit, indSub, &imax);
		
		hipblasIdamax(handle, N, d_Qit, 1, &imax); 
		imax = imax-1;
		Qmax = dev_Qit[imax];
		
		/*thrust::device_ptr<double> max_ptr = thrust::max_element(dev_Qit, dev_Qit + N);
		imax = &max_ptr[0] - &dev_Qit[0];
		Qmax = max_ptr[0];*/
		
		//cout<<"Qmax = "<<Qmax<<"\t imax = "<<imax<<endl;
		
		if (dev_indSub[imax])
			dev_indSub[imax] = 0;
		else
			break;
			//cout<<"indSub break flag!"<<endl;
		
		//if (Qmax > Q)
		//{
		//	Q = Qmax;
		//	copy_S_flag = true;
		//	//continue;						
		//}
		//else if (copy_S_flag)
		//{
		//	copy_S_flag = false;
		//	checkCudaErrors( hipMemcpy(d_S, d_Sit, sizeof(double)*N, hipMemcpyDeviceToDevice));			
		//}
				
		dev_Sit[imax] = -1.0*dev_Sit[imax];
		
		if (Qmax > Q)
		{
			Q = Qmax;
			dev_S[imax] = dev_Sit[imax];
		}
		else
			break;

		ITER++;
	}
	cout<<"\nFine tune ITER: "<<ITER<<endl;
	//if (copy_S_flag)
	//	checkCudaErrors( hipMemcpy(d_S, d_Sit, sizeof(double)*N, hipMemcpyDeviceToDevice));

	hipFree(d_temp_vector);
	hipFree(d_Qit);
	hipFree(d_Sit);
	hipFree(d_indSub);
	//delete []Sit;
	//delete []Qit;
	//delete []indSub;
	cout<<"fine tune Q: "<<Q<<endl;
	return Q;
}

//bool Sub_Partition_GPU(int N, int * ind, int * R, int * C, float *V, double *K, double *sumBG, double m, int * Result, int * Num_module)
//{
//	int devID;
//	hipDeviceProp_t deviceProps;
//	devID = findCudaDevice();
//	// get number of SMs on this GPU
//	checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
//	
//	checkCublasErrors(hipblasCreate(&handle), "Create cublas handle err!\n");
//	checkCusparseErrors( hipsparseCreate(&s_handle), "Create cusparse handle err!\n" );
//	checkCusparseErrors(hipsparseCreateMatDescr(&s_descr), "CreateMatDescr err!\n"); 
//	hipsparseSetMatType(s_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
//    hipsparseSetMatIndexBase(s_descr,HIPSPARSE_INDEX_BASE_ZERO); 
//
//	int i,j;
//	int nnz = R[N]-R[0];
//	double *Vd = new double [nnz];
//	for (i = 0; i < nnz; i++)
//		Vd[i] = (double) V[i];
//	
//	int *d_R, *d_C;
//	double *d_V;
//	double *d_K, *d_sumBG;
//
//	checkCudaErrors( hipMalloc( (void**) &d_R, sizeof(int) * (N + 1)));
//	checkCudaErrors( hipMalloc( (void**) &d_C, sizeof(int) * (R[N]-R[0])));
//	checkCudaErrors( hipMalloc( (void**) &d_V, sizeof(double) * (N)));
//	checkCudaErrors( hipMalloc( (void**) &d_sumBG, sizeof(double) * (N)));
//	checkCudaErrors( hipMalloc( (void**) &d_K, sizeof(double) * (N)));
//	
//	checkCudaErrors( hipMemcpy( d_R, R, sizeof(int) * (N + 1), hipMemcpyHostToDevice)) ;
//	checkCudaErrors( hipMemcpy( d_C, C, sizeof(int) * R[N], hipMemcpyHostToDevice)) ;
//	checkCudaErrors( hipMemcpy( d_V, Vd, sizeof(double) * R[N], hipMemcpyHostToDevice)) ;
//	checkCudaErrors( hipMemcpy( d_K, K, sizeof(double) * N, hipMemcpyHostToDevice)) ;
//	checkCudaErrors( hipMemcpy( d_sumBG, sumBG, sizeof(double) * N, hipMemcpyHostToDevice)) ;
//
//	double * d_eigv;
//	checkCudaErrors( hipMalloc( (void**) &d_eigv, sizeof(double) * N));
//	
//	double lambda = 0;
//	lambda = Lead_Vector_GPU(N, nnz, d_R, d_C, d_V, d_K,  d_sumBG, m, BETA_Adjust, d_eigv);
//	//lambda = Lead_Vector(N, R, C, V, K,  sumBG, m, BETA_Adjust, eigv);
//	lambda += BETA_Adjust;
//	// If lambda < 0, calucate the leading eigenvalue for  B - lambda * I
//	if (lambda < 0)
//		lambda += Lead_Vector_GPU(N, nnz, d_R, d_C, d_V, d_K,  d_sumBG, m, lambda, d_eigv);
//		//lambda += Lead_Vector(N, R, C, V, K,  sumBG, m, lambda, eigv);
//	cout<<"Eigen Value: "<<lambda<<'\t';
//	fout<<"Eigen Value: "<<lambda<<'\t';
//
//	// Decide whether this round of partition is successful 
//	int subN = 0, subP = 0;
//	
//	/*for (i = 0; i < N; i++)
//	{
//		subP += (eigv[i] >= 0);  		
//		subN += (eigv[i] < 0);	 		
//		eigv[i] = ((eigv[i] >= 0) ? 1 : -1);
//		S[i] = ((eigv[i] >= 0) ? 1 : -1);
//	}*/
//	
//	
//	
//
//	sign_eigv<<<blocknum,threadnum>>>(N,d_eigv);
//	double S_sum = 0;
//	//stat = hipblasDasum(handle, N, d_eigv, 1, &S_sum);  //asum sum of absolute value;
//	//if (stat != HIPBLAS_STATUS_SUCCESS)
//	//	return stat;
//	
//	subP = (N + S_sum)/2;
//	subN = (N - S_sum)/2;
//	cout<<"subP: "<<subP<<endl;
//	cout<<"subN: "<<subN<<endl;
//	if (subP+subN != N)
//	{
//		cout<<"S allocation error!";
//		system("pause");
//	}
//
//	//calculate dQ;
//	double dQ = 0;
//
//	dQ = calculate_dQ_GPU(d_eigv, N, nnz, d_R, d_C, d_V, d_K, d_sumBG, m);
//	cout<< "dQ = "<<dQ<<endl;
//	if (dQ>DQ_MIN)  //fine tune results 
//	{
//		 dQ = fine_tune_S_GPU(dQ, N, nnz, d_R, d_C, d_V, d_K, m, d_eigv);	
//	}
//	subP = 0;
//	subN = 0;
//	
//	/*for (i = 0; i < N; i++)
//	{
//		subP += (S[i] >= 0);  		
//		subN += (S[i] < 0);	 		
//	}*/
//
//	checkCublasErrors( hipblasDasum(handle, N, d_eigv, 1, &S_sum), "sum S err in sub_partition!");
//		
//	subP = (N + S_sum)/2;
//	subN = (N - S_sum)/2;
//
//	dQ = calculate_dQ_GPU(d_eigv, N, nnz, d_R, d_C, d_V, d_K, d_sumBG, m);
//	cout<< "after fine tune, dQ = "<<dQ<<endl;
//	
//	bool isSplit = (dQ > DQ_MIN && subP > MIN_GROUP && subN > MIN_GROUP);
//
//	cout<<"Divide?: "<<isSplit<<'\t';
//	fout<<"Divide?: "<<isSplit<<'\t';
//	// If not divided, return; otherwise update Result and Max_Result
//	double *S = new double [N];
//	if (isSplit)
//	{
//		checkCudaErrors( hipMemcpy( S, d_eigv, sizeof(double) * N, hipMemcpyDeviceToHost)) ;
//		(*Num_module) += 1;
//		if (subP>subN)
//			for (i = 0; i < N; i++)
//				Result[ind[i]] = ((S[i] >= 0 ) ? Result[ind[i]] : (*Num_module));
//		else
//			for (i = 0; i < N; i++)
//				Result[ind[i]] = ((S[i] < 0 ) ? Result[ind[i]] : (*Num_module));
//	}
//	
//	delete []S;
//	hipblasDestroy(handle);
//
//
//	delete []Vd;
//	checkCudaErrors(hipFree(d_eigv));
//	
//	checkCudaErrors(hipFree(d_R));
//	checkCudaErrors(hipFree(d_C));
//	checkCudaErrors(hipFree(d_V));
//	checkCudaErrors(hipFree(d_sumBG));
//	checkCudaErrors(hipFree(d_K));
//	return isSplit;
//}

bool Sub_Partition_GPU_test(int N, int * ind, int * R, int * C, float *V, double *K, double *sumBG, double m, int * Result, int * Num_module)
{
		
	checkCublasErrors(hipblasCreate(&handle), "Create cublas handle err!\n");
	checkCusparseErrors( hipsparseCreate(&s_handle), "Create cusparse handle err in calculate dQ function" );
	checkCusparseErrors(hipsparseCreateMatDescr(&s_descr), "CreateMatDescr err in calculate dQ function"); 
	hipsparseSetMatType(s_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(s_descr,HIPSPARSE_INDEX_BASE_ZERO); 

	int i,j;
	int nnz = R[N]-R[0];
	double *Vd = new double [nnz];
	for (i = 0; i < nnz; i++)
		Vd[i] = (double) V[i];
	
	int *d_R, *d_C;
	double *d_V;
	double *d_K, *d_sumBG;

	checkCudaErrors( hipMalloc( (void**) &d_R, sizeof(int) * (N + 1)));
	checkCudaErrors( hipMalloc( (void**) &d_C, sizeof(int) * nnz));
	checkCudaErrors( hipMalloc( (void**) &d_V, sizeof(double) * nnz));
	checkCudaErrors( hipMalloc( (void**) &d_sumBG, sizeof(double) * (N)));
	checkCudaErrors( hipMalloc( (void**) &d_K, sizeof(double) * (N)));
	
	checkCudaErrors( hipMemcpy( d_R, R, sizeof(int) * (N + 1), hipMemcpyHostToDevice)) ;
	checkCudaErrors( hipMemcpy( d_C, C, sizeof(int) * nnz, hipMemcpyHostToDevice)) ;
	checkCudaErrors( hipMemcpy( d_V, Vd, sizeof(double) * nnz, hipMemcpyHostToDevice)) ;
	checkCudaErrors( hipMemcpy( d_K, K, sizeof(double) * N, hipMemcpyHostToDevice)) ;
	checkCudaErrors( hipMemcpy( d_sumBG, sumBG, sizeof(double) * N, hipMemcpyHostToDevice)) ;

	double * d_eigv;
	checkCudaErrors( hipMalloc( (void**) &d_eigv, sizeof(double) * N));
	
	double * eigv = new double [N];
	//signed char *S = new signed char [N];
	double lambda = 0;
	lambda = Lead_Vector_GPU(N, nnz, d_R, d_C, d_V, d_K,  d_sumBG, m, BETA_Adjust, d_eigv);
	//lambda = Lead_Vector(N, R, C, V, K,  sumBG, m, BETA_Adjust, eigv);
	lambda += BETA_Adjust;
	// If lambda < 0, calucate the leading eigenvalue for  B - lambda * I
	if (lambda < 0)
		lambda += Lead_Vector_GPU(N, nnz, d_R, d_C, d_V, d_K,  d_sumBG, m, lambda, d_eigv);
		//lambda += Lead_Vector(N, R, C, V, K,  sumBG, m, lambda, eigv);
		
	cout<<"Eigen Value: "<<lambda<<'\t';
	fout<<"Eigen Value: "<<lambda<<'\t';

	// Decide whether this round of partition is successful 
	int subN = 0, subP = 0;
		
	sign_eigv<<<blocknum,threadnum>>>(N,d_eigv);
	double S_sum = 0;
	thrust::device_ptr<double> dev_ptr(d_eigv);
	S_sum = thrust::reduce(dev_ptr, dev_ptr+N, (double) 0, thrust::plus<double>());
	
	subP = (N + S_sum)/2;
	subN = (N - S_sum)/2;

	cout<<"subP: "<<subP<<endl;
	cout<<"subN: "<<subN<<endl;

	
	if (subP+subN != N)
	{
		cout<<"S allocation error!";
		system("pause");
	}
	//calculate dQ;
	double dQ = 0;
	//checkCudaErrors( hipMemcpy( d_eigv, eigv, sizeof(double) * N, hipMemcpyHostToDevice)) ;
	dQ = calculate_dQ_GPU(d_eigv, N, nnz, d_R, d_C, d_V, d_K, d_sumBG, m);
	cout<< "dQ = "<<dQ<<endl;
	//dQ = calculate_dQ(S, N, R, C, V, K, sumBG, m);
	//cout<< "dQ = "<<dQ<<endl;
	
	/////////////////////////////////////////////////////////////////////
	//                        fine tune results                        //
	if (dQ>DQ_MIN)  
	{
		 dQ = fine_tune_S_GPU(dQ, N, nnz, d_R, d_C, d_V, d_K, m, d_eigv);	
	}
			
	//dQ = calculate_dQ_GPU(d_eigv, N, nnz, d_R, d_C, d_V, d_K, d_sumBG, m);
	cout<< "after fine tune, dQ = "<<dQ<<endl;

	S_sum = thrust::reduce(dev_ptr, dev_ptr+N, (double) 0, thrust::plus<double>());
	subP = (N + S_sum)/2;
	subN = (N - S_sum)/2;
	cout<<"subP: "<<subP<<endl;
	cout<<"subN: "<<subN<<endl;
	////////////////////////////////////////////////////////////////////////


	bool isSplit = (dQ > DQ_MIN && subP > MIN_GROUP && subN > MIN_GROUP);

	cout<<"Divide?: "<<isSplit<<'\t';
	fout<<"Divide?: "<<isSplit<<'\t';
	// If not divided, return; otherwise update Result and Max_Result
	/*double *S = new double [N];
	if (isSplit)
	{
		checkCudaErrors( hipMemcpy( S, d_eigv, sizeof(double) * N, hipMemcpyDeviceToHost)) ;
		(*Num_module) += 1;
		if (subP>subN)
			for (i = 0; i < N; i++)
				Result[ind[i]] = ((S[i] >= 0 ) ? Result[ind[i]] : (*Num_module));
		else
			for (i = 0; i < N; i++)
				Result[ind[i]] = ((S[i] < 0 ) ? Result[ind[i]] : (*Num_module));
	}
	
	delete []S;
	hipblasDestroy(handle);*/

	if (isSplit)
	{
		checkCudaErrors( hipMemcpy( eigv, d_eigv, sizeof(double) * N, hipMemcpyDeviceToHost)) ;
		(*Num_module) += 1;
		if (subP>subN)
			for (i = 0; i < N; i++)
				Result[ind[i]] = ((eigv[i] >= 0 ) ? Result[ind[i]] : (*Num_module));
		else
			for (i = 0; i < N; i++)
				Result[ind[i]] = ((eigv[i] < 0 ) ? Result[ind[i]] : (*Num_module));
	}
	//delete []S;
	delete []eigv;
	delete []Vd;
	
	checkCudaErrors(hipFree(d_eigv));
	
	checkCudaErrors(hipFree(d_R));
	checkCudaErrors(hipFree(d_C));
	checkCudaErrors(hipFree(d_V));
	checkCudaErrors(hipFree(d_sumBG));
	checkCudaErrors(hipFree(d_K));
	hipblasDestroy(handle) ;
	hipsparseDestroy(s_handle) ;
	return isSplit;
}


/* 
This function does the partition, no return value.
R and C represent the adjacency matrix in CSR format.
Result stores the partition results.
*/
//void Partition_GPU(long long N, int * R, int * C, float * V, int * Result)
//{
//	long long M = R[N];
//	long long i = 0,j = 0;
//	
//	double * K;
//	K = new double [N];
//	double m = 0;
//	memset(K, 0, sizeof(double)*N);
//	for (i = 0; i < N; i++)
//	{
//		for (j = R[i]; j < R[i+1]; j++)
//		{
//			K[i]+= V[j];
//		}
//		m += K[i];
//	}
//	//double m_inv = 1.0/m;
//	cout<<"check K: "<<K[0]<<'\t'<<K[N/2]<<'\t'<<K[N-1]<<endl; //check
//	cout<<"check m: "<<m<<endl;
//	//memset(Result, 0, sizeof(int) * N);
//	fill(Result, Result+N, 1); //initialize Result
//	//int * Adjust_Result = new int [N];   //can be optimized,old version bfs style
//	int result_idx = 1;
//	int Num_module = 1;
//	int NumG = N;
//	int * index = new int [N];
//	
//	int * ind = new int [N];            //later try to put in the loop, using NumG
//	for (i = 0; i < N; i++)
//		ind[i] = i;
//	//int NumG = 0;
//	int Round = 1;						// The iteration round
//	//int Max_Result = 1;					// Maximum index of modules
//	int * R_new = new int [N+1];
//	memcpy(R_new, R, sizeof(int)*(N+1));
//	int * C_new = new int [M];
//	memcpy(C_new, C, sizeof(int)*(M));
//	float *V_new = new float [M];
//	memcpy(V_new,V,sizeof(float)*M);
//	double *K_new = new double [N];
//	memcpy(K_new, K, sizeof(double)*N);
//	double *sumBG = new double [N];
//	memset(sumBG, 0, sizeof(double)*N);
//	
//	bool isSplit;
//	int ITER = 0;
//	while (Round <= Num_module)
//	{		
//		/**********************************************************************************/
//		/************************************ Partition ***********************************/
//		if (NumG>1)
//		{
//			cout<<"\nRound:\t"<<Round<<'\t'<<"Iter:\t"<<ITER<<endl;
//			cout<<"number of nodes:\t"<<NumG<<'\t';
//			cout<<"number of non-zero elements:\t"<<R_new[NumG]<<'\t';
//			cout<<"density of this submodule:\t"<<R_new[NumG]*1.0/( (double) NumG * NumG)<<'\t';
//			fout<<"\nRound:\t"<<Round<<'\t';
//			fout<<"number of nodes:\t"<<NumG<<'\t';
//			fout.flush();
//			
//			Setup(1);
//			Start(1);
//			//isSplit = Sub_Partition(NumG, ind, R_new, C_new, V_new, K_new, sumBG, m, Result, &Num_module); //if split, Num_module+1;
//			isSplit = Sub_Partition_GPU_test(NumG, ind, R_new, C_new, V_new, K_new, sumBG, m, Result, &Num_module); //if split, Num_module+1;
//			Stop(1);
//			
//			cout<<"sub_partition time:   "<<GetElapsedTime(1)<<"s"<<endl;
//			fout<<"sub_partition time:   "<<GetElapsedTime(1)<<"s"<<endl;
//			//if (!isSplit)					// If divided, record the adjusted result
//			//	Adjust_Result[Round] = result_idx++;
//			//Num_module += isSplit;		// Update the total number modules , old version
//		}
//		if (!isSplit)
//			Round++;
//		/**********************************************************************************/
//		/************************** Find the next sub_module ***************************/
//		NumG = 0;
//		fill(index, index+N, -1);
//		for (i = 0; i < N; i++)
//		{
//			if (Result[i] == Round)
//			{	
//				ind[NumG] = i;
//				index[i] = NumG++;   // index[i] >= 0 if node i is involved in this round
//				//NumG ++;
//			}				
//		}
//		if (!NumG)
//		{
//			cout<<"No voxel in the submodule next round";
//			cout<<"iter: "<<ITER<<", and Num_module: "<<Num_module<<" should be equal.\n";
//			continue;
//		}
//		int ii = 0;
//		int jj = 0;
//		R_new[0] = 0;
//		for(i = 0;i < N; i++)
//		{
//			if(index[i] < 0)
//				continue;
//			K_new[ii] = K[i];                 
//			for (j = R[i];j < R[i+1];j++)
//			{
//				if(index[C[j]] < 0)
//					continue;
//				C_new[jj] = index[C[j]];
//				V_new[jj] = V[j];
//				if(C_new[jj] > NumG)                      //check flag
//					cout<<C_new[jj]<<'\t'<<C[j]<<"C_new exceed NumG!\n";				
//				jj++;
//			}
//			R_new[++ii] = jj;			
//		}
//		if (ii!=NumG)    //check flag
//			cout<<"sub module voxel# not match!";
//
//		//update R_new, C_new, V_new, and k (i.e., bsub);
//		/**********************************************************************************/
//		/******************************** diag(sum(bsub)) *********************************/
//		double temp1 = 0, temp2 = 0;
//		for (i = 0; i < NumG; i++)
//				temp2 += (K_new[i]);
//
//		for (i = 0; i < NumG; i++)
//		{
//			//if (!G[i])
//				//continue;
//			//sumBG[i] = 0;
//			temp1 = 0;
//			for (j = R_new[i]; j < R_new[i+1]; j++)
//				temp1 += V_new[j];			
//			sumBG[i] = temp1 - K_new[i] * temp2 / m;
//		}
//		
//		ITER++;
//
//		//ofstream debugfile;
//		//ostringstream s1;
//		//
//		//s1<<"Round"<<Round<<"_Iter"<<ITER;
//		////string debugfilename = "round";
//		//debugfile.open(s1.str().append("_ind"),ios::binary|ios::out);
//		//debugfile.write((char *)ind, NumG*sizeof(int));
//		//debugfile.close();
//		//int Rlength = NumG+1;
//		//int Clength = R_new[NumG];
//		//debugfile.open(s1.str().append("_csr"),ios::binary|ios::out);
//		//debugfile.write((char *)&Rlength, sizeof(int));
//		//debugfile.write((char *)R_new, Rlength*sizeof(int));
//		//debugfile.write((char *)&Clength, sizeof(int));
//		//debugfile.write((char *)C_new, R_new[NumG]*sizeof(int));
//		//debugfile.write((char *)&Clength, sizeof(int));
//		//debugfile.write((char *)V_new, R_new[NumG]*sizeof(float));
//		//debugfile.close();
//		//debugfile.open(s1.str().append("_k"),ios::binary|ios::out);
//		//debugfile.write((char *)K_new, NumG*sizeof(double));
//		//debugfile.close();
//		//debugfile.open(s1.str().append("_diag"),ios::binary|ios::out);
//		//debugfile.write((char *)sumBG, NumG*sizeof(double));
//		//debugfile.close();
//
//		
//	}
//	
//	double Q = 0;
//	for (i = 0; i < N; i++)
//		for (j = R[i]; j < R[i+1]; j++)
//			Q += V[j] * (Result[i] == Result[C[j]]);
//	for (i = 0; i < N; i++)
//		for (j = 0; j < N; j++)
//			Q -= (Result[i] == Result[j]) * (K[i]) * (K[j]) /m;
//	Q = Q / m;
//
//	cout<<"\nNumber of Modules: "<<Num_module<<",\tQ="<<Q<<endl;
//	fout<<"\nNumber of Modules: "<<Num_module<<",\tQ="<<Q<<endl;
//
//	//for (i = 0; i < N; i++)	
//	//	Result[i] = Adjust_Result[Result[i]];
//	
//	//delete []Adjust_Result;
//	delete []R_new;
//	delete []C_new;
//	delete []V_new;
//	delete []K_new;
//	delete []K;
//	delete []sumBG;
//	delete []ind;
//	delete []index;
//}
